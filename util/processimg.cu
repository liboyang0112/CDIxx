#include "hip/hip_runtime.h"
#include <complex>
#include <cassert>
#include <stdio.h>
#include <time.h>
#include <stdio.h>
#include "imgio.h"
#include "cudaConfig.h"
#include "experimentConfig.h"
#include "cuPlotter.h"
#include "cub_wrap.h"

int main(int argc, char** argv )
{
  hipFree(0); // to speed up the cuda malloc; https://forums.developer.nvidia.com/t/cudamalloc-slow/40238
  int nmerge = atoi(argv[3]);
  int row, col, row1, col1;
  Real* bkg = readImage(argv[1], row, col);
  size_t sz = row*col*sizeof(Real);
  myCuDMalloc(Real,d_bkg,sz);
  hipMemcpy(d_bkg, bkg, sz, hipMemcpyHostToDevice);
  ccmemMngr.returnCache(bkg);
  Real* sig = readImage(argv[2], row1, col1);
  sz = row1*col1*sizeof(Real);
  myCuDMalloc(Real, d_sig, sz);
  myCuDMalloc(Real, d_stretched, sz);
  hipMemcpy(d_sig, sig, sz, hipMemcpyHostToDevice);
  ccmemMngr.returnCache(sig);
  init_cuda_image(rcolor, 1);
  if(row!=row1){
    if(row > row1){
      myCuDMalloc(Real, tmp, row1*col1);
      resize_cuda_image(row1, col1);
      crop(d_bkg, tmp, row, col);
      memMngr.returnCache(d_bkg);
      d_bkg = tmp;
      row = row1;
      col = col1;
    }else{
      myCuDMalloc(Real, tmp, row*col);
      resize_cuda_image(row, col);
      crop(d_sig, tmp, row1, col1);
      memMngr.returnCache(d_sig);
      d_sig = tmp;
    }
  }else{
      resize_cuda_image(row, col);
  }

  add(d_sig, d_bkg, -1);
  //plt.init(row, col);
  //plt.plotFloat(d_sig, MOD, 0, 1, "logimage", 1);
  complexFormat mid;
  myCuDMalloc(Real, d_bit, sz);
  myCuDMalloc(Real, d_mask, sz);
  rect spt;
  spt.startx = row/2-100;
  spt.starty = col/2-100;
  spt.endx = row/2+100;
  spt.endy = col/2+100;
  myCuDMalloc(rect, cuda_spt, 1);
  hipMemcpy(cuda_spt, &spt, sizeof(rect), hipMemcpyHostToDevice);
  hipMemcpy(d_bit, d_sig, sz, hipMemcpyDeviceToDevice);
  createMask(d_mask, cuda_spt);
  applyMask(d_bit, d_mask);
  applyThreshold(d_bit, d_sig, 0.2);
  //plt.plotFloat(d_bit, MOD, 0, 1, "bit", 1);
  mid = findMiddle(d_bit, row*col);
  memMngr.returnCache(d_bit);
  memMngr.returnCache(d_mask);
  memMngr.returnCache(cuda_spt);
  if(argc >= 7){
    mid.y -= std::stof(argv[6])/row;
    mid.x += std::stof(argv[7])/col;
  }
  int step = nmerge*4;
  int outrow = (row-int(abs(mid.x)*row)*2)/step*step;
  int outcol = (col-int(abs(mid.y)*col)*2)/step*step;
  //int outrow = row/step*step;
  //int outcol = col/step*step;
  outrow = outcol = min(outrow,outcol);
  resize_cuda_image(outrow, outcol);
  myCuDMalloc(Real, tmp, outrow*outcol);
  printf("mid= %f,%f\n",mid.x,mid.y);
  Real shiftx = int(mid.x*row)-mid.x*row;
  Real shifty = int(mid.y*col)-mid.y*col;
  printf("shift = %f,%f\n",shiftx, shifty);
  crop(d_sig, tmp, row, col,mid.x,mid.y);
  //crop(d_sig, tmp, row, col);
  myCuDMalloc(complexFormat, tmp1, outrow*outcol);
  extendToComplex(tmp, tmp1);
  init_fft(outrow,outcol);
  if(argv[5][0]=='1') shiftMiddle(tmp1);
  else shiftWave(tmp1, shiftx, shifty);
  getReal(tmp, tmp1);
  int finsize = outrow/nmerge;
  resize_cuda_image(finsize,finsize);
  mergePixel(d_sig, tmp, outrow, outcol, nmerge);
  plt.init(finsize, finsize);
  myCuDMalloc(complexFormat, xc, finsize*finsize);
  init_fft(finsize,finsize);
  extendToComplex(d_sig,xc);
  myFFT(xc, xc);
  stretch(d_sig, d_stretched, 1.9, 50);
  plt.plotFloat(d_sig, MOD, 0, 1, "logimagemerged", 1, 0, 1);
  plt.plotFloat(d_stretched, MOD, 0, 4, "logimagemerged_str", 1, 0, 1);
  plt.plotComplex(xc, MOD2, 1, 1./finsize, "autocorrelation", 1, 0, 1);
  plt.saveFloat(d_sig, argv[4]);
  return 0;
}

