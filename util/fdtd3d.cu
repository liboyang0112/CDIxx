#include "hip/hip_runtime.h"
#include <fstream>
#include <cstring>
#include <math.h>
#include "cudaConfig.h"
#include "memManager.h"
#include "cuPlotter.h"
using namespace std;

__global__ void updateH(Real* Hx, Real* Hy, Real* Hz, Real* Ex, Real* Ey, Real* Ez, int nx, int ny, int nz)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if(x >= nx || y >= ny || z >= nz) return;
  int index = x + nx*y + nx*ny*z;
  Real mH = 0.5;
  if(z < nz-1 && x > 0 && y < ny-1){
    Hx[index] -= mH*(Ez[index+nx]-Ez[index]-Ey[index+nx*ny]+Ey[index]);
  }
  if(z < nz-1 && x < nx-1 && y > 0){
    Hy[index] -= mH*(Ex[index+nx*ny]-Ex[index]-Ez[index+1]+Ez[index]); //dEz/dy
  }
  if(z > 0 && x < nx-1 && y < ny-1){
    Hz[index] -= mH*(Ey[index+1]-Ey[index]-Ex[index+nx]+Ex[index]); //dEz/dx
  }
}
__global__ void updateE(Real* Hx, Real* Hy, Real* Hz, Real* Ex, Real* Ey, Real* Ez, int nx, int ny, int nz)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if(x >= nx || y >= ny || z >= nz) return;
  int index = x + nx*y + nx*ny*z;
  Real mE = 0.5;
  if(z > 0 && x < nx-1 && y > 0){
    Ex[index] += mE*(Hz[index]-Hz[index-nx]-Hy[index]+Hy[index-nx*ny]); //dEz/dy
  }
  if(z > 0 && x > 0 && y > 0){
    Ey[index] += mE*(Hx[index]-Hx[index-nx*ny]-Hz[index]+Hz[index-1]);
  }
  if(z < nz-1 && x > 0 && y > 0){
    Ez[index] += mE*(Hy[index]-Hy[index-1]-Hx[index]+Hx[index-nx]);
  }
}
__global__ void applyPMLx1Ez(Real* Hy, Real* Ez, Real* Ezprevx1, int nx, int ny, int nz){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(y >= ny || z >= nz) return;
  int edgeIdx = z*nx*ny + y*nx + nx - 1;  //large stride, unavoidable
  Real a = Ez[edgeIdx];
  Hy[edgeIdx-1] += (Ezprevx1[y+z*ny] + a)/2;
  Ezprevx1[y+z*ny] = a;
  Ez[edgeIdx] = 0;
}
__global__ void applyPMLx1Ey(Real* Hz, Real* Ey, Real* Eyprevx1, int nx, int ny, int nz){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(y >= ny || z >= nz) return;
  int edgeIdx = z*nx*ny + y*nx + nx - 1;  //large stride, unavoidable
  Real a = Ey[edgeIdx];
  Hz[edgeIdx-1] -= (Eyprevx1[y+z*ny] + a)/2;
  Eyprevx1[y+z*ny] = a;
  Ey[edgeIdx] = 0;
}
__global__ void applyPMLx0Hz(Real* Hz, Real* Ey, Real* Hzprevx0, int nx, int ny, int nz){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(y >= ny || z >= nz) return;
  int edgeIdx = z*nx*ny + y*nx+5;  //large stride, unavoidable
  Real a = Hz[edgeIdx];
  Ey[edgeIdx+1] += (Hzprevx0[y+z*ny] + a)/2;
  Hzprevx0[y+z*ny] = a;
  Hz[edgeIdx] = 0;
}
__global__ void applyPMLx0Hy(Real* Hy, Real* Ez, Real* Hyprevx0, int nx, int ny, int nz){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(y >= ny || z >= nz) return;
  int edgeIdx = z*nx*ny + y*nx+5;  //large stride, unavoidable
  Real a = Ez[edgeIdx];
  Hy[edgeIdx] -= (Hyprevx0[y+z*ny] + a)/2;
  Hyprevx0[y+z*ny] = a;
  Ez[edgeIdx] = 0;
}
__global__ void applyPMLy1Ez(Real* Hx, Real* Ez, Real* Ezprevy1, int nx, int ny, int nz){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || z >= nz) return;
  int edgeIdx = nx*ny*z + nx*(ny-1)+ x;
  Real a = Ez[edgeIdx];
  Hx[edgeIdx-nx] -= (Ezprevy1[x+nx*z] + a)/2;
  Ezprevy1[x+nx*z] = a;
  Ez[edgeIdx] = 0;
}
__global__ void applyPMLy1Ex(Real *Hz, Real* Ex, Real* Exprevy1, int nx, int ny, int nz){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || z >= nz) return;
  int edgeIdx = nx*ny*z + nx*(ny-1)+ x;
  Real a = Ex[edgeIdx];
  Hz[edgeIdx-nx] += (Exprevy1[x+nx*z] + a)/2;
  Exprevy1[x+nx*z] = a;
  Ex[edgeIdx] = 0;
}
__global__ void applyPMLy0Hz(Real* Hz, Real* Ex, Real* Hzprevy0, int nx, int ny, int nz){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || z >= nz) return;
  int edgeIdx = nx*ny*z + x;
  Real a = Hz[edgeIdx];
  Ex[edgeIdx+nx] -= (Hzprevy0[x+nx*z] + a)/2;
  Hzprevy0[x+nx*z] = a;
  Hz[edgeIdx] = 0;
}
__global__ void applyPMLy0Hx(Real *Hx, Real* Ez, Real* Hxprevy0, int nx, int ny, int nz){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || z >= nz) return;
  int edgeIdx = nx*ny*z + x;
  Real a = Hx[edgeIdx];
  Ez[edgeIdx+nx] += (Hxprevy0[x+nx*z] + a)/2;
  Hxprevy0[x+nx*z] = a;
  Hx[edgeIdx] = 0;
}
__global__ void applyPMLz1Ex(Real* Hy, Real* Ex, Real* Exprevz1, int nx, int ny, int nz){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || y >= ny) return;
  int edgeIdx = nx*ny*(nz-1) + nx*y + x;
  Real a = Ex[edgeIdx];
  Hy[edgeIdx-nx*ny] -= (Exprevz1[x+nx*y] + a)/2;
  Exprevz1[x+nx*y] = a;
  Ex[edgeIdx] = 0;
}
__global__ void applyPMLz1Ey(Real* Hx, Real* Ey, Real* Eyprevz1, int nx, int ny, int nz){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || y >= ny) return;
  int edgeIdx = nx*ny*(nz-1) + nx*y + x;
  Real a = Ey[edgeIdx];
  Hx[edgeIdx-nx*ny] += (Eyprevz1[x+nx*y] + a)/2;
  Eyprevz1[x+nx*y] = a;
  Ey[edgeIdx] = 0;
}
__global__ void applyPMLz0Hx(Real* Hx, Real* Ey, Real* Hxprevz0, int nx, int ny, int nz){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || y >= ny) return;
  int edgeIdx = nx*y + x;
  Real a = Hx[edgeIdx];
  Ey[edgeIdx+nx*ny] -= (Hxprevz0[x+nx*y] + a)/2;
  Hxprevz0[x+nx*y] = a;
  Hx[edgeIdx] = 0;
}
__global__ void applyPMLz0Hy(Real* Hy, Real* Ex, Real* Hyprevz0, int nx, int ny, int nz){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || y >= ny) return;
  int edgeIdx = nx*y + x;
  Real a = Hy[edgeIdx];
  Ex[edgeIdx+nx*ny] += (Hyprevz0[x+nx*y] + a)/2;
  Hyprevz0[x+nx*y] = a;
  Hy[edgeIdx] = 0;
}
__global__ void applyPeriodicx_H(Real* Hx, Real* Hy, int nx){
  int y =threadIdx.x;
  Hx[y*nx] = Hx[(y+1)*nx-1];
  y++;
  Hy[(y+1)*nx-1] = Hy[y*nx];
}
__global__ void applyPeriodicx_E(Real* Ez, int nx){
  int y =threadIdx.x+1;
  Ez[y*nx] = Ez[(y+1)*nx-1];
}
__global__ void applyPeriodicy_H(Real* Hx, Real* Hy, int displace){
  int x =threadIdx.x;
  Hy[x] = Hy[x+displace];
  x++;
  Hx[x+displace] = Hx[x];
}
__global__ void applyPeriodicy_E(Real* Ez, int displace){
  int x =threadIdx.x+1;
  Ez[x] = Ez[x+displace];
}
__global__ void applySource(Real* Ez, size_t idx, Real val){
  Ez[idx] += val;
}
__global__ void applySourceV(Real* Ez, Real* Hy, int nx, int pos, Real val, Real val1){
  int y = threadIdx.x;
  int idx = y*nx + pos;
  Ez[idx] += val;
  Hy[idx-1] += val1;
}
cuFunc(getXYSlice,(Real* slice, Real* data, int nx, int ny, int iz), (slice, data, nx, ny, iz), {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || y >= ny) return;
  int index = x + nx*y;
  slice[index] = data[index+iz*nx*ny];
})
cuFunc(getXZSlice,(Real* slice, Real* data, int nx, int ny, int nz, int iy), (slice, data, nx, ny, nz, iy), {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || z >= nz) return;
  int index = x + nx*z;
  slice[index] = data[x+nx*iy+nx*ny*z];
})
int main(){
  int nsteps = 5000;
  const int nx = 200;
  const int ny = 200;
  const int nz = 200;
  dim3 nblk, nthd, nblkx,nblky,nblkz, nthd2d;
  nthd.x = 256;
  nthd.y = 1;
  nthd.z = 1;
  nthd2d.x = 256;
  nthd2d.y = 1;
  nblk.x = (nx-1)/nthd.x+1;
  nblk.y = (ny-1)/nthd.y+1;
  nblk.z = (nz-1)/nthd.z+1;
  nblkx.x = (ny-1)/nthd2d.x+1;
  nblkx.y = (nz-1)/nthd2d.y+1;
  nblky.x = (nx-1)/nthd2d.x+1;
  nblky.y = (nz-1)/nthd2d.y+1;
  nblkz.x = (nx-1)/nthd2d.x+1;
  nblkz.y = (ny-1)/nthd2d.y+1;

  size_t nnode = nx*ny*nz;
  size_t memsz = nnode*sizeof(Real);
  Real* Hz = (Real*)memMngr.borrowCleanCache(memsz);
  Real* Ex = (Real*)memMngr.borrowCleanCache(memsz);
  Real* Ey = (Real*)memMngr.borrowCleanCache(memsz);
  Real* Ez = (Real*)memMngr.borrowCleanCache(memsz);
  Real* Hx = (Real*)memMngr.borrowCleanCache(memsz);
  Real* Hy = (Real*)memMngr.borrowCleanCache(memsz);
  //record boundaries for PML
  Real* Ezprevx1 = (Real*)memMngr.borrowCleanCache(ny*nz*sizeof(Real));
  Real* Eyprevx1 = (Real*)memMngr.borrowCleanCache(ny*nz*sizeof(Real));
  Real* Ezprevy1 = (Real*)memMngr.borrowCleanCache(nz*nx*sizeof(Real));
  Real* Exprevy1 = (Real*)memMngr.borrowCleanCache(nz*nx*sizeof(Real));
  Real* Exprevz1 = (Real*)memMngr.borrowCleanCache(nx*ny*sizeof(Real));
  Real* Eyprevz1 = (Real*)memMngr.borrowCleanCache(nx*ny*sizeof(Real));
  Real* Hzprevx0 = (Real*)memMngr.borrowCleanCache(ny*nz*sizeof(Real));
  Real* Hyprevx0 = (Real*)memMngr.borrowCleanCache(ny*nz*sizeof(Real));
  Real* Hzprevy0 = (Real*)memMngr.borrowCleanCache(nz*nx*sizeof(Real));
  Real* Hxprevy0 = (Real*)memMngr.borrowCleanCache(nz*nx*sizeof(Real));
  Real* Hxprevz0 = (Real*)memMngr.borrowCleanCache(nx*ny*sizeof(Real));
  Real* Hyprevz0 = (Real*)memMngr.borrowCleanCache(nx*ny*sizeof(Real));
  Real* slice = (Real*)memMngr.borrowCache(nx*ny*sizeof(Real));
  bool saveField = 0;
  int sourcePos = 100+nx*100 + nx*ny*100;
  resize_cuda_image(ny,nx);
  plt.init(ny,nx);
  init_cuda_image();
  
  int ezvid = plt.initVideo("Ez.mp4v");
  int hxvid = plt.initVideo("Hx.mp4v");
  int hyvid = plt.initVideo("Hy.mp4v");
  for(int i = 0; i < nsteps; i++){
    saveField = i%5==0;
    applySource<<<1,1>>>(Ez, sourcePos, 500*sin(M_PI/30*i));//50*exp(-pow(double(i-100)/30,2))); 
    //point source
    //applySourceV<<<1,ny>>>(Ez, Hy, nx, 50, exp(-pow(double(i-100)/30,2)), -exp(-pow(double(i-99.5)/30,2))); //plain wave source
    applyPMLx0Hy<<<nblkx,nthd2d>>>(Hy, Ez, Hyprevx0, nx, ny, nz);
    //applyPMLx1Ez<<<nblkx,nthd2d>>>(Hy, Ez, Ezprevx1, nx, ny, nz);
    //applyPMLy0Hx<<<nblky,nthd2d>>>(Hx, Ez, Hxprevy0, nx, ny, nz);
    //applyPMLy1Ez<<<nblky,nthd2d>>>(Hx, Ez, Ezprevy1, nx, ny, nz);
    //applyPMLz0Hy<<<nblky,nthd2d>>>(Hy, Ex, Hyprevz0, nx, ny, nz);
    //applyPMLz1Ey<<<nblky,nthd2d>>>(Hx, Ey, Eyprevz1, nx, ny, nz);

    updateH<<<nblk,nthd>>>(Hx, Hy, Hz, Ex, Ey, Ez, nx, ny, nz);    //------------UPDATE H-----------
    applyPMLx0Hz<<<nblkx,nthd2d>>>(Hz, Ey, Hzprevx0, nx, ny, nz);
    //applyPMLx1Ey<<<nblkx,nthd2d>>>(Hz, Ey, Eyprevx1, nx, ny, nz);
    //applyPMLy0Hz<<<nblky,nthd2d>>>(Hz, Ex, Hzprevy0, nx, ny, nz);
    //applyPMLy1Ex<<<nblky,nthd2d>>>(Hz, Ex, Exprevy1, nx, ny, nz);
    //applyPMLz0Hx<<<nblky,nthd2d>>>(Hx, Ey, Hxprevz0, nx, ny, nz);
    //applyPMLz1Ex<<<nblky,nthd2d>>>(Hy, Ex, Exprevz1, nx, ny, nz);

    //applyPMLy0<<<1,nx>>>(Hx, Hy, Ez, Hprevy0, nx);
    //applyPeriodicx_H<<<1,ny-1>>>(Hx, Hy, nx);
    //applyPeriodicy_H<<<1,nx-1>>>(Hx, Hy, nx*(ny-1));
    updateE<<<nblk,nthd>>>(Hx, Hy, Hz, Ex, Ey, Ez, nx, ny, nz);
    if(i==nsteps-1){
      getXZSlice(slice, Ey , nx, ny, nz, 50);
      //getXYSlice(slice, Ez , nx, ny, 100);
      plt.toVideo = -1;
      plt.plotFloat(slice, REAL, 0, 1, "Eylast",0,0,1);
    }
    if(saveField) {
      getXYSlice(slice, Ez , nx, ny, 50);
      //getXZSlice(slice, Ey , nx, ny, nz, 50);
      plt.toVideo = ezvid;
      plt.plotFloat(slice, REAL, 0, 1, "",0,0,1);
      //getXYSlice(slice, Hx , nx, ny, 50);
      //plt.toVideo = hxvid;
      //plt.plotFloat(slice, REAL, 0, 1, ("Hx"+to_string(i)).c_str(),0,0,1);
      //getXYSlice(slice, Hx , nx, ny, 50);
      //plt.toVideo = hyvid;
      //plt.plotFloat(slice, REAL, 0, 1, ("Hy"+to_string(i)).c_str(),0,0,1);
    }
  }
}
