#include "hip/hip_runtime.h"
#include <fstream>
#include <cstring>
#include <math.h>
#include "cudaConfig.h"
#include "memManager.h"
#include "cuPlotter.h"
using namespace std;

__global__ void updateH(Real* Hx, Real* Hy, Real* Hz, Real* Ex, Real* Ey, Real* Ez, int nx, int ny, int nz)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if(x >= nx || y >= ny || z >= nz) return;
  int index = x + nx*y + nx*ny*z;
  Real mH = 0.5;
  if(z < nz-1 && x > 0 && y < ny-1){
    Hx[index] -= mH*(Ez[index+nx]-Ez[index]-Ey[index+nx*ny]+Ey[index]);
  }
  if(z < nz-1 && x < nx-1 && y > 0){
    Hy[index] -= mH*(Ex[index+nx*ny]-Ex[index]-Ez[index+1]+Ez[index]); //dEz/dy
  }
  if(z > 0 && x < nx-1 && y < ny-1){
    Hz[index] -= mH*(Ey[index+1]-Ey[index]-Ex[index+nx]+Ex[index]); //dEz/dx
  }
}
__global__ void updateE(Real* Hx, Real* Hy, Real* Hz, Real* Ex, Real* Ey, Real* Ez, int nx, int ny, int nz)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if(x >= nx || y >= ny || z >= nz) return;
  int index = x + nx*y + nx*ny*z;
  Real mE = 0.5;
  if(z > 0 && x < nx-1 && y > 0){
    Ex[index] += mE*(Hz[index]-Hz[index-nx]-Hy[index]+Hy[index-nx*ny]); //dEz/dy
  }
  if(z > 0 && x > 0 && y > 0){
    Ey[index] += mE*(Hx[index]-Hx[index-nx*ny]-Hz[index]+Hz[index-1]);
  }
  if(z < nz-1 && x > 0 && y > 0){
    Ez[index] += mE*(Hy[index]-Hy[index-1]-Hx[index]+Hx[index-nx]);
  }
}
__global__ void applyPMLx1(Real* Hz, Real* Ey, Real* Hy, Real* Ez, Real* Ezprevx1, Real* Eyprevx1, int nx, int ny, int nz){
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(y >= ny || z >= nz) return;
  int edgeIdx = z*nx*ny + y*nx + nx - 1;  //large stride, unavoidable
  Real a = Ez[edgeIdx];
  Hy[edgeIdx-1] += (Ezprevx1[y+z*ny] + a)/2;
  Ezprevx1[y+z*ny] = a;
  Ez[edgeIdx] = 0;
  a = Ey[edgeIdx];
  Hz[edgeIdx] += (Eyprevx1[y+z*ny] + a)/2;
  Eyprevx1[y+z*ny] = a;
  Ey[edgeIdx] = 0;
}
__global__ void applyPMLy1(Real* Hx, Real* Ez, Real *Hz, Real* Ex, Real* Ezprevy1, Real* Hzprevy1, int nx, int ny, int nz){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || z >= nz) return;
  int edgeIdx = nx*ny*z + nx*(ny-1)+ x;
  Real a = Ez[edgeIdx];
  Hx[edgeIdx-nx] -= (Ezprevy1[x+nx*z] + a)/2;
  Ezprevy1[x+nx*z] = a;
  Ez[edgeIdx] = 0;
  a = Hz[edgeIdx];
  Ex[edgeIdx] -= (Hzprevy1[x+nx*z] + a)/2;
  Hzprevy1[x+nx*z] = a;
  Hz[edgeIdx] = 0;
}
__global__ void applyPMLz1(Real* Hx, Real* Ey, Real* Hy, Real* Ex, Real* Hyprevz1, Real* Eyprevz1, int nx, int ny, int nz){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || y >= ny) return;
  int edgeIdx = nx*ny*(nz-1) + nx*y + x;
  Real a = Ey[edgeIdx];
  Hx[edgeIdx-nx*ny] -= (Eyprevz1[x+nx*y] + a)/2;
  Eyprevz1[x+nx*y] = a;
  Ey[edgeIdx] = 0;
  a = Hy[edgeIdx];
  Ex[edgeIdx] -= (Hyprevz1[x+nx*y] + a)/2;
  Hyprevz1[x+nx*y] = a;
  Hy[edgeIdx] = 0;
}
__global__ void applyPeriodicx_H(Real* Hx, Real* Hy, int nx){
  int y =threadIdx.x;
  Hx[y*nx] = Hx[(y+1)*nx-1];
  y++;
  Hy[(y+1)*nx-1] = Hy[y*nx];
}
__global__ void applyPeriodicx_E(Real* Ez, int nx){
  int y =threadIdx.x+1;
  Ez[y*nx] = Ez[(y+1)*nx-1];
}
__global__ void applyPeriodicy_H(Real* Hx, Real* Hy, int displace){
  int x =threadIdx.x;
  Hy[x] = Hy[x+displace];
  x++;
  Hx[x+displace] = Hx[x];
}
__global__ void applyPeriodicy_E(Real* Ez, int displace){
  int x =threadIdx.x+1;
  Ez[x] = Ez[x+displace];
}
__global__ void applySource(Real* Ez, size_t idx, Real val){
  Ez[idx] += val;
}
__global__ void applySourceV(Real* Ez, Real* Hy, int nx, int pos, Real val, Real val1){
  int y = threadIdx.x;
  int idx = y*nx + pos;
  Ez[idx] += val;
  Hy[idx-1] += val1;
}
cuFunc(getXYSlice,(Real* slice, Real* data, int nx, int ny, int iz), (slice, data, nx, ny, iz), {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || y >= ny) return;
  int index = x + nx*y;
  slice[index] = data[index+iz*nx*ny];
})
cuFunc(getXZSlice,(Real* slice, Real* data, int nx, int ny, int nz, int iy), (slice, data, nx, ny, nz, iy), {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int z = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= nx || z >= nz) return;
  int index = x + nx*z;
  slice[index] = data[x+nx*iy+nx*ny*z];
})
int main(){
  int nsteps = 5000;
  const int nx = 200;
  const int ny = 200;
  const int nz = 200;
  dim3 nblk, nthd, nblkx,nblky,nblkz, nthd2d;
  nthd.x = 256;
  nthd.y = 1;
  nthd.z = 1;
  nthd2d.x = 256;
  nthd2d.y = 1;
  nblk.x = (nx-1)/nthd.x+1;
  nblk.y = (ny-1)/nthd.y+1;
  nblk.z = (nz-1)/nthd.z+1;
  nblkx.x = (ny-1)/nthd2d.x+1;
  nblkx.y = (nz-1)/nthd2d.y+1;
  nblky.x = (nx-1)/nthd2d.x+1;
  nblky.y = (nz-1)/nthd2d.y+1;
  nblkz.x = (nx-1)/nthd2d.x+1;
  nblkz.y = (ny-1)/nthd2d.y+1;

  size_t nnode = nx*ny*nz;
  size_t memsz = nnode*sizeof(Real);
  Real* Hz = (Real*)memMngr.borrowCleanCache(memsz);
  Real* Ex = (Real*)memMngr.borrowCleanCache(memsz);
  Real* Ey = (Real*)memMngr.borrowCleanCache(memsz);
  Real* Ez = (Real*)memMngr.borrowCleanCache(memsz);
  Real* Hx = (Real*)memMngr.borrowCleanCache(memsz);
  Real* Hy = (Real*)memMngr.borrowCleanCache(memsz);
  //record boundaries for PML
  Real* Ezprevx1 = (Real*)memMngr.borrowCleanCache(ny*nz*sizeof(Real));
  Real* Eyprevx1 = (Real*)memMngr.borrowCleanCache(ny*nz*sizeof(Real));
  Real* Ezprevy1 = (Real*)memMngr.borrowCleanCache(nz*nx*sizeof(Real));
  Real* Exprevy1 = (Real*)memMngr.borrowCleanCache(nz*nx*sizeof(Real));
  Real* Exprevz1 = (Real*)memMngr.borrowCleanCache(nx*ny*sizeof(Real));
  Real* Eyprevz1 = (Real*)memMngr.borrowCleanCache(nx*ny*sizeof(Real));
  Real* Hzprevx0 = (Real*)memMngr.borrowCleanCache(ny*nz*sizeof(Real));
  Real* Hyprevx0 = (Real*)memMngr.borrowCleanCache(ny*nz*sizeof(Real));
  Real* Hzprevy0 = (Real*)memMngr.borrowCleanCache(nz*nx*sizeof(Real));
  Real* Hxprevy0 = (Real*)memMngr.borrowCleanCache(nz*nx*sizeof(Real));
  Real* Hxprevz0 = (Real*)memMngr.borrowCleanCache(nx*ny*sizeof(Real));
  Real* Hyprevz0 = (Real*)memMngr.borrowCleanCache(nx*ny*sizeof(Real));
  Real* slice = (Real*)memMngr.borrowCache(nx*ny*sizeof(Real));
  bool saveField = 0;
  int sourcePos = 50+nx*50 + nx*ny*50;
  resize_cuda_image(ny,nx);
  plt.init(ny,nx);
  init_cuda_image();
  
  int ezvid = plt.initVideo("Ez.mp4v");
  int hxvid = plt.initVideo("Hx.mp4v");
  int hyvid = plt.initVideo("Hy.mp4v");
  for(int i = 0; i < nsteps; i++){
    saveField = i%5==0;
    applySource<<<1,1>>>(Ez, sourcePos, 500*sin(M_PI/30*i));//50*exp(-pow(double(i-100)/30,2))); 
    //point source
    //applySourceV<<<1,ny>>>(Ez, Hy, nx, 50, exp(-pow(double(i-100)/30,2)), -exp(-pow(double(i-99.5)/30,2))); //plain wave source
    applyPMLx1<<<nblkx,nthd2d>>>(Hz, Ey, Hy, Ez, Ezprevx1, Eyprevx1, nx, ny, nz);
    applyPMLy1<<<nblky,nthd2d>>>(Hx, Ez, Hz, Ex, Ezprevy1, Exprevy1, nx, ny, nz);
    applyPMLz1<<<nblkz,nthd2d>>>(Hy, Ex, Hx, Ey, Eyprevz1, Exprevz1, nx, ny, nz);
    //applyPeriodicy_E<<<1,nx-1>>>(Ez, nx*(ny-1));
    //applyPeriodicx_E<<<1,ny-1>>>(Ez, nx);
    updateH<<<nblk,nthd>>>(Hx, Hy, Hz, Ex, Ey, Ez, nx, ny, nz);
    //applyPMLx0<<<1,ny>>>(Hx, Hy, Ez, Hprevx0, nx);
    //applyPMLy0<<<1,nx>>>(Hx, Hy, Ez, Hprevy0, nx);
    //applyPeriodicx_H<<<1,ny-1>>>(Hx, Hy, nx);
    //applyPeriodicy_H<<<1,nx-1>>>(Hx, Hy, nx*(ny-1));
    updateE<<<nblk,nthd>>>(Hx, Hy, Hz, Ex, Ey, Ez, nx, ny, nz);
    if(i==nsteps-1){
      //getXZSlice(slice, Ey , nx, ny, nz, 50);
      getXYSlice(slice, Ez , nx, ny, 50);
      plt.toVideo = -1;
      plt.plotFloat(slice, REAL, 0, 1, "Eylast",0,0,1);
    }
    if(saveField) {
      getXYSlice(slice, Ez , nx, ny, 50);
      //getXZSlice(slice, Ey , nx, ny, nz, 50);
      plt.toVideo = ezvid;
      plt.plotFloat(slice, REAL, 0, 1, "",0,0,1);
      //getXYSlice(slice, Hx , nx, ny, 50);
      //plt.toVideo = hxvid;
      //plt.plotFloat(slice, REAL, 0, 1, ("Hx"+to_string(i)).c_str(),0,0,1);
      //getXYSlice(slice, Hx , nx, ny, 50);
      //plt.toVideo = hyvid;
      //plt.plotFloat(slice, REAL, 0, 1, ("Hy"+to_string(i)).c_str(),0,0,1);
    }
  }
}
