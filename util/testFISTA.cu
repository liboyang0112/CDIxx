#include <complex>
#include <cassert>
#include <stdio.h>
#include <time.h>
#include <random>

#include <stdio.h>
#include <libconfig.h++>
#include "hipfft/hipfft.h"
#include "common.h"
#include <ctime>
#include "cudaConfig.h"
#include "experimentConfig.h"
#include "cuPlotter.h"
#include "mnistData.h"
#include "tvFilter.h"

#include "cdi.h"

void applyC(Real* input, Real* output){
  hipMemcpy(output, input, memMngr.getSize(input), hipMemcpyDeviceToDevice);
}

int main(int argc, char** argv )
{
  CDI setups(argv[1]);
  hipFree(0); // to speed up the hipMalloc; https://forums.developer.nvidia.com/t/cudamalloc-slow/40238
  if(argc < 2){
    printf("please feed the object intensity and phase image\n");
  }
  setups.readFiles();
  setups.init();
  setups.prepareIter();
  FISTA(setups.patternData,setups.patternData, 0.003, 100, 0);
  plt.plotFloat(setups.patternData, MOD, 1, setups.exposure, "test",1);
  return 0;
}

