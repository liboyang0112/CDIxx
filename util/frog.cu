#include "hip/hip_runtime.h"
#include "cudaConfig.h" //cuda related
#include "cuPlotter.h" //plt
#include "cub_wrap.h"
#include <complex>
#include <fstream>
#define ALPHA 0.5
#define BETA 1
#define DELTA 1e-3
#define GAMMA 0.5
using namespace std;

Real* simTrace(int ndelay, int nspect){
  myDMalloc(Real, traces, nspect*ndelay);
  return traces;
}

cuFunc(dgenTrace, (Real* gate, Real* E, complexFormat* fulltrace, Real* delay), (gate, E, fulltrace,delay), {
  cudaIdx();
  int tidx = y-delay[x];
  if(tidx >= cuda_column || tidx < 0) {
    fulltrace[index].x = 0;
  }else
    fulltrace[index].x = gate[tidx] * E[y];
  fulltrace[index].y = 0;
})

cuFunc(dgencTrace, (complexFormat* gate, complexFormat* E, complexFormat* fulltrace, Real* delay), (gate, E, fulltrace,delay), {
  cudaIdx();
  int tidx = y-delay[x];
  if(tidx >= cuda_column || tidx < 0) {
    fulltrace[index].x = 0;
    fulltrace[index].y = 0;
  }else
    fulltrace[index] = hipCmulf(gate[tidx],E[y]);
})


cuFunc(genEComplex,(complexFormat* E),(E),{
  cuda1Idx();
  int rindex = cuda_row-index-1;
  int bias = index-cuda_row/2;
  Real sigma = 20;
  Real midwl = 100;
  Real chirp = 3e-4;
  Real CEP = M_PI;
  Real phase = 2*M_PI/midwl*index + CEP + 2*M_PI*chirp*(index-400)*(index-400);
  Real envolope = (exp(-sq(bias-50)/(2*sq(sigma)))+exp(-sq(bias+50)/(2*sq(sigma))));
  E[rindex].x = envolope*cos(-phase);
  E[rindex].y = envolope*sin(-phase);
})

cuFunc(genE,(Real* E),(E),{
  cuda1Idx();
  int bias = index-cuda_row/2;
  Real sigma = 20;
  Real chirp = 1e-5;
  Real midfreq = 22;
  Real CEP = M_PI;
  E[index] = (exp(-sq(bias-100)/(2*sq(sigma)))+exp(-sq(bias+100)/(2*sq(sigma))))*cos(2*M_PI/midfreq*index + 2*M_PI/chirp*index*index + CEP);
})

cuFunc(setDelay,(Real* delay),(delay),{
  cuda1Idx();
  delay[index] = Real(index)-cuda_row/2;
})

cuFunc(convertFOy, (complexFormat* data), (data), {
  cuda1Idx();
  int y = index%cuda_column;
  if(y >= cuda_column/2) return;
  complexFormat tmp = data[index];
  data[index] = data[index+cuda_column/2];
  data[index+cuda_column/2] = tmp;
})
cuFunc(updateGE, (complexFormat* E, complexFormat* gate, complexFormat* trace, Real* delays, int i, Real alpha), (E,gate,trace,delays,i, alpha), {
  cuda1Idx();
  int tidx = index - delays[i];
  if(tidx >= cuda_row || tidx < 0) return;
  if(index < 100 || index > cuda_row-100) {
    E[index].x = 0;
    E[index].y = 0;
    return;
  }
  complexFormat tmp1 = E[index];
  tmp1.y = -tmp1.y;
  Real alpha1 = 0.1*alpha/(sqSum(tmp1.x,tmp1.y)+1e-2);
  tmp1 = hipCmulf(tmp1,trace[i*cuda_row+index]);

  gate[tidx].x += alpha1*tmp1.x;
  gate[tidx].y += alpha1*tmp1.y;

  complexFormat tmp = gate[tidx];
  tmp.y = -tmp.y;
  alpha /= (sqSum(tmp.x,tmp.y)+3e-3);
  tmp = hipCmulf(tmp,trace[i*cuda_row+index]);

  E[index].x += alpha*tmp.x;
  E[index].y += alpha*tmp.y;

})
cuFunc(updateE, (complexFormat* E, complexFormat* gate, complexFormat* trace, Real* delays, int i, Real alpha), (E,gate,trace,delays,i, alpha), {
  cuda1Idx();
  int tidx = index - delays[i];
  if(tidx >= cuda_row || tidx < 0) return;
  if(index < 100 || index > cuda_row-100) {
    E[index].x = 0;
    E[index].y = 0;
    return;
  }
  complexFormat tmp = gate[tidx];
  tmp.y = -tmp.y;
  //alpha /= (sqSum(tmp.x,tmp.y)+1e-2);
  tmp = hipCmulf(tmp,trace[i*cuda_row+index]);
  E[index].x += alpha*tmp.x;
  E[index].y += alpha*tmp.y;
})

cuFunc(initGate, (complexFormat* gate), (gate), {
  cuda1Idx();
  int window = 100;
  if(index > cuda_row/2-window && index < cuda_row/2+window){
    gate[index].x = 1;
  }else gate[index].x = 0;
  gate[index].y = 0;
})

cuFunc(removeNeg, (complexFormat* data), (data), {
  cuda1Idx();
  if(index >= cuda_row/4){
    data[index].x = 0;
    data[index].y = 0;
  }
})

cuFunc(shiftmid, (complexFormat* Eprime, complexFormat* E, int tx),(Eprime, E, tx),{
  cuda1Idx();
  int tidx = index+tx;
  if(tidx < 0 || tidx >= cuda_row) {
    E[index].x = 0;
    E[index].y = 0;
  }
  else E[index] = Eprime[tidx];
})

cuFunc(average, (complexFormat* Eprime, complexFormat* E, Real gamma),(Eprime, E, gamma),{
  cuda1Idx();
  Eprime[index].x = E[index].x = (Eprime[index].x+E[index].x)/2;
  Eprime[index].y = E[index].y = (Eprime[index].y+E[index].y)/2;
})

cuFunc(stepMove, (complexFormat* Eprime, complexFormat* E, Real gamma),(Eprime, E, gamma),{
  cuda1Idx();
  Real diff = Eprime[index].x - E[index].x;
  if(diff > gamma) diff = gamma;
  else if(diff < -gamma) diff = -gamma;
  Eprime[index].x -= diff;
  diff = Eprime[index].y - E[index].y;
  if(diff > gamma) diff = gamma;
  else if(diff < -gamma) diff = -gamma;
  Eprime[index].y -= diff;
})


void solveE(complexFormat* E, Real* traceIntensity, complexFormat* trace, Real* delays){
  int ndelay = cuda_imgsz.x;
  int nspect = cuda_imgsz.y;
  complexFormat* gate = (complexFormat*) memMngr.borrowSame(E);
  complexFormat* Eprime = (complexFormat*) memMngr.borrowSame(E);
  complexFormat* traceprime = (complexFormat*) memMngr.borrowSame(trace);
  resize_cuda_image(nspect,1);
  initGate(gate);
  hipfftHandle singleplan;
  hipfftPlan1d(&singleplan, nspect, FFTformat, 1);
  //hipMemcpy(E, gate, nspect*sizeof(complexFormat), hipMemcpyDeviceToDevice);
  int niter = 100;
  //applyNorm(traceIntensity, nspect);
  getMod2((Real*)Eprime, gate);
  Real maxgate = findMax((Real*)Eprime, nspect);
  myCuDMalloc(hiprandStateMRG32k3a, devstates, nspect*ndelay);
  Real resprev = 0;
  Real step = 0.8;
  FILE* file = fopen("residual.txt", "w");
  for(int i = 0; i < niter; i++){
    resize_cuda_image(ndelay,nspect);
    dgencTrace(gate, E, trace, delays);
    myFFT(trace, traceprime);
    applyModAbs(traceprime, traceIntensity, devstates);
    applyNorm(traceprime, 1./sqrt(nspect));
    myIFFT(traceprime, traceprime);
    add(traceprime, trace, -1);
    resize_cuda_image(nspect,1);
    for(int j = 0; j < ndelay; j++){
      updateE(E, gate, traceprime, delays, j, step/maxgate);
    }
    resize_cuda_image(ndelay,nspect);
    getMod2((Real*)trace,traceprime);
    Real residual = findSum((Real*)trace, nspect*ndelay);
    if(residual < 1e-4) break;
    if(i>4){
      if(resprev < residual) step*=0.5;
      Real ratio = residual / fabs(resprev - residual);
      if(ratio > 10 && resprev > residual) step *= 2.;
      resprev = residual;
      if(step > 0.5) step = 0.5;
      //fprintf(file, "residual = %f, ratio = %f, step = %f\n", residual, ratio, step);
      fprintf(file, "%d %f\n", i, residual);
    }
    if(residual!=residual) exit(0);
    resize_cuda_image(nspect,1);
    myCufftExec(singleplan, E, Eprime, HIPFFT_FORWARD);
    removeNeg(Eprime);
    if(i %20 == 0) {
      myCufftExec(singleplan, Eprime, Eprime, HIPFFT_BACKWARD);
      Real mid = 0;
      mid = findMiddle(Eprime,nspect).x;
      shiftmid(Eprime, E, mid*nspect);
    }else{
      myCufftExec(singleplan, Eprime, E, HIPFFT_BACKWARD);
    }
    applyNorm(E, 1./nspect);
    hipMemcpy(gate, E, nspect*sizeof(complexFormat), hipMemcpyDeviceToDevice);
    getMod2((Real*)Eprime, gate);
    maxgate = findMax((Real*)Eprime, nspect)+1e-2;
  }
  fclose(file);
  myFFT(trace, trace);
  resize_cuda_image(ndelay,nspect);
  applyNorm(trace, 1./sqrt(nspect));
  convertFOy(trace);
}

void genTrace(complexFormat* E, complexFormat* fulltrace, Real* delays){
  dgencTrace(E, E, fulltrace,delays);
  myFFT(fulltrace, fulltrace);
  applyNorm(fulltrace, 1./sqrt(cuda_imgsz.y));
}

void genTrace(Real* E, complexFormat* fulltrace, Real* delays){
  dgenTrace(E, E, fulltrace,delays);
  myFFT(fulltrace, fulltrace);
  applyNorm(fulltrace, 1./sqrt(cuda_imgsz.y));
}

int main(int argc, char** argv )
{
  init_cuda_image();  //always needed
  int ndelay = 250;
  myDMalloc(Real, delays, ndelay);
  int nspect = 1000;
  int nfulldelay = 2000;
  myCuDMalloc(Real, d_fulldelays, nfulldelay);
  myCuDMalloc(Real, d_delays, ndelay);
  myCuDMalloc(complexFormat, d_cE, nspect);
  myCuDMalloc(complexFormat, d_fulltraces, nspect*nfulldelay);
  myCuDMalloc(Real, d_traceIntensity, nspect*ndelay);
  myCuDMalloc(complexFormat, d_traces, nspect*ndelay);
  //Generate E, simulate trace
  resize_cuda_image(nspect,1);
  //myCuDMalloc(Real, d_E, nspect);
  //genE(d_E);
  genEComplex(d_cE);
  myDMalloc(complexFormat, ccE, nspect);
  hipMemcpy(ccE, d_cE, sizeof(complexFormat)*nspect, hipMemcpyDeviceToHost);
  std::ofstream file("input.txt", std::ios::out);
  for(int i = 0; i < cuda_imgsz.x; i++){
    file<<i << " " << ccE[i].x << " " << ccE[i].y << " " << hypot(ccE[i].x,ccE[i].y) << " " << arg(complex<float>(ccE[i].x, ccE[i].y))<<std::endl;
  }
  file.close();
  resize_cuda_image(nfulldelay,1);
  setDelay(d_fulldelays);
  init_fft(nspect,1,nfulldelay);
  resize_cuda_image(nfulldelay,nspect);
  plt.init(nfulldelay,nspect);
  //genTrace(d_E, d_fulltraces, d_fulldelays);
  genTrace(d_cE, d_fulltraces, d_fulldelays);
  convertFOy(d_fulltraces);
  plt.plotComplex(d_fulltraces, MOD2, 0, 1, "trace_truth", 1, 0, 1);
  //select delay, reconstruct E;
  //srand(time(NULL));
  srand(8);
  for(int i = 0; i < ndelay; i++){
    delays[i] = i*nfulldelay/ndelay - nfulldelay/2 + 2.*rand()/RAND_MAX;
  }
  hipMemcpy(d_delays, delays, ndelay*sizeof(Real), hipMemcpyHostToDevice);
  resize_cuda_image(ndelay,nspect);
  plt.init(ndelay,nspect);
  init_fft(nspect,1,ndelay);
  genTrace(d_cE, d_traces, d_delays);
  getMod2(d_traceIntensity, d_traces);
  convertFOy(d_traces);
  plt.plotComplex(d_traces, MOD2, 0, 1, "trace_sampled", 1, 0, 1);
  hipMemset(d_cE, 0, nspect*sizeof(complexFormat));
  hipMemset(d_traces, 0, nspect*ndelay*sizeof(complexFormat));
  solveE(d_cE, d_traceIntensity, d_traces, d_delays);
  plt.plotComplex(d_traces, MOD2, 0, 1, "trace_recon", 1, 0, 1);
  hipMemcpy(ccE, d_cE, sizeof(complexFormat)*nspect, hipMemcpyDeviceToHost);
  std::ofstream file1("output.txt", std::ios::out);
  for(int i = 0; i < nspect; i++){
    file1<<i << " " << ccE[i].x << " " << ccE[i].y << " " << hypot(ccE[i].x,ccE[i].y) << " " << arg(complex<float>(ccE[i].x, ccE[i].y))<<std::endl;
  }
  file1.close();
  init_fft(nspect,1,nfulldelay);
  resize_cuda_image(nfulldelay,nspect);
  plt.init(nfulldelay,nspect);
  genTrace(d_cE, d_fulltraces, d_fulldelays);
  convertFOy(d_fulltraces);
  plt.plotComplex(d_fulltraces, MOD2, 0, 1, "trace_recon_full", 1, 0, 1);
}

