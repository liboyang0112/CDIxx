#include "hip/hip_runtime.h"
#include "cudaConfig.h" //cuda related
#include "cudaDefs.h"
#include "cuPlotter.h" //plt
#include <complex>
#include <hiprand/hiprand_kernel.h>
#include "cub_wrap.h"
#include <fstream>
#include <vector>
#include <random>
#include <algorithm>
#define ALPHA 0.5
#define BETA 1
#define DELTA 1e-3
#define GAMMA 0.5
using namespace std;

Real* simTrace(int ndelay, int nspect){
  myDMalloc(Real, traces, nspect*ndelay);
  return traces;
}

void saveWave(const char* fname, complexFormat* ccE, int n){
  std::ofstream file1(fname, std::ios::out);
  for(int i = 0; i < n; i++){
    auto dat = ((complex<float>*)ccE)[i];
    file1<<i << " " << dat.real() << " " << dat.imag() << " " << abs(dat) << " " << arg(dat)<<std::endl;
  }
  file1.close();
}

cuFuncc(dgenTrace, (Real* gate, Real* E, complexFormat* fulltrace, Real* delay),(Real* gate, Real* E, hipComplex* fulltrace, Real* delay), (gate, E, (hipComplex*)fulltrace,delay), {
  cudaIdx();
  int tidx = y-delay[x];
  if(tidx >= cuda_column || tidx < 0) {
    fulltrace[index].x = 0;
  }else
    fulltrace[index].x = gate[tidx] * E[y];
  fulltrace[index].y = 0;
})

cuFuncc(dgencTraceSingle, (complexFormat* gate, complexFormat* E, complexFormat* trace, Real* delay, int i),(hipComplex* gate, hipComplex* E, hipComplex* trace, Real* delay, int i), ((hipComplex*)gate, (hipComplex*)E, (hipComplex*)trace,delay, i), {
  cuda1Idx();
  int tidx = index-delay[i];
  if(tidx >= cuda_row || tidx < 0) {
    trace[index].x = 0;
    trace[index].y = 0;
  }else
    trace[index] = hipCmulf(gate[tidx],E[index]);
})

cuFuncc(dgencTrace, (complexFormat* gate, complexFormat* E, complexFormat* fulltrace, Real* delay),(hipComplex* gate, hipComplex* E, hipComplex* fulltrace, Real* delay), ((hipComplex*)gate, (hipComplex*)E, (hipComplex*)fulltrace,delay), {
  cudaIdx();
  int tidx = y-delay[x];
  if(tidx >= cuda_column || tidx < 0) {
    fulltrace[index].x = 0;
    fulltrace[index].y = 0;
  }else
    fulltrace[index] = hipCmulf(gate[tidx],E[y]);
})


cuFuncc(genEComplex,(complexFormat* E),(hipComplex* E),((hipComplex*)E),{
  cuda1Idx();
  int rindex = cuda_row-index-1;
  int bias = index-cuda_row/2;
  Real sigma = 50;
  Real midf = 0;
  Real chirp = 2e-4;
  Real CEP = M_PI;
  Real phase = 2*M_PI*midf*index + CEP + 2*M_PI*chirp*(index-500)*(index-500);
  Real envolope = (exp(-sq(bias-50)/(2*sq(sigma)))+exp(-sq(bias+50)/(2*sq(sigma))));
  E[rindex].x = envolope*cos(phase);
  E[rindex].y = envolope*sin(phase);
})

cuFunc(genE,(Real* E),(E),{
  cuda1Idx();
  int bias = index-cuda_row/2;
  Real sigma = 20;
  Real chirp = 1e-5;
  Real midfreq = 22;
  Real CEP = M_PI;
  E[index] = (exp(-sq(bias-100)/(2*sq(sigma)))+exp(-sq(bias+100)/(2*sq(sigma))))*cos(2*M_PI/midfreq*index + 2*M_PI/chirp*index*index + CEP);
})

cuFunc(setDelay,(Real* delay),(delay),{
  cuda1Idx();
  delay[index] = Real(index)-cuda_row/2;
})

cuFuncc(convertFOy, (complexFormat* data),(hipComplex* data), ((hipComplex*)data), {
  cuda1Idx();
  int y = index%cuda_column;
  if(y >= cuda_column/2) return;
  hipComplex tmp = data[index];
  data[index] = data[index+cuda_column/2];
  data[index+cuda_column/2] = tmp;
})
cuFuncc(updateGE, (complexFormat* E, complexFormat* gate, complexFormat* trace, Real* delays, int i, Real alpha),(hipComplex* E, hipComplex* gate, hipComplex* trace, Real* delays, int i, Real alpha), ((hipComplex*)E,(hipComplex*)gate,(hipComplex*)trace,delays,i, alpha), {
  cuda1Idx();
  int tidx = index - delays[i];
  if(tidx >= cuda_row || tidx < 0) return;
  if(index < 100 || index > cuda_row-100) {
    E[index].x = 0;
    E[index].y = 0;
    return;
  }
  hipComplex tmp1 = E[index];
  tmp1.y = -tmp1.y;
  Real alpha1 = 0.1*alpha/(sqSum(tmp1.x,tmp1.y)+1e-2);
  tmp1 = hipCmulf(tmp1,trace[i*cuda_row+index]);

  gate[tidx].x += alpha1*tmp1.x;
  gate[tidx].y += alpha1*tmp1.y;

  hipComplex tmp = gate[tidx];
  tmp.y = -tmp.y;
  alpha /= (sqSum(tmp.x,tmp.y)+3e-3);
  tmp = hipCmulf(tmp,trace[i*cuda_row+index]);

  E[index].x += alpha*tmp.x;
  E[index].y += alpha*tmp.y;

})
cuFuncc(updateE, (complexFormat* E, complexFormat* gate, complexFormat* trace, Real* delays, int i, Real alpha),(hipComplex* E, hipComplex* gate, hipComplex* trace, Real* delays, int i, Real alpha), ((hipComplex*)E,(hipComplex*)gate,(hipComplex*)trace,delays,i, alpha), {
  cuda1Idx();
  int tidx = index - delays[i];
  if(tidx >= cuda_row || tidx < 0) return;
  if(index < 100 || index > cuda_row-100) {
    E[index].x = 0;
    E[index].y = 0;
    return;
  }
  hipComplex tmp = gate[tidx];
  tmp.y = -tmp.y;
  alpha /= sqrt(sqSum(tmp.x,tmp.y)+1e-1);
  tmp = hipCmulf(tmp,trace[index]);
  E[index].x += alpha*tmp.x;
  E[index].y += alpha*tmp.y;
})

cuFuncc(initGate, (complexFormat* gate),(hipComplex* gate), ((hipComplex*)gate), {
  cuda1Idx();
  int window = 100;
  if(index > cuda_row/2-window && index < cuda_row/2+window){
    gate[index].x = 1;
  }else gate[index].x = 0;
  gate[index].y = 0;
})

cuFuncc(removeHighFreq, (complexFormat* data),(hipComplex* data), ((hipComplex*)data), {
  cuda1Idx();
  if(index >= cuda_row/4 && index < cuda_row/4*3){
    data[index].x = 0;
    data[index].y = 0;
  }
})

cuFuncc(shiftmid, (complexFormat* Eprime, complexFormat* E, int tx),(hipComplex* Eprime, hipComplex* E, int tx),((hipComplex*)Eprime, (hipComplex*)E, tx),{
  cuda1Idx();
  int tidx = index+tx;
  if(tidx < 0 || tidx >= cuda_row) {
    E[index].x = 0;
    E[index].y = 0;
  }
  else E[index] = Eprime[tidx];
})

cuFuncc(average, (complexFormat* Eprime, complexFormat* E, Real gamma),(hipComplex* Eprime, hipComplex* E, Real gamma),((hipComplex*)Eprime, (hipComplex*)E, gamma),{
  cuda1Idx();
  Eprime[index].x = E[index].x = (Eprime[index].x+E[index].x)/2;
  Eprime[index].y = E[index].y = (Eprime[index].y+E[index].y)/2;
})

cuFuncc(stepMove, (complexFormat* Eprime, complexFormat* E, Real gamma),(hipComplex* Eprime, hipComplex* E, Real gamma),((hipComplex*)Eprime, (hipComplex*)E, gamma),{
  cuda1Idx();
  Real diff = Eprime[index].x - E[index].x;
  if(diff > gamma) diff = gamma;
  else if(diff < -gamma) diff = -gamma;
  Eprime[index].x -= diff;
  diff = Eprime[index].y - E[index].y;
  if(diff > gamma) diff = gamma;
  else if(diff < -gamma) diff = -gamma;
  Eprime[index].y -= diff;
})


void solveE(complexFormat* E, Real* traceIntensity, Real* spectrum, complexFormat* trace, Real* delays, int singleplan){
  int ndelay = cuda_imgsz.x;
  int nspect = cuda_imgsz.y;
  myDMalloc(complexFormat, ccE, nspect);
  complexFormat* gate = (complexFormat*) memMngr.borrowSame(E);
  complexFormat* Eprime = (complexFormat*) memMngr.borrowSame(E);
  complexFormat* traceprime = (complexFormat*) memMngr.borrowSame(E);
  resize_cuda_image(nspect,1);
  initGate(gate);
  myMemcpyD2D(E, gate, nspect*sizeof(complexFormat));
  int niter = 100;
  void* devstates = newRand(nspect);
  Real step = 0.9;
  resize_cuda_image(nspect,1);
  initRand(devstates, time(NULL));
  vector<int> sf(ndelay);
  for(int i = 0; i < ndelay; i++) sf[i] = i;
  for(int i = 0; i < niter; i++){
    shuffle(sf.begin(),sf.end(), std::default_random_engine(time(NULL)));
    for(int j = 0; j < ndelay; j++){
      int sfd = sf[j];
      dgencTraceSingle(gate, E, trace, delays, sfd);
      myFFTM(singleplan, trace, traceprime);
      applyModAbs(traceprime, traceIntensity+sfd*nspect, devstates);
      applyNorm(traceprime, 1./sqrt(nspect));
      myIFFTM(singleplan, traceprime, traceprime);
      add(traceprime, trace, -1);
      updateE(E, gate, traceprime, delays, sfd, step);
      myMemcpyD2D(gate, E, nspect*sizeof(complexFormat));
    }
    if(spectrum){
      myFFTM(singleplan, E, Eprime);
      applyModAbs(Eprime, spectrum, devstates);
      applyNorm(Eprime, 1./sqrt(nspect));
      myIFFTM(singleplan, Eprime, E);
    }
    myFFTM(singleplan, E, Eprime);
    removeHighFreq(Eprime);
    if(i %20 == 0) {
      myIFFTM(singleplan, Eprime, Eprime);
      Real mid = complex<float>(findMiddle(Eprime,nspect)).real();
      shiftmid(Eprime, E, mid*nspect);
    }else{
      myIFFTM(singleplan, Eprime, E);
    }
    applyNorm(E, 1./nspect);
    myMemcpyD2D(gate, E, nspect*sizeof(complexFormat));
  }
  resize_cuda_image(ndelay,nspect);
  dgencTrace(gate, E, trace, delays);
  myFFT(trace, trace);
  applyNorm(trace, 1./sqrt(nspect));
  convertFOy(trace);
}

void genTrace(complexFormat* E, complexFormat* fulltrace, Real* delays){
  dgencTrace(E, E, fulltrace,delays);
  myFFT(fulltrace, fulltrace);
  applyNorm(fulltrace, 1./sqrt(cuda_imgsz.y));
}

void genTrace(Real* E, complexFormat* fulltrace, Real* delays){
  dgenTrace(E, E, fulltrace,delays);
  myFFT(fulltrace, fulltrace);
  applyNorm(fulltrace, 1./sqrt(cuda_imgsz.y));
}

int main(int argc, char** argv )
{
  init_cuda_image();  //always needed
  int ndelay = 200;
  myDMalloc(Real, delays, ndelay);
  int nspect = 1000;
  int nfulldelay = 1000;
  myCuDMalloc(Real, d_fulldelays, nfulldelay);
  myCuDMalloc(Real, d_delays, ndelay);
  myCuDMalloc(complexFormat, d_cE, nspect);
  myCuDMalloc(complexFormat, d_spect, nspect);
  myCuDMalloc(Real, d_spectrum, nspect);
  myCuDMalloc(complexFormat, d_fulltraces, nspect*nfulldelay);
  myCuDMalloc(Real, d_traceIntensity, nspect*ndelay);
  myCuDMalloc(complexFormat, d_traces, nspect*ndelay);
  //Generate E, simulate trace
  resize_cuda_image(nspect,1);
  //myCuDMalloc(Real, d_E, nspect);
  //genE(d_E);
  genEComplex(d_cE);
  myDMalloc(complexFormat, ccE, nspect);
  myMemcpyD2H(ccE, d_cE, sizeof(complexFormat)*nspect);
  saveWave("input.txt", ccE, nspect);
  int singleplan;
  createPlan1d(&singleplan, nspect);
  myFFTM(singleplan, d_cE, d_spect);
  applyNorm(d_spect, 1./sqrt(nspect));
  getMod2(d_spectrum, d_spect);
  cudaConvertFO(d_spect);
  convertFOPhase(d_spect);
  myMemcpyD2H(ccE, d_spect, sizeof(complexFormat)*nspect);
  saveWave("inputSpect.txt", ccE, nspect);
  resize_cuda_image(nfulldelay,1);
  setDelay(d_fulldelays);
  init_fft(nspect,1,nfulldelay);
  resize_cuda_image(nfulldelay,nspect);
  plt.init(nfulldelay,nspect);
  //genTrace(d_E, d_fulltraces, d_fulldelays);
  genTrace(d_cE, d_fulltraces, d_fulldelays);
  convertFOy(d_fulltraces);
  plt.plotComplex(d_fulltraces, MOD2, 0, 1, "trace_truth", 1, 0, 1);
  //select delay, reconstruct E;
  srand(time(NULL));
  for(int i = 0; i < ndelay; i++){
    delays[i] = i*nfulldelay/ndelay - nfulldelay/2;
  }
  myMemcpyH2D(d_delays, delays, ndelay*sizeof(Real));
  resize_cuda_image(ndelay,nspect);
  plt.init(ndelay,nspect);
  init_fft(nspect,1,ndelay);
  genTrace(d_cE, d_traces, d_delays);
  getMod2(d_traceIntensity, d_traces);
  convertFOy(d_traces);
  plt.plotComplex(d_traces, MOD2, 0, 1, "trace_sampled", 1, 0, 1);
  clearCuMem(d_cE,  nspect*sizeof(complexFormat));
  clearCuMem(d_traces,  nspect*ndelay*sizeof(complexFormat));
  solveE(d_cE, d_traceIntensity, 0, d_traces, d_delays, singleplan);
  //solveE(d_cE, d_traceIntensity, d_spectrum, d_traces, d_delays, singleplan);
  plt.plotComplex(d_traces, MOD2, 0, 1, "trace_recon", 1, 0, 1);
  myMemcpyD2H(ccE, d_cE, sizeof(complexFormat)*nspect);
  saveWave("output.txt", ccE, nspect);
  myFFTM(singleplan, d_cE, d_spect);
  resize_cuda_image(nspect,1);
  applyNorm(d_spect, 1./sqrt(nspect));
  cudaConvertFO(d_spect);
  convertFOPhase(d_spect);
  myMemcpyD2H(ccE, d_spect, sizeof(complexFormat)*nspect);
  saveWave("outputSpect.txt", ccE, nspect);
  init_fft(nspect,1,nfulldelay);
  resize_cuda_image(nfulldelay,nspect);
  plt.init(nfulldelay,nspect);
  genTrace(d_cE, d_fulltraces, d_fulldelays);
  convertFOy(d_fulltraces);
  plt.plotComplex(d_fulltraces, MOD2, 0, 1, "trace_recon_full", 1, 0, 1);
}

