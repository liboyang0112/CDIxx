#include "hip/hip_runtime.h"
#include "cudaConfig.h" //cuda related
#include "cudaDefs.h"
#include "cuPlotter.h" //plt
#include <complex>
#include <hiprand/hiprand_kernel.h>
#include "cub_wrap.h"
#include <fstream>
#include <vector>
#include <random>
#include <algorithm>
#define ALPHA 0.5
#define BETA 1
#define DELTA 1e-3
#define GAMMA 0.5
using namespace std;

void saveWave(const char* fname, complexFormat* ccE, int n){
  std::ofstream file1(fname, std::ios::out);
  for(int i = 0; i < n; i++){
    auto dat = ((complex<float>*)ccE)[i];
    file1<<i << " " << dat.real() << " " << dat.imag() << " " << abs(dat) << " " << arg(dat)<<std::endl;
  }
  file1.close();
}
cuFuncc(dgencTraceSingle, (complexFormat* gate, complexFormat* E, complexFormat* trace, Real delay),(hipComplex* gate, hipComplex* E, hipComplex* trace, Real delay), ((hipComplex*)gate, (hipComplex*)E, (hipComplex*)trace,delay), {
  cuda1Idx();
  int tidx = index-delay;
  if(tidx >= cuda_row || tidx < 0) {
    trace[index].x = 0;
    trace[index].y = 0;
  }else
    trace[index] = hipCmulf(gate[tidx],E[index]);
})

cuFuncc(dgencTrace, (complexFormat* gate, complexFormat* E, complexFormat* fulltrace, Real* delay),(hipComplex* gate, hipComplex* E, hipComplex* fulltrace, Real* delay), ((hipComplex*)gate, (hipComplex*)E, (hipComplex*)fulltrace,delay), {
  cudaIdx();
  int tidx = y-delay[x];
  if(tidx >= cuda_column || tidx < 0) {
    fulltrace[index].x = 0;
    fulltrace[index].y = 0;
  }else
    fulltrace[index] = hipCmulf(gate[tidx],E[y]);
})


cuFuncc(genEComplex,(complexFormat* E),(hipComplex* E),((hipComplex*)E),{
  cuda1Idx();
  int bias = index-cuda_row/2;
  Real sigma = 5;
  Real midf = 0;
  Real chirp = 2e-3;
  Real CEP = M_PI;
  Real phase = 2*M_PI*midf*index + CEP + 2*M_PI*chirp*bias*bias;
  Real envolope = (exp(-sq(bias-10)/(2*sq(sigma)))+0.5*exp(-sq(bias+10)/(2*sq(sigma))));
  E[index].x = envolope*cos(phase);
  E[index].y = envolope*sin(phase);
})

cuFunc(setDelay,(Real* delay),(delay),{
  cuda1Idx();
  delay[index] = index-cuda_row/2;
})

cuFuncc(convertFOy, (complexFormat* data),(hipComplex* data), ((hipComplex*)data), {
  cuda1Idx();
  int y = index%cuda_column;
  if(y >= cuda_column/2) return;
  hipComplex tmp = data[index];
  data[index] = data[index+cuda_column/2];
  data[index+cuda_column/2] = tmp;
})
cuFuncc(applySoftThreshold, (complexFormat* data, Real thres),(hipComplex* data, Real thres), ((hipComplex*)data, thres), {
  cuda1Idx();
  Real mod = hipCabsf(data[index]);
  if(mod > thres){
    mod = (1-thres/mod)/cuda_row;
    data[index].x *= mod;
    data[index].y *= mod;
  }else data[index].x = data[index].y = 0;
})
cuFuncc(updateGE, (complexFormat* E, complexFormat* gate, complexFormat* trace, Real delay, Real alpha),(hipComplex* E, hipComplex* gate, hipComplex* trace, Real delay, Real alpha), ((hipComplex*)E,(hipComplex*)gate,(hipComplex*)trace,delay, alpha), {
  cuda1Idx();
  int tidx = index - delay;
  if(tidx >= cuda_row || tidx < 0) return;
  if(index < cuda_row/4 || index > 3*cuda_row/4) {
    E[index].x = E[index].y = 0;
    return;
  }
  hipComplex tmp = E[index];
  tmp.y = -tmp.y;
  tmp = hipCmulf(tmp,trace[index]);
  gate[tidx].x += alpha*tmp.x;
  gate[tidx].y += alpha*tmp.y;

  tmp = gate[tidx];
  tmp.y = -tmp.y;
  tmp = hipCmulf(tmp,trace[index]);

  E[index].x += alpha*tmp.x;
  E[index].y += alpha*tmp.y;

})
cuFuncc(updateE, (complexFormat* E, complexFormat* gate, complexFormat* trace, Real delay, Real alpha),(hipComplex* E, hipComplex* gate, hipComplex* trace, Real delay, Real alpha), ((hipComplex*)E,(hipComplex*)gate,(hipComplex*)trace,delay, alpha), {
  cuda1Idx();
  int tidx = index - delay;
  if(tidx >= cuda_row || tidx < 0) return;
  hipComplex tmp = gate[tidx];
  tmp.y = -tmp.y;
  tmp = hipCmulf(tmp,trace[index]);
  E[index].x += alpha*tmp.x;
  E[index].y += alpha*tmp.y;
})

cuFuncc(initE, (complexFormat* gate),(hipComplex* gate), ((hipComplex*)gate), {
  cuda1Idx();
  int window = 32;
  if(index > cuda_row/2-window && index < cuda_row/2+window){
    gate[index].x = 1;
  }else gate[index].x = 0;
  gate[index].y = 0;
})

cuFuncc(removeHighFreq, (complexFormat* data),(hipComplex* data), ((hipComplex*)data), {
  cuda1Idx();
  if(index >= cuda_row/4 && index < cuda_row/4*3){
    data[index].x = 0;
    data[index].y = 0;
  }
})

cuFuncc(shiftmid, (complexFormat* Eprime, complexFormat* E, int tx),(hipComplex* Eprime, hipComplex* E, int tx),((hipComplex*)Eprime, (hipComplex*)E, tx),{
  cuda1Idx();
  int tidx = index+tx;
  if(tidx < 0 || tidx >= cuda_row) {
    E[index].x = 0;
    E[index].y = 0;
  }
  else E[index] = Eprime[tidx];
})

cuFuncc(average, (complexFormat* Eprime, complexFormat* E, Real gamma),(hipComplex* Eprime, hipComplex* E, Real gamma),((hipComplex*)Eprime, (hipComplex*)E, gamma),{
  cuda1Idx();
  Eprime[index].x = E[index].x = (Eprime[index].x+E[index].x)/2;
  Eprime[index].y = E[index].y = (Eprime[index].y+E[index].y)/2;
})
cuFuncc(applyModAbsxrange,(complexFormat* source, Real* target, void* state, int xrange, Real gamma),(hipComplex* source, Real* target, void* state, int xrange, Real gamma),((hipComplex*)source, target, state, xrange, gamma),{
    cuda1Idx();
    Real mod = hypot(source[index].x, source[index].y);
    if(index < xrange+cuda_row/2 && index >= cuda_row/2 - xrange){
      Real thres = gamma*sqrtf(cuda_row);
      if(mod > thres){
        mod = (1-thres/mod)/cuda_row;
        source[index].x *= mod;
        source[index].y *= mod;
      }else source[index].x = source[index].y = 0;
      return;
    }
    Real rat = target[index];
    if(rat > 0) rat = sqrt(rat);
    else rat = 0;
    if(mod==0) {
    Real randphase = state?hiprand_uniform((hiprandStateMRG32k3a*)state + index)*2*M_PI:0;
    source[index].x = rat*cos(randphase);
    source[index].y = rat*sin(randphase);
    return;
    }
    rat /= mod;
    source[index].x *= rat;
    source[index].y *= rat;
    })

void solveE(complexFormat* E, Real* traceIntensity, Real* spectrum, complexFormat* trace, Real* delays, int nfulldelay, int singleplan, int nspectm){
  int ndelay = cuda_imgsz.x;
  int nspect = cuda_imgsz.y;
  myDMalloc(complexFormat, ccE, nspect);
  complexFormat* gate = trace+nspect;
  complexFormat* Eprime = gate+nspect;
  complexFormat* traceprime = Eprime+nspect;
  resize_cuda_image(nspect,1);
  initE(E);
  myMemcpyD2D(gate, E, nspect*sizeof(complexFormat));
  int niter = 3000;
  double step = spectrum?10:3.;
  vector<int> sf(ndelay);
  for(int i = 0; i < ndelay; i++) sf[i] = i;
  std::mt19937 mtrnd( std::random_device {} () );
  for(int i = 0; i < niter; i++){
    shuffle(sf.begin(),sf.end(), mtrnd);
    for(int j = 0; j < ndelay; j++){
      double randv = double(rand())/RAND_MAX;
      int thisdelay = delays[sf[j]];
      getMod2((Real*)Eprime, E);
      Real maxv = findMax((Real*)Eprime, nspect);
      dgencTraceSingle(gate, E, trace, thisdelay);
      myFFTM(singleplan, trace, traceprime);
      applyModAbsxrange(traceprime, traceIntensity+sf[j]*nspect, 0, nspectm, 1e-3);
      myIFFTM(singleplan, traceprime, traceprime);
      add(traceprime, trace, -1);
      updateGE(E, gate, traceprime, thisdelay, (step*randv)/maxv);
      average(E,gate,0.5);
      if(spectrum){
        myFFTM(singleplan, E, Eprime);
        applyModAbs(Eprime, spectrum);
        applyNorm(Eprime, 1./sqrt(nspect));
        myIFFTM(singleplan, Eprime, E);
        myMemcpyD2D(gate, E, nspect*sizeof(complexFormat));
      }
    }
    myFFTM(singleplan, E, Eprime);
    if(spectrum){
      applyModAbs(Eprime, spectrum);
      applyNorm(Eprime, 1./sqrt(nspect));
    }else{
      removeHighFreq(Eprime);
      applyNorm(Eprime, 1./nspect);
    }
    if(i %10 == 0) {
      myIFFTM(singleplan, Eprime, Eprime);
      Real mid = complex<float>(findMiddle(Eprime,nspect)).real();
      shiftmid(Eprime, E, mid*nspect);
    }else{
      myIFFTM(singleplan, Eprime, E);
    }
    myMemcpyD2D(gate, E, nspect*sizeof(complexFormat));
  }
}

void genTrace(complexFormat* E, complexFormat* fulltrace, Real* delays, int nspectm = 0){
  dgencTrace(E, E, fulltrace,delays);
  myFFT(fulltrace, fulltrace);
  applyNorm(fulltrace, 1./sqrt(cuda_imgsz.y));
  convertFOy(fulltrace);
  if(nspectm) zeroEdgey(fulltrace, nspectm);
  convertFOy(fulltrace);
}

int main(int argc, char** argv )
{
  init_cuda_image();  //always needed
  int ndelay = 5;
  int nspect = 128;
  int nspectm=58;
  int nfulldelay = 128;
  //declare and allocate variables
  myDMalloc(Real, delays, ndelay);
  myCuDMalloc(Real, d_fulldelays, nfulldelay);
  myCuDMalloc(Real, d_delays, ndelay);
  myCuDMalloc(complexFormat, d_cE, nspect);
  myCuDMalloc(complexFormat, d_spect, nspect);
  myCuDMalloc(Real, d_spectrum, nspect);
  myCuDMalloc(complexFormat, d_fulltraces, nspect*nfulldelay);
  myCuDMalloc(Real, d_traceIntensity, nspect*ndelay);
  myCuDMalloc(complexFormat, d_traces, nspect*ndelay);
  //Generate electric field, and write to file
  resize_cuda_image(nspect,1);
  genEComplex(d_cE);
  myDMalloc(complexFormat, ccE, nspect);
  myMemcpyD2H(ccE, d_cE, sizeof(complexFormat)*nspect);
  saveWave("input.txt", ccE, nspect);

  //calculate the spectrum, and write to file
  int singleplan;
  createPlan1d(&singleplan, nspect);
  myFFTM(singleplan, d_cE, d_spect);
  applyNorm(d_spect, 1./sqrt(nspect));
  getMod2(d_spectrum, d_spect);
  cudaConvertFO(d_spect);
  convertFOPhase(d_spect);
  myMemcpyD2H(ccE, d_spect, sizeof(complexFormat)*nspect);
  saveWave("inputSpect.txt", ccE, nspect);
  //calculate the complete FROG trace and plot
  resize_cuda_image(nfulldelay,1);
  setDelay(d_fulldelays);
  init_fft(nspect,1,nfulldelay);
  resize_cuda_image(nfulldelay,nspect);
  plt.init(nfulldelay,nspect);
  genTrace(d_cE, d_fulltraces, d_fulldelays);
  convertFOy(d_fulltraces);
  plt.plotComplex(d_fulltraces, MOD2, 0, 1, "trace_truth", 1, 0, 1);
  //downsampling, calculate downsampled trace and plot.
  for(int i = 0; i < ndelay; i++){
    delays[i] = i*(nfulldelay-1)/(ndelay-1) - nfulldelay/2;
  }
  myMemcpyH2D(d_delays, delays, ndelay*sizeof(Real));
  resize_cuda_image(ndelay,nspect);
  plt.init(ndelay,nspect);
  init_fft(nspect,1,ndelay);
  genTrace(d_cE, d_traces, d_delays, nspectm);
  getMod2(d_traceIntensity, d_traces);
  applyNorm(d_traceIntensity, 1./nspect);
  convertFOy(d_traces);
  plt.plotComplex(d_traces, MOD2, 0, 1, "trace_sampled", 1, 0, 1);
  //Reconstruct electric field
  clearCuMem(d_cE,  nspect*sizeof(complexFormat));
  clearCuMem(d_traces,  nspect*ndelay*sizeof(complexFormat));
  //solveE(d_cE, d_traceIntensity, 0, d_traces, delays, nfulldelay, singleplan, nspectm); // spectrum is unknown
  solveE(d_cE, d_traceIntensity, d_spectrum, d_traces, delays, nfulldelay, singleplan, nspectm); //spectrum is known

  //save electric field to file
  myMemcpyD2H(ccE, d_cE, sizeof(complexFormat)*nspect);
  saveWave("output.txt", ccE, nspect);
  //save spectrum to file
  myFFTM(singleplan, d_cE, d_spect);
  resize_cuda_image(nspect,1);
  applyNorm(d_spect, 1./sqrt(nspect));
  cudaConvertFO(d_spect);
  convertFOPhase(d_spect);
  myMemcpyD2H(ccE, d_spect, sizeof(complexFormat)*nspect);
  saveWave("outputSpect.txt", ccE, nspect);
  //calculate reconstructed complete trace
  init_fft(nspect,1,nfulldelay);
  resize_cuda_image(nfulldelay,nspect);
  plt.init(nfulldelay,nspect);
  genTrace(d_cE, d_fulltraces, d_fulldelays);
  convertFOy(d_fulltraces);
  plt.plotComplex(d_fulltraces, MOD2, 0, 1, "trace_recon_full", 1, 0, 1);
}
