#include"cnpy.h"
#include"cudaConfig.h"
#include"format.h"
#include"common.h"
#include<complex>
#include<cstdlib>
#include<iostream>
#include<map>
#include<string>
#include<fstream>
#include"monoChromo.h"

int main(int argc, const char* argv[])
{
    cnpy::npz_t my_npz = cnpy::npz_load("Example_whitelight_experiment.npz"); //arrays are saved in double
    cnpy::NpyArray b = my_npz["B"];
    cnpy::NpyArray spectra = my_npz["spect_I"];
    cnpy::NpyArray lambdas = my_npz["spect_lambda"];
    printf("numpy word size= %ld, double=%ld\n", b.word_size, sizeof(double));
    int row = b.shape[0];
    int col = b.shape[1];
    int nlambda = lambdas.shape[0];
    double* spectrad = spectra.data<double>();
    double* lambdasd = lambdas.data<double>();
    printf("image size = (%d, %d), spectra size = %d\n", row, col, nlambda);
    int mynlambda = int(row*(lambdasd[nlambda-1]/lambdasd[0]-1))/2;
    Real dlambda = 2./row;
    Real* myspectra = (Real*)ccmemMngr.borrowCache(sizeof(Real)*mynlambda);
    Real* mylambdas = (Real*)ccmemMngr.borrowCache(sizeof(Real)*mynlambda);
    Real nextlmd = 1;
    int ilmd = 0;
    for(int i = 0; i < nlambda; i++){
      Real currlmd = lambdasd[i]/lambdasd[0];
      if(currlmd >= nextlmd){
        mylambdas[ilmd] = nextlmd;
        nextlmd+=dlambda;
        myspectra[ilmd++] = spectrad[i];
      }
    }
    Real intensitysum = 0;
    for(int i = 0; i < mynlambda; i++){
      intensitysum+=myspectra[i];
    }
    for(int i = 0; i < mynlambda; i++){
      myspectra[i]/=intensitysum;
    }
    std::ofstream file;
    file.open("spectra.txt", std::ios::out);
    for(int i = 0; i < mynlambda; i++){
      //myspectra[i] = 0.01;
      file<<mylambdas[i]<<" "<<myspectra[i]<<std::endl;
    }
    file.close();
    Real* realb = (Real*)memMngr.borrowCache(sizeof(Real)*row*col);
    double* doubleb = (double*)memMngr.useOnsite(sizeof(double)*row*col);
    hipMemcpy(doubleb, b.data<double>(), sizeof(double)*row*col, hipMemcpyHostToDevice);
    init_cuda_image(row, col, 65535, 1);
    init_fft(row,col);
    cudaF(assignVal)(realb, doubleb);
    cudaF(applyNorm)(realb, 1./intensitysum);
    monoChromo mwl;
    printf("init monochrom\n");
    mwl.init(row, col, mynlambda, mylambdas, myspectra);
    plt.init(row, col);
    complexFormat* complexpattern = (complexFormat*)memMngr.borrowCache(sizeof(double)*row*col);
    complexFormat* solved = (complexFormat*)memMngr.borrowCache(sizeof(double)*row*col);
    cudaF(extendToComplex)(realb, complexpattern);
    plt.plotComplex(complexpattern,REAL,0,1,"logbroadpattern",1);
    plt.plotComplex(complexpattern,REAL,0,1,"broadpattern",0);
    printf("solving matrix\n");
    mwl.solveMWL(complexpattern, solved, 0, 30, 1, 0);
    for(int i = 0; i < mynlambda; i++){
      myspectra[i] = 0.01;
    }
    mwl.solveMWL(complexpattern, solved, solved, 300, 0, 1);
    plt.plotComplex(solved,REAL,0,1,"logmonopattern",1);
    plt.plotComplex(solved,REAL,0,1,"monopattern",0);
    cudaF(getMod)(realb, solved);
    plt.saveFloat(realb, "pattern");
    myCufftExec( *plan, complexpattern, complexpattern, HIPFFT_BACKWARD);
    cudaF(applyNorm)(complexpattern,1./col);
    plt.plotComplex(complexpattern, MOD, 1, 1, "autocbroad", 1);
    myCufftExec( *plan, solved, complexpattern, HIPFFT_BACKWARD);
    cudaF(applyNorm)(complexpattern,1./col);
    plt.plotComplex(complexpattern, MOD, 1, 1, "autocsolved", 1);
    file.open("spectra_new.txt", std::ios::out);
    for(int i = 0; i < mynlambda; i++){
      file<<mylambdas[i]<<" "<<myspectra[i]<<std::endl;
    }
    file.close();

    return 0;
}
