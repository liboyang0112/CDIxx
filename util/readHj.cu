#include"cnpy.h"
#include"cudaConfig.h"
#include"format.h"
#include"cuPlotter.h"
#include"common.h"
#include<complex>
#include<cstdlib>
#include<iostream>
#include<random>
#include<chrono>
#include<map>
#include<string>
#include<fstream>
#include"monoChromo.h"

int main(int argc, const char* argv[])
{
    cnpy::npz_t my_npz = cnpy::npz_load("Example_whitelight_experiment.npz"); //arrays are saved in double
    cnpy::NpyArray b = my_npz["B"];
    cnpy::NpyArray spectra = my_npz["spect_I"];
    cnpy::NpyArray lambdas = my_npz["spect_lambda"];
    printf("numpy word size= %ld, double=%ld\n", b.word_size, sizeof(double));
    int row = b.shape[0];
    int col = b.shape[1];
    int nlambda = lambdas.shape[0];
    double* spectrad = spectra.data<double>();
    double* lambdasd = lambdas.data<double>();
    printf("image size = (%d, %d), spectra size = %d\n", row, col, nlambda);
    int mynlambda = int(row*(lambdasd[nlambda-1]/lambdasd[0]-1))/2/3;
    Real dlambda = 2./row*3;
    Real* myspectra = (Real*)ccmemMngr.borrowCache(sizeof(Real)*mynlambda);
    Real* myspectra1 = (Real*)ccmemMngr.borrowCache(sizeof(Real)*mynlambda);
    Real* mylambdas = (Real*)ccmemMngr.borrowCache(sizeof(Real)*mynlambda);
    Real nextlmd = 1;
    int ilmd = 0;
    for(int i = 0; i < nlambda; i++){
      Real currlmd = lambdasd[i]/lambdasd[0];
      if(currlmd >= nextlmd){
        mylambdas[ilmd] = nextlmd;
        nextlmd+=dlambda;
        myspectra[ilmd++] = spectrad[i];
      }
    }
    Real intensitysum = 0;
    for(int i = 0; i < mynlambda; i++){
      intensitysum+=myspectra[i];
    }
    for(int i = 0; i < mynlambda; i++){
      myspectra[i]/=intensitysum;
    }
    std::ofstream file;
    file.open("spectra.txt", std::ios::out);
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::normal_distribution<double> distribution(0.0, 1.);
    for(int i = 0; i < mynlambda; i++){
      //myspectra[i]+= distribution(generator)*0.002;
      file<<mylambdas[i]<<" "<<myspectra[i]<<std::endl;
    }
    file.close();
    Real* realb = (Real*)memMngr.borrowCache(sizeof(Real)*row*col);
    double* doubleb = (double*)memMngr.useOnsite(sizeof(double)*row*col);
    hipMemcpy(doubleb, b.data<double>(), sizeof(double)*row*col, hipMemcpyHostToDevice);
    init_cuda_image(row, col, 65535, 1);
    cudaF(assignVal)(realb, doubleb);
    cudaF(applyNorm)(realb, 1./intensitysum);
    monoChromo mwl;
    printf("init monochrom\n");
    mwl.init(row, col, mynlambda, mylambdas, myspectra);
    plt.init(row, col);
    complexFormat* complexpattern = (complexFormat*)memMngr.borrowCache(sizeof(double)*row*col);
    complexFormat* solved = (complexFormat*)memMngr.borrowCache(sizeof(double)*row*col);
    cudaF(extendToComplex)(realb, complexpattern);
    plt.plotComplex(complexpattern,REAL,0,1,"logbroadpattern",1);
    plt.plotComplex(complexpattern,REAL,0,1,"broadpattern",0);
    printf("solving matrix\n");
    mwl.solveMWL(complexpattern, solved, 1, 20, 1, 0);
    for(int i = 0; i < mynlambda; i++){
      myspectra[i] = 1./mynlambda;
    }
    //mwl.solveMWL(complexpattern, solved, 0, 1000, 0, 1);

    //myspectra = mwl.spectra;
    //for(int i = 0; i < mynlambda; i++){
    //  myspectra1[i] = 0;
    //  for(int j = -2; j < 3; j ++){
    //    if(i+j >= 0 && i+j < mynlambda)
    //    myspectra1[i] += myspectra[i+j]/5;
    //  }
    //}
    //for(int i = 0; i < mynlambda; i++){
    //  myspectra[i] = 0;
    //  for(int j = -2; j < 3; j ++){
    //    if(i+j >= 0 && i+j < mynlambda){
    //      myspectra[i] += myspectra1[i+j]/5;
    //    }
    //  }
    //}
    //mwl.solveMWL(complexpattern, solved, 1, 500, 1, 0);
    plt.plotComplex(solved,REAL,0,1,"logmonopattern",1);
    plt.plotComplex(solved,REAL,0,1,"monopattern",0);
    cudaF(getMod)(realb, solved);
    plt.saveFloat(realb, "pattern");
    myCufftExec( *plan, complexpattern, complexpattern, HIPFFT_BACKWARD);
    cudaF(applyNorm)(complexpattern,1./col);
    plt.plotComplex(complexpattern, MOD, 1, 1, "autocbroad", 1);
    myCufftExec( *plan, solved, complexpattern, HIPFFT_BACKWARD);
    cudaF(applyNorm)(complexpattern,1./col);
    plt.plotComplex(complexpattern, MOD, 1, 1, "autocsolved", 1);
    mwl.writeSpectra("spectra_new.txt");
    file.close();

    return 0;
}
