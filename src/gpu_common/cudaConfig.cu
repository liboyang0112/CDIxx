#include "hip/hip_runtime.h"
#include "cudaConfig.h"
#include "cudaDefs.h"
#include "imgio.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>
cudaVars* cudaVar = 0;
cudaVars* cudaVarLocal = 0;
dim3 numBlocks;
const dim3 threadsPerBlock = 256;
complexFormat *cudaData = 0;
hipfftHandle *plan, *planR2C;
int2 cuda_imgsz = {0,0};
void cuMemManager::c_malloc(void*& ptr, size_t sz) { gpuErrchk(hipMalloc((void**)&ptr, sz)); }
void cuMemManager::c_memset(void*& ptr, size_t sz) { gpuErrchk(hipMemset(ptr, 0, sz)); }
cuMemManager memMngr;
void myMemcpyH2D(void* d, void* s, size_t sz){
  hipMemcpy(d, s, sz, hipMemcpyHostToDevice);
}
void myMemcpyD2D(void* d, void* s, size_t sz){
  hipMemcpy(d, s, sz, hipMemcpyDeviceToDevice);
}
void myMemcpyD2H(void* d, void* s, size_t sz){
  hipMemcpy(d, s, sz, hipMemcpyDeviceToHost);
}
void resize_cuda_image(int rows, int cols){
  cuda_imgsz.x = rows;
  cuda_imgsz.y = cols;
  numBlocks.x=(rows*cols-1)/threadsPerBlock.x+1;
  //numBlocks.y=(cols-1)/threadsPerBlock.x+1;
}
void init_cuda_image(int rcolor, Real scale){
  const int sz = sizeof(cudaVars);
  if(!cudaVar){
    cudaVar = (cudaVars*) memMngr.borrowCache(sz);
    cudaVarLocal = (cudaVars*) malloc(sz);
    cudaVarLocal->threshold = 0.5;
    cudaVarLocal->beta_HIO = 1;
    if(scale==scale) cudaVarLocal->scale = scale;
    if(rcolor!=0) cudaVarLocal->rcolor = rcolor;
    else cudaVarLocal->rcolor=65535;
    hipMemcpy(cudaVar, cudaVarLocal, sz, hipMemcpyHostToDevice);
    return;
  }
  if(rcolor!=0) cudaVarLocal->rcolor = rcolor;
  if(scale==scale) cudaVarLocal->scale = scale;
  hipMemcpy(cudaVar, cudaVarLocal, sz, hipMemcpyHostToDevice);
};
void* newRand(size_t sz){
  return memMngr.borrowCache(sz * sizeof(hiprandStateMRG32k3a));
}
using namespace std;
static int rows_fft, cols_fft, batch_fft;
__device__ __host__ bool rect::isInside(int x, int y){
  if(x > startx && x <= endx && y > starty && y <= endy) return true;
  return false;
}
__device__ __host__ bool C_circle::isInside(int x, int y){
  Real dr = hypot(Real(x-x0),Real(y-y0));
  if(dr < r) return true;
  return false;
}
void init_fft(int rows, int cols, int batch){
  if(rows!=rows_fft||cols!=cols_fft||batch_fft!=batch){
    if(!plan){
      plan = new hipfftHandle();
      planR2C = new hipfftHandle();
    }else{
      hipfftDestroy(*plan);
      hipfftDestroy(*planR2C);
    }
    if(cols == 1){
      int dim[2] = {rows, batch};
      hipfftPlanMany ( plan, 1, &rows, dim, 1, rows, dim, 1, rows, FFTformat, batch);
      //hipfftPlan1d ( planR2C, rows, FFTformatR2C, batch);
    }else{
      hipfftPlan2d ( plan, rows, cols, FFTformat);
      hipfftPlan2d ( planR2C, rows, cols, FFTformatR2C);
    }
    cols_fft = cols;
    rows_fft = rows;
  }
}
void createPlan(int* handle, int row, int col){
  hipfftPlan2d (handle, row, col, FFTformat);
}
void createPlan1d(int* handle, int n){
  hipfftPlan1d(handle, n, FFTformat, 1);
}
void destroyPlan(int handle){
  hipfftDestroy(handle);
}
void myFFTM(int handle, void* in, void* out){
  myCufftExec(handle, (hipComplex*)in, (hipComplex*)out, HIPFFT_FORWARD);
}
void myIFFTM(int handle, void* in, void* out){
  myCufftExec(handle, (hipComplex*)in, (hipComplex*)out, HIPFFT_BACKWARD);
}
void myFFT(void* in, void* out){
  myCufftExec(*plan, (hipComplex*)in, (hipComplex*)out, HIPFFT_FORWARD);
}
void myIFFT(void* in, void* out){
  myCufftExec(*plan, (hipComplex*)in, (hipComplex*)out, HIPFFT_BACKWARD);
}
void myFFTR2C(void* in, void* out){
  myCufftExecR2C(*planR2C, (Real*)in, (hipComplex*)out);
}
cuFuncTemplate(cudaConvertFO, (T* data, T* out),(data,out==0?data:out),{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= (cuda_row*cuda_column)/2) return;
  int x = index%cuda_row;
  int y = index/cuda_row;
  index = x*cuda_column+y;
  int indexp = (x >= (cuda_row>>1)? x - (cuda_row>>1) : (x + (cuda_row>>1)))*cuda_column + y + (cuda_column>>1);
  T tmp = data[index];
  out[index]=data[indexp];
  out[indexp]=tmp;
})
template void cudaConvertFO<Real>(Real*, Real*);
template<> void cudaConvertFO<complexFormat>(complexFormat* data, complexFormat* out){
  cudaConvertFOWrap<<<numBlocks, threadsPerBlock>>>(cudaVar, cuda_imgsz.x, cuda_imgsz.y, (hipComplex*)data, (hipComplex*)(out==0?data:out));
}

template <typename T1, typename T2>
__global__ void assignValWrap(int cuda_row, int cuda_column, T1* out, T2* input){
  cuda1Idx()
	out[index] = input[index];
}
template <typename T1, typename T2>
void assignVal(T1* out, T2* input){
  assignValWrap<<<numBlocks,threadsPerBlock>>>(cuda_imgsz.x, cuda_imgsz.y,out,input);
}
template void assignVal<Real,Real>(Real*, Real*);
template void assignVal<Real,double>(Real*, double*);
template<> void assignVal<complexFormat, complexFormat>(complexFormat* out, complexFormat* input){
  assignValWrap<<<numBlocks,threadsPerBlock>>>(cuda_imgsz.x, cuda_imgsz.y,(hipComplex*)out,(hipComplex*)input);
}

cuFuncTemplate(crop,(T* src, T* dest, int row, int col, Real midx, Real midy),(src,dest,row,col,midx,midy),{
  cudaIdx()
  int shiftx = int(row*midx);
  if(shiftx + cuda_row/2 > row/2) shiftx = (row-cuda_row)/2;
  else if(shiftx - cuda_row/2 < - row/2) shiftx = (cuda_row-row)/2;
  int shifty = int(col*midy);
  if(shifty + cuda_column/2 > col/2) shifty = (row-cuda_column)/2;
  else if(shifty - cuda_column/2 < - col/2) shifty = (cuda_column-col)/2;
	int targetindex = (x+(row-cuda_row)/2+shiftx)*col + y+(col-cuda_column)/2+shifty;
	dest[index] = src[targetindex];
})
template void crop<Real>(Real*, Real*, int, int, Real, Real);
template<> void crop<complexFormat>(complexFormat* src, complexFormat* dest, int row, int col, Real midx, Real midy){
  cropWrap<<<numBlocks, threadsPerBlock>>>(cudaVar, cuda_imgsz.x, cuda_imgsz.y, (hipComplex*)src, (hipComplex*)dest, row, col, midx, midy);
}


cuFuncc(multiplyShift,(complexFormat* object, Real shiftx, Real shifty),(hipComplex* object, Real shiftx, Real shifty),((hipComplex*)object,shiftx,shifty),{
    cudaIdx();
    Real phi = -2*M_PI*(shiftx*(x-cuda_row/2)/cuda_row+shifty*(y-cuda_column/2)/cuda_column);
    hipComplex tmp = {cos(phi),sin(phi)};
    object[index] = hipCmulf(object[index],tmp);
    })


void shiftWave(complexFormat* wave, Real shiftx, Real shifty){
  myFFT(wave, wave);
  cudaConvertFO(wave);
  multiplyShift(wave, shiftx, shifty);
  cudaConvertFO(wave);
  applyNorm(wave, 1./(cuda_imgsz.x*cuda_imgsz.y));
  myIFFT(wave, wave);
}

cuFuncc(rotateToReal,(complexFormat* data),(hipComplex* data),((hipComplex*)data),{
    cuda1Idx();
    data[index].x = hipCabsf(data[index]);
    data[index].y = 0;
    })

cuFuncc(removeImag,(complexFormat* data),(hipComplex* data),((hipComplex*)data),{
    cuda1Idx();
    data[index].y = 0;
    })

void shiftMiddle(complexFormat* wave){
  cudaConvertFO(wave);
  myFFT(wave, wave);
  rotateToReal(wave);
  applyNorm(wave, 1./(cuda_imgsz.x*cuda_imgsz.y));
  myIFFT(wave, wave);
  cudaConvertFO(wave);
}

__global__ void createGaussKernel(Real* data, int sz, Real sigma){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= sz*sz) return;
  int dx = idx/sz-(sz>>1);
  int dy = (idx%sz)-(sz>>1);
  data[idx] = exp(-Real(dx*dx+dy*dy)/(sigma*sigma));
}

void createGauss(Real* data, int sz, Real sigma){
  createGaussKernel<<<(sz*sz-1)/threadsPerBlock.x+1,threadsPerBlock>>>(data, sz, sigma);
}
void applyGaussConv(Real* input, Real* output, Real* gaussMem, Real sigma){
  int size = floor(sigma*6); // r=3 sigma to ensure the contribution outside kernel is negligible (0.01 of the maximum)
  size = size>>1;
  int width = (size<<1)+1;
  createGauss(gaussMem, width, sigma);
  applyConvolution((sq(width-1+16)+(width*width))*sizeof(Real), input, output, gaussMem, size, size);
}

cuFuncc(fillRedundantR2C,(complexFormat* data, complexFormat* dataout, Real factor),(hipComplex* data, hipComplex* dataout, Real factor),((hipComplex*)data,(hipComplex*)dataout,factor),{
    cudaIdx()
    int targetIndex = x*(cuda_column/2+1)+y;
    if(y <= cuda_column/2) {
    dataout[index].x = data[targetIndex].x*factor;
    dataout[index].y = data[targetIndex].y*factor;
    return;
    }
    if(x==0) {
    targetIndex = cuda_column-y;
    }else{
    targetIndex = (cuda_row-x)*(cuda_column/2+1)+cuda_column-y;
    }
    dataout[index].x = data[targetIndex].x*factor;
    dataout[index].y = -data[targetIndex].y*factor;
    })

cuFuncShared(applyConvolution,(Real *input, Real *output, Real* kernel, int kernelwidth, int kernelheight),
    (input,output,kernel,kernelwidth,kernelheight),
    {
    extern __shared__ float tile[];
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int blockindex = threadIdx.x*blockDim.y+threadIdx.y;
    int tilewidth = kernelwidth*2+blockDim.x;
    int tilesize = tilewidth*tilewidth;
    int blocksize = blockDim.x*blockDim.y;
    for(int filltarget = blockindex; filltarget < tilesize; filltarget+=blocksize){
    int fillx = x - threadIdx.x - kernelwidth + filltarget/tilewidth;
    int filly = y - threadIdx.y - kernelwidth + filltarget%tilewidth;
    if(fillx < 0 || filly < 0 || filly >= cuda_column || fillx >= cuda_row) tile[filltarget] = 0;
    else tile[filltarget] = input[fillx*cuda_column+filly];
    }
    if(blockindex < (2*kernelwidth+1)*(2*kernelheight+1)) tile[tilesize+blockindex] = kernel[blockindex];
    if(x >= cuda_row || y >= cuda_column) return;
    int index = x*cuda_column+y;
    __syncthreads();
    int Idx = (threadIdx.x)*(tilewidth) + threadIdx.y;
    int IdxK = 0;
    Real n_output = 0;
    for(int x1 = -kernelwidth; x1 <= kernelwidth; x1++){
      for(int y1 = -kernelheight; y1 <= kernelheight; y1++){
        n_output+=tile[Idx++]*tile[tilesize+IdxK++];
      }
      Idx+=tilewidth-2*kernelheight-1;
    }
    output[index] = n_output;
    })

cuFuncc(applyNorm,(complexFormat* data, Real factor),(hipComplex* data, Real factor),((hipComplex*)data,factor),{
    cuda1Idx()
    data[index].x*=factor;
    data[index].y*=factor;
    })
cuFunc(applyNorm,(Real* data, Real factor),(data,factor),{
    cuda1Idx()
    data[index]*=factor;
    })
cuFunc(interpolate,(Real* out, Real* data0, Real* data1, Real dx),(out, data0,data1,dx),{
    cuda1Idx()
    out[index] = data0[index]*(1-dx) + data1[index]*dx;
    })
cuFuncc(interpolate,(complexFormat* out, complexFormat* data0, complexFormat* data1, Real dx),(hipComplex* out, hipComplex* data0, hipComplex* data1, Real dx),((hipComplex*)out, (hipComplex*)data0,(hipComplex*)data1,dx),{
    cuda1Idx()
    out[index].x = data0[index].x*(1-dx) + data1[index].x*dx;
    out[index].y = data0[index].y*(1-dx) + data1[index].y*dx;
    })
cuFunc(adamUpdateV,(Real* v, Real* grad, Real beta2),(v,grad,beta2),{
    cuda1Idx()
    Real tmp = grad[index];
    v[index] = tmp*tmp*(1-beta2) + beta2*v[index];
    })
cuFuncc(adamUpdateV,(Real* v, complexFormat* grad, Real beta2),(Real* v, hipComplex* grad, Real beta2),(v,(hipComplex*)grad,beta2),{
    cuda1Idx()
    Real tmp = grad[index].x;
    v[index] = tmp*tmp*(1-beta2) + beta2*v[index];
    })
cuFuncc(adamUpdate,(complexFormat* xn, complexFormat* m, Real* v, Real lr, Real eps),(hipComplex* xn, hipComplex* m, Real* v, Real lr, Real eps),((hipComplex*)xn,(hipComplex*)m,v,lr,eps),{
    cuda1Idx()
    xn[index].x += lr*m[index].x/(sqrt(v[index])+eps);
    })
cuFuncc(ceiling,(complexFormat* data, Real ceilval),(hipComplex* data, Real ceilval),((hipComplex*)data,ceilval),{
    cuda1Idx()
    Real factor = ceilval/hypot(data[index].x, data[index].y);
    if(factor>1) return;
    data[index].x*=factor;
    data[index].y*=factor;
    })
cuFuncc(multiplyReal,(Real* store, complexFormat* src, complexFormat* target),(Real* store, hipComplex* src, hipComplex* target),(store,(hipComplex*)src,(hipComplex*)target),{
    cuda1Idx();
    store[index] = src[index].x*target[index].x;
    })

cuFuncc(multiply,(complexFormat* src, complexFormat* target),(hipComplex* src, hipComplex* target),((hipComplex*)src,(hipComplex*)target),{
    cuda1Idx()
    src[index] = hipCmulf(src[index], target[index]);
    })
cuFuncc(forcePositive,(complexFormat* a),(hipComplex* a),((hipComplex*)a),{
    cuda1Idx()
    if(a[index].x<0) a[index].x=0;
    a[index].y = 0;
    })

cuFunc(forcePositive,(Real* a),(a),{
    cuda1Idx()
    if(a[index]<0) a[index]=0;
    })

cuFuncc(multiply,(complexFormat* store, complexFormat* src, complexFormat* target),(hipComplex* store, hipComplex* src, hipComplex* target),((hipComplex*)store,(hipComplex*)src,(hipComplex*)target),{
    cuda1Idx()
    store[index] = hipCmulf(src[index], target[index]);
    })

cuFunc(multiply,(Real* store, Real* src, Real* target),(store,src,target),{
    cuda1Idx()
    store[index] = src[index]* target[index];
    })

cuFuncc(multiplyPropagatePhase,(complexFormat* amp, Real a, Real b),(hipComplex* amp, Real a, Real b),((hipComplex*)amp,a,b),{ // a=z/lambda, b = (lambda/s)^2, s is the image size
    cudaIdx();
    hipComplex phasefactor;
    Real phase = a*sqrt(1-(sq(x-(cuda_row>>1))+sq(y-(cuda_column>>1)))*b);
    phasefactor.x = cos(phase);
    phasefactor.y = sin(phase);
    amp[index] = hipCmulf(amp[index],phasefactor);
    })

cuFuncc(extendToComplex,(Real* a, complexFormat* b),(Real* a, hipComplex* b),(a,(hipComplex*)b),{
    cuda1Idx()
    b[index].x = a[index];
    b[index].y = 0;
    })

cuFunc(add,(Real* a, Real* b, Real c),(a,b,c),{
    cuda1Idx()
    a[index]+=b[index]*c;
    })

cuFunc(add,(Real* store, Real* a, Real* b, Real c),(store, a,b,c),{
    cuda1Idx()
    store[index] = a[index]+b[index]*c;
    })

cuFuncc(createWaveFront,(Real* d_intensity, Real* d_phase, complexFormat* objectWave, int row, int col, int shiftx, int shifty, Real phaseFactor),(Real* d_intensity, Real* d_phase, hipComplex* objectWave, int row, int col, int shiftx, int shifty, Real phaseFactor),(d_intensity,d_phase,(hipComplex*)objectWave,row,col,shiftx,shifty,phaseFactor),{
    cudaIdx()
    int marginx = (cuda_row-row)/2+shiftx;
    int marginy = (cuda_column-col)/2+shifty;
    if(x<marginx || x >= marginx+row || y < marginy || y >= marginy+col){
    objectWave[index].x = objectWave[index].y = 0;
    return;
    }
    int targetindex = (x-marginx)*col + y-marginy;
    Real phase = phaseFactor*sqSum(x-marginx-(row>>1),y-marginy-(col>>1));
    Real mod = d_intensity?sqrtf(max(0.,d_intensity[targetindex])):1;
    if(d_phase){
    phase += (d_phase[targetindex]-0.5)*2*M_PI;
    }
    if(phase){
    objectWave[index].x = mod*cos(phase);
    objectWave[index].y = mod*sin(phase);
    }else{
    objectWave[index].x = mod;
    objectWave[index].y = 0;
    }
})

cuFuncc(createWaveFront,(Real* d_intensity, Real* d_phase, complexFormat* objectWave, Real oversampling, Real shiftx, Real shifty, Real phaseFactor),(Real* d_intensity, Real* d_phase, hipComplex* objectWave, Real oversampling, Real shiftx, Real shifty, Real phaseFactor),(d_intensity,d_phase,(hipComplex*)objectWave,oversampling,shiftx,shifty,phaseFactor),{
    cudaIdx()
    Real marginratio = (1-1./oversampling)/2;
    int marginx = (marginratio+shiftx)*cuda_row;
    int marginy = (marginratio+shifty)*cuda_column;
    if(x<marginx || x >= cuda_row-marginx || y < marginy || y >= cuda_column-marginy){
    objectWave[index].x = objectWave[index].y = 0;
    return;
    }
    int row = ceil(cuda_row/oversampling);
    int col = ceil(cuda_column/oversampling);
    int targetindex = (x-marginx)*col + y-marginy;
    Real phase = phaseFactor*sqSum(x-marginx-(row>>1),y-marginy-(col>>1));
    Real mod = d_intensity?sqrtf(max(0.,d_intensity[targetindex])):1;
    if(d_phase) phase += (d_phase[targetindex]-0.5)*2*M_PI;
    //Real phase = d_phase? (d_phase[targetindex]-0.5) : 0;
    if(phase){
    objectWave[index].x = mod*cos(phase);
    objectWave[index].y = mod*sin(phase);
    }else{
    objectWave[index].x = mod;
    objectWave[index].y = 0;
    }
})

void readComplexWaveFront(const char* intensityFile, const char* phaseFile, Real* &d_intensity, Real* &d_phase, int &objrow, int &objcol){
  size_t sz = 0;
  if(intensityFile) {
    Real* intensity = readImage(intensityFile, objrow, objcol);
    sz = objrow*objcol*sizeof(Real);
    d_intensity = (Real*)memMngr.borrowCache(sz); //use the memory allocated;
    hipMemcpy(d_intensity, intensity, sz, hipMemcpyHostToDevice);
    ccmemMngr.returnCache(intensity);
  }
  if(phaseFile) {
    int tmprow,tmpcol;
    Real* phase = readImage(phaseFile, tmprow,tmpcol);
    if(!intensityFile) {
      sz = tmprow*tmpcol*sizeof(Real);
      objrow = tmprow;
      objcol = tmpcol;
    }
    d_phase = (Real*)memMngr.borrowCache(sz);
    size_t tmpsz = tmprow*tmpcol*sizeof(Real);
    if(tmpsz!=sz){
      Real* d_phasetmp = (Real*)memMngr.borrowCache(tmpsz);
      gpuErrchk(hipMemcpy(d_phasetmp,phase,tmpsz,hipMemcpyHostToDevice));
      resize_cuda_image(objrow, objcol);
      if(tmpsz > sz){
        crop(d_phasetmp, d_phase, tmprow, tmpcol);
      }else{
        pad(d_phasetmp, d_phase, tmprow, tmpcol);
      }
      memMngr.returnCache(d_phasetmp);
    }
    else {
      gpuErrchk(hipMemcpy(d_phase,phase,sz,hipMemcpyHostToDevice));
    }

    ccmemMngr.returnCache(phase);
  }
  gpuErrchk(hipGetLastError());
}

cuFunc(initRand,(void* state, unsigned long long seed),(state,seed),{
    cuda1Idx()
    hiprand_init(seed,index,0,&((hiprandStateMRG32k3a*)state)[index]);
    })

cuFunc(applyPoissonNoise_WO,(Real* wave, Real noiseLevel, void* state, Real scale),
    (wave,noiseLevel,state,scale),{
    cuda1Idx()
    if(scale==0) scale = vars->scale;
    wave[index]=scale*(int(wave[index]*vars->rcolor/scale) + hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel)-noiseLevel)/vars->rcolor;
    })
cuFunc(ccdRecord, (uint16_t* data, Real* wave, int noiseLevel, void* state, Real exposure),
    (data,wave,noiseLevel,state,exposure),{
    cuda1Idx()
    int dataint = hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel) + vars->rcolor*wave[index]*exposure;
    if(dataint >= vars->rcolor) dataint = vars->rcolor-1;
    data[index] = dataint-noiseLevel;
    });
cuFuncc(ccdRecord, (uint16_t* data, complexFormat* wave, int noiseLevel, void* state, Real exposure),(uint16_t* data, hipComplex* wave, int noiseLevel, void* state, Real exposure),
    (data,(hipComplex*)wave,noiseLevel,state,exposure),{
    cuda1Idx()
    hipComplex tmp = wave[index];
    int dataint = hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel) + vars->rcolor*(tmp.x*tmp.x+tmp.y*tmp.y)*exposure;
    if(dataint >= vars->rcolor) dataint = vars->rcolor-1;
    data[index] = dataint-noiseLevel;
    });
cuFunc(ccdRecord, (Real* data, Real* wave, int noiseLevel, void* state, Real exposure),
    (data,wave,noiseLevel,state,exposure),{
    cuda1Idx()
    int dataint = hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel) + vars->rcolor*wave[index]*exposure;
    if(dataint >= vars->rcolor) dataint = vars->rcolor-1;
    data[index] = Real(dataint-noiseLevel)/vars->rcolor;
    });
cuFuncc(ccdRecord, (Real* data, complexFormat* wave, int noiseLevel, void* state, Real exposure),(Real* data, hipComplex* wave, int noiseLevel, void* state, Real exposure),
    (data,(hipComplex*)wave,noiseLevel,state,exposure),{
    cuda1Idx()
    hipComplex tmp = wave[index];
    int dataint = hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel) + vars->rcolor*(tmp.x*tmp.x+tmp.y*tmp.y)*exposure;
    if(dataint >= vars->rcolor) dataint = vars->rcolor-1;
    data[index] = Real(dataint-noiseLevel)/vars->rcolor;
    });
cuFuncc(ccdRecord, (complexFormat* data, complexFormat* wave, int noiseLevel, void* state, Real exposure),(hipComplex* data, hipComplex* wave, int noiseLevel, void* state, Real exposure),
    ((hipComplex*)data,(hipComplex*)wave,noiseLevel,state,exposure),{
    cuda1Idx()
    hipComplex tmp = wave[index];
    int dataint = hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel) + vars->rcolor*(tmp.x*tmp.x+tmp.y*tmp.y)*exposure;
    if(dataint >= vars->rcolor) dataint = vars->rcolor-1;
    data[index].x = Real(dataint-noiseLevel)/vars->rcolor;
    data[index].y = 0;
    });
cuFunc(applyPoissonNoise,(Real* wave, Real noiseLevel, void* state, Real scale),
    (wave,noiseLevel,state,scale),{
    cuda1Idx()
    hiprand_init(1,index,0,&((hiprandStateMRG32k3a*)state)[index]);
    if(scale==0) scale = vars->scale;
    wave[index]+=scale*(hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel)-noiseLevel)/vars->rcolor;
    })

cuFuncc(getMod,(Real* mod, complexFormat* amp),(Real* mod, hipComplex* amp),(mod,(hipComplex*)amp),{
    cuda1Idx()
    mod[index] = hipCabsf(amp[index]);
    })
cuFuncc(getReal,(Real* mod, complexFormat* amp),(Real* mod, hipComplex* amp),(mod,(hipComplex*)amp),{
    cuda1Idx()
    mod[index] = amp[index].x;
    })
cuFuncc(getImag,(Real* mod, complexFormat* amp),(Real* mod, hipComplex* amp),(mod,(hipComplex*)amp),{
    cuda1Idx()
    mod[index] = amp[index].y;
    })
cuFuncc(assignReal,(Real* mod, complexFormat* amp),(Real* mod, hipComplex* amp),(mod,(hipComplex*)amp),{
    cuda1Idx()
    amp[index].x = mod[index];
    })
cuFuncc(assignImag,(Real* mod, complexFormat* amp),(Real* mod, hipComplex* amp),(mod,(hipComplex*)amp),{
    cuda1Idx()
    amp[index].y = mod[index];
    })
cuFuncc(getMod2,(Real* mod2, complexFormat* amp),(Real* mod2, hipComplex* amp),(mod2,(hipComplex*)amp),{
    cuda1Idx()
    hipComplex tmp = amp[index];
    mod2[index] = tmp.x*tmp.x + tmp.y*tmp.y;
    })
cuFuncc(addMod2,(Real* mod2, complexFormat* amp, Real norm),(Real* mod2, hipComplex* amp, Real norm),(mod2,(hipComplex*)amp,norm),{
    cuda1Idx()
    hipComplex tmp = amp[index];
    mod2[index] += tmp.x*tmp.x*norm + tmp.y*tmp.y*norm;
    })
cuFunc(getMod2,(Real* mod2, Real* mod),(mod2,mod),{
    cuda1Idx()
    mod2[index] = sq(mod[index]);
    })

cuFuncc(bitMap,(Real* store, complexFormat* amp, Real threshold),(Real* store, hipComplex* amp, Real threshold),(store,(hipComplex*)amp, threshold),{
    cuda1Idx()
    hipComplex tmp = amp[index];
    store[index] = tmp.x*tmp.x+tmp.y*tmp.y > threshold*threshold;
    })

cuFunc(bitMap,(Real* store, Real* amp, Real threshold),(store,amp, threshold),{
    cuda1Idx()
    store[index] = amp[index] > threshold;
    })

cuFunc(applyThreshold,(Real* store, Real* input, Real threshold),(store,input,threshold),{
    cuda1Idx()
    store[index] = input[index] > threshold? input[index] : 0;
    })

cuFunc(linearConst,(Real* store, Real* data, Real fact, Real shift),(store, data, fact, shift),{
    cuda1Idx();
    store[index] = fact*data[index]+shift;
    })

cuFuncc(applyModAbs,(complexFormat* source, Real* target, void* state),(hipComplex* source, Real* target, void* state),((hipComplex*)source, target, state),{
    cuda1Idx();
    Real mod = hypot(source[index].x, source[index].y);
    Real rat = target[index];
    if(rat > 0) rat = sqrt(rat);
    else rat = 0;
    if(mod==0) {
    Real randphase = state?hiprand_uniform(&((hiprandStateMRG32k3a*)state)[index])*2*M_PI:0;
    source[index].x = rat*cos(randphase);
    source[index].y = rat*sin(randphase);
    return;
    }
    rat /= mod;
    source[index].x *= rat;
    source[index].y *= rat;
    })
cuFuncc(applyModAbsinner,(complexFormat* source, Real* target,  int row, int col, Real norm, void* state),(hipComplex* source, Real* target,  int row, int col, Real norm, void* state),((hipComplex*)source,target,row,col,norm, state),{
    cudaIdx()
    int targetx = x >= cuda_row/2 ? x - (cuda_row - row) : x;
    int targety = y >= cuda_column/2 ? y - (cuda_column - col) : y;
    Real rat = target[index]*norm;
    index = targetx*col+targety;
    Real mod = hypot(source[index].x, source[index].y);
    if(rat > 0) rat = sqrt(rat);
    else rat = 0;
    if(mod==0) {
    Real randphase = state?hiprand_uniform(&((hiprandStateMRG32k3a*)state)[index])*2*M_PI:0;
    source[index].x = rat*cos(randphase);
    source[index].x = rat*sin(randphase);
    return;
    }
    rat /= mod;
    source[index].x *= rat;
    source[index].y *= rat;
    })

cuFuncc(applyMod,(complexFormat* source, Real* target, Real *bs, bool loose, int iter, int noiseLevel),(hipComplex* source, Real* target, Real *bs, bool loose, int iter, int noiseLevel),
    ((hipComplex*)source, target, bs, loose, iter, noiseLevel),{
    cuda1Idx()
    Real maximum = vars->scale*0.95;
    Real mod2 = target[index];
    if(mod2<0) mod2=0;
    if(loose && bs && bs[index]>0.5) {
    //if(iter > 500) return;
    //else mod2 = maximum+1;
    return;
    }
    Real tolerance = (1.+sqrtf(noiseLevel))*vars->scale/vars->rcolor; // fluctuation caused by bit depth and noise
    hipComplex sourcedata = source[index];
    Real srcmod2 = sourcedata.x*sourcedata.x + sourcedata.y*sourcedata.y;
    if(mod2>=maximum) {
    if(loose) mod2 = max(maximum,srcmod2);
    else tolerance*=1000;
    }
    if(srcmod2 == 0){
    source[index].x = sqrt(mod2);
    source[index].y = 0;
    return;
    }
    Real diff = mod2-srcmod2;
    Real val = mod2;
    if(diff>tolerance){
      val -= tolerance;
    }else if(diff < -tolerance ){
      val += tolerance;
    }
    val = sqrt(val/srcmod2);
    source[index].x = (0.+val)/1.*sourcedata.x;
    source[index].y = (0.+val)/1.*sourcedata.y;
    })
cuFuncc(add,(complexFormat* a, complexFormat* b, Real c ),(hipComplex* a, hipComplex* b, Real c ),((hipComplex*)a,(hipComplex*)b,c),{
    cuda1Idx()
    a[index].x+=b[index].x*c;
    a[index].y+=b[index].y*c;
    })
cuFuncc(convertFOPhase, (complexFormat* data),(hipComplex* data),((hipComplex*)data),{
    cudaIdx()
    if((x+y)%2==1) {
    data[index].x = -data[index].x;
    data[index].y = -data[index].y;
    }
    })
cuFuncc(add,(complexFormat* store, complexFormat* a, complexFormat* b, Real c ),(hipComplex* store, hipComplex* a, hipComplex* b, Real c ),((hipComplex*)store,(hipComplex*)a,(hipComplex*)b,c),{
    cuda1Idx()
    store[index].x=a[index].x + b[index].x*c;
    store[index].y=a[index].y + b[index].y*c;
    })
cuFunc(addRemoveOE, (Real* src, Real* sub, Real mult), (src, sub,mult), {
    cuda1Idx();
    if(sub[index] < 0.99){
    src[index]+=sub[index]*mult;
    }else{
    src[index] = 0;
    }
    });
cuFuncc(applyRandomPhase,(complexFormat* wave, Real* beamstop, void* state),(hipComplex* wave, Real* beamstop, void* state),
    ((hipComplex*)wave, beamstop, state),{
    cuda1Idx()
    hipComplex tmp = wave[index];
    if(beamstop && beamstop[index]>vars->threshold) {
    tmp.x = tmp.y = 0;
    }
    else{
    Real mod = hipCabsf(wave[index]);
    Real randphase = hiprand_uniform(&((hiprandStateMRG32k3a*)state)[index])*2*M_PI;
    tmp.x = mod*cos(randphase);
    tmp.y = mod*sin(randphase);
    }
    wave[index] = tmp;
    })

__device__ hipComplex getFact(Real phase, int l){
  hipComplex nom;
  if(phase != 0){
    nom.x = cos(phase)-1;
    nom.y = sin(phase);
    Real mod2 = sqSum(nom.x, nom.y);
    nom.x = nom.x/mod2;
    nom.y = nom.y/mod2;  //omitted a - sign
    Real nomy = 2*sin(phase*l/2);
    nom.x = nom.y*nomy;
    nom.y = nom.x*nomy;
  }else{
    nom.x = l;
    nom.y = 0;
  }
  return nom;
}
cuFunc(stretch,(Real* src, Real* dest, Real rat, int prec),(src,dest,rat,prec),{
    cudaIdx()
    int targetx = Real(x-cuda_row/2)/rat+cuda_row/2;
    int targety = Real(y-cuda_column/2)/rat+cuda_column/2;
    Real f = cuda_row*cuda_column*rat*rat;
    dest[index] = 0;
    Real sum = 0;
    Real sum1 = 0;
    for(int tx = targetx - prec; tx < targetx+prec; tx++){
      Real phase = 2*M_PI*(Real(x-cuda_row/2)/rat-tx+cuda_row/2)/cuda_row;
      hipComplex factor1 = getFact(phase, cuda_row);
      for(int ty = targety - prec; ty < targety+prec; ty++){
        phase = 2*M_PI*(Real(y-cuda_column/2)/rat-ty+cuda_column/2)/cuda_column;
        hipComplex factor2 = getFact(phase, cuda_row);
        factor2 = hipCmulf(factor1,factor2);
        if(x == 1 && y == 1) {
        sum += factor2.x/f;
        sum1 += factor2.y/f;
        printf("%d, %d, %f, %f\n", tx, ty, factor2.x/f, factor2.y/f);
        }
        dest[index] += src[tx*cuda_row+ty]*factor2.x /f;
      }
    }
    if(x == 1 && y == 1) printf("sum: %f, %f, %f\n", sum, sum1, sqSum(sum, sum1));
    })
cuFunc(cropinner,(Real* src, Real* dest, int row, int col, Real norm),(src,dest,row,col,norm),{
    cudaIdx()
    int targetx = x >= cuda_row/2 ? x + row - cuda_row : x;
    int targety = y >= cuda_column/2 ? y + col - cuda_column : y;
    int targetidx = targetx * col + targety;
    dest[index] = src[targetidx]*norm;
    })
cuFunc(mergePixel, (Real* output, Real* input, int row, int col, int nmerge),(output, input, row, col, nmerge),{
    cudaIdx()
    int idx0 = x*nmerge*col+y*nmerge;
    Real out = 0;
    for(int dx = 0; dx < nmerge; dx ++){
    for(int dy = 0; dy < nmerge; dy ++){
    out += input[idx0];
    idx0++;
    }
    idx0+=col-nmerge;
    }
    output[index] = out/(nmerge*nmerge);
    })

cuFuncc(cropinner,(complexFormat* src, complexFormat* dest, int row, int col, Real norm),(hipComplex* src, hipComplex* dest, int row, int col, Real norm),((hipComplex*)src,(hipComplex*)dest,row,col,norm),{
    cudaIdx()
    int targetx = x >= cuda_row/2 ? x + row - cuda_row : x;
    int targety = y >= cuda_column/2 ? y + col - cuda_column : y;
    int targetidx = targetx * col + targety;
    dest[index].x = src[targetidx].x*norm;
    dest[index].y = src[targetidx].y*norm;
    })
cuFunc(padinner,(Real* src, Real* dest, int row, int col, Real norm),(src,dest,row,col,norm),{
    cudaIdx()
    if((x >= row/2 && x < (cuda_row - row/2)) || (y >= col/2 && y < (cuda_column - col/2))){
    dest[index] = 0;
    return;
    }
    int targetx = x >= cuda_row/2 ? x - (cuda_row - row) : x;
    int targety = y >= cuda_column/2 ? y - (cuda_column - col) : y;
    dest[index] = src[targetx*col+targety]*norm;
    })

cuFuncc(padinner, (complexFormat* src, complexFormat* dest, int row, int col, Real norm),(hipComplex* src, hipComplex* dest, int row, int col, Real norm), ((hipComplex*)src, (hipComplex*)dest, row, col, norm),{
    cudaIdx()
    if((x >= row/2 && x < (cuda_row - row/2)) || (y >= col/2 && y < (cuda_column - col/2))){
    dest[index].x = dest[index].y = 0;
    return;
    }
    int targetx = x >= cuda_row/2 ? x - (cuda_row - row) : x;
    int targety = y >= cuda_column/2 ? y - (cuda_column - col) : y;
    int targetidx = targetx*col+targety;
    dest[index].x = src[targetidx].x*norm;
    dest[index].y = src[targetidx].y*norm;
    })

cuFuncTemplate(createMask, (Real* data, T* spt, bool isFrequency),(data,spt,isFrequency),{
  cudaIdx()
  if(isFrequency){
    if(x>=cuda_row/2) x-=cuda_row/2;
    else x+=cuda_row/2;
    if(y>=cuda_column/2) y-=cuda_column/2;
    else y+=cuda_column/2;
  }
  data[index]=spt->isInside(x,y);
})
template void createMask<rect>(Real*, rect*, bool isFrequency);
template void createMask<C_circle>(Real*, C_circle*, bool isFrequency);
cuFuncTemplate(createMaskBar, (Real* data, T* spt, bool isFrequency),(data,spt,isFrequency),{
  cudaIdx()
  if(isFrequency){
    if(x>=cuda_row/2) x-=cuda_row/2;
    else x+=cuda_row/2;
    if(y>=cuda_column/2) y-=cuda_column/2;
    else y+=cuda_column/2;
  }
  data[index]=!spt->isInside(x,y);
})
template void createMaskBar<rect>(Real*, rect*, bool isFrequency);
template void createMaskBar<C_circle>(Real*, C_circle*, bool isFrequency);
cuFunc(applyMask, (Real* data, Real* mask, Real threshold),(data,mask,threshold),{
  cuda1Idx();
  if(mask[index]<=threshold) data[index] = 0;
})

cuFuncc(applyMask, (complexFormat* data, Real* mask, Real threshold),(hipComplex* data, Real* mask, Real threshold),((hipComplex*)data,mask,threshold),{
  cuda1Idx();
  if(mask[index]<=threshold) data[index].x = data[index].y = 0;
})
cuFunc(applyMaskBar, (Real* data, Real* mask, Real threshold),(data,mask,threshold),{
  cuda1Idx();
  if(mask[index]>threshold) data[index] = 0;
})
cuFuncc(applyMaskBar, (Real* data, complexFormat* mask, Real threshold),(Real* data, hipComplex* mask, Real threshold),(data,(hipComplex*)mask,threshold),{
  cuda1Idx();
  if(mask[index].x>threshold) data[index] = 0;
})
cuFuncc(applyMaskBar, (complexFormat* data, Real* mask, Real threshold),(hipComplex* data, Real* mask, Real threshold),((hipComplex*)data,mask,threshold),{
  cuda1Idx();
  if(mask[index]>threshold) data[index].x = data[index].y = 0;
})
cuFuncc(zeroEdge,(complexFormat* a, int n),(hipComplex* a, int n),((hipComplex*)a,n),{
  cudaIdx()
  if(x<n || x>=cuda_row-n || y < n || y >= cuda_column-n)
    a[index] = hipComplex();
})
cuFunc(zeroEdge,(Real* a, int n),(a,n),{
  cudaIdx()
  if(x<n || x>=cuda_row-n || y < n || y >= cuda_column-n)
    a[index] = 0;
})

cuFuncTemplate(pad,(T* src, T* dest, int row, int col, int shiftx, int shifty),(src, dest, row, col, shiftx, shifty),{
  cudaIdx()
	int marginx = (cuda_row-row)/2+shiftx;
	int marginy = (cuda_column-col)/2+shifty;
	if(x < marginx || x >= row+marginx || y < marginy || y >= col+marginy){
		dest[index] = T();
		return;
	}
	int targetindex = (x-marginx)*col + y-marginy;
	dest[index] = src[targetindex];
})
template void pad<Real>(Real*, Real*, int, int, int, int);
template<> void pad<complexFormat>(complexFormat* src, complexFormat* dest, int row, int col, int shiftx, int shifty){
  padWrap<<<numBlocks, threadsPerBlock>>>(cudaVar, cuda_imgsz.x, cuda_imgsz.y, (hipComplex*)src, (hipComplex*)dest, row, col, shiftx, shifty);
};

cuFuncTemplate(refine,(T* src, T* dest, int refinement),(src,dest,refinement),{
  cudaIdx()
	int indexlu = (x/refinement)*(cuda_row/refinement) + y/refinement;
	int indexld = (x/refinement)*(cuda_row/refinement) + y/refinement+1;
	int indexru = (x/refinement+1)*(cuda_row/refinement) + y/refinement;
	int indexrd = (x/refinement+1)*(cuda_row/refinement) + y/refinement+1;
	Real dx = Real(x%refinement)/refinement;
	Real dy = Real(y%refinement)/refinement;
	dest[index] = 
		src[indexlu]*(1-dx)*(1-dy)
		+((y<cuda_column-refinement)?src[indexld]*(1-dx)*dy:0)
		+((x<cuda_row-refinement)?src[indexru]*dx*(1-dy):0)
		+((y<cuda_column-refinement&&x<cuda_row-refinement)?src[indexrd]*dx*dy:0);
})
template void refine<Real>(Real*, Real*, int);
