
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
float bandwidthtest(int GB){
  size_t sz = 1024*1024;
  sz*= 1024*GB;
  void* memory = malloc(sz); //3G
  void* memorydev;
  hipMalloc(&memorydev, sz); //3G
  time_t current_time = time(NULL);
  hipMemcpy(memorydev, memory, sz, hipMemcpyHostToDevice);
  hipMemcpy(memory, memorydev, sz, hipMemcpyDeviceToHost);
  hipMemcpy(memorydev, memory, sz, hipMemcpyHostToDevice);
  hipMemcpy(memory, memorydev, sz, hipMemcpyDeviceToHost);
  hipMemcpy(memorydev, memory, sz, hipMemcpyHostToDevice);
  hipMemcpy(memory, memorydev, sz, hipMemcpyDeviceToHost);
  printf("time eclipsed: %ld\n", time(NULL)-current_time);
  return 6.*GB/(time(NULL) - current_time);
}
