#include <ctime>
#include "fmt/core.h"
float bandwidthtest(int GB){
  size_t sz = 1024*1024;
  sz*= 1024*GB;
  void* memory = malloc(sz);
  void* memorydev;
  hipMalloc(&memorydev, sz);
  time_t current_time = time(NULL);
  hipMemcpy(memorydev, memory, sz, hipMemcpyHostToDevice);
  hipMemcpy(memory, memorydev, sz, hipMemcpyDeviceToHost);
  hipMemcpy(memorydev, memory, sz, hipMemcpyHostToDevice);
  hipMemcpy(memory, memorydev, sz, hipMemcpyDeviceToHost);
  hipMemcpy(memorydev, memory, sz, hipMemcpyHostToDevice);
  hipMemcpy(memory, memorydev, sz, hipMemcpyDeviceToHost);
  fmt::println("time eclipsed: {}", time(NULL)-current_time);
  return 6.*GB/(time(NULL) - current_time);
}
