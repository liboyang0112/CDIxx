#include "hip/hip_runtime.h"
#include "experimentConfig.h"
#include "cudaConfig.h"

// pixsize*pixsize*M_PI/(d*lambda) and 2*d*M_PI/lambda
cuFunc(multiplyPatternPhase_Device,(cudaVars* vars, complexFormat* amp, Real r_d_lambda, Real d_r_lambda),(vars,amp,r_d_lambda,d_r_lambda),{
  cudaIdx()
  Real phase = (pow(x-(cuda_row>>1),2)+pow(y-(cuda_column>>1),2))*r_d_lambda+d_r_lambda;
  complexFormat p = {cos(phase),sin(phase)};
  amp[index] = hipCmulf(amp[index], p);
})

cuFunc(multiplyPatternPhaseOblique_Device,(cudaVars* vars, complexFormat* amp, Real r_d_lambda, Real d_r_lambda, Real costheta),(vars,amp,r_d_lambda,d_r_lambda,costheta),{ // pixsize*pixsize*M_PI/(d*lambda) and 2*d*M_PI/lambda and costheta = z/r
  cudaIdx()
  Real phase = (pow((x-(cuda_row>>1)*costheta),2)+pow(y-(cuda_column>>1),2))*r_d_lambda+d_r_lambda;
  complexFormat p = {cos(phase),sin(phase)};
  amp[index] = hipCmulf(amp[index], p);
})

cuFunc(multiplyFresnelPhase_Device,(cudaVars* vars, complexFormat* amp, Real phaseFactor),(vars,amp,phaseFactor),{ // pixsize*pixsize*M_PI/(d*lambda) and 2*d*M_PI/lambda
  cudaIdx()
  Real phase = phaseFactor*(pow(x-(cuda_row>>1),2)+pow(y-(cuda_column>>1),2));
  complexFormat p = {cos(phase),sin(phase)};
  if(hipCabsf(amp[index])!=0) amp[index] = hipCmulf(amp[index], p);
})

cuFunc(multiplyFresnelPhaseOblique_Device,(cudaVars* vars, complexFormat* amp, Real phaseFactor, Real costheta_r),(vars,amp,phaseFactor,costheta_r),{ // costheta_r = 1./costheta = r/z
  cudaIdx()
  Real phase = phaseFactor*(pow((x-(cuda_row>>1))*costheta_r,2)+pow(y-(cuda_column>>1),2));
  complexFormat p = {cos(phase),sin(phase)};
  if(hipCabsf(amp[index])!=0) amp[index] = hipCmulf(amp[index], p);
})

void opticalPropagate(void* field, Real lambda, Real d, Real imagesize){
  cudaF(multiplyFresnelPhase_Device,(complexFormat*)field, M_PI/lambda/d*(imagesize*imagesize/cudaVarLocal->rows/cudaVarLocal->cols));
  cudaF(cudaConvertFO,(complexFormat*)field);
  myCufftExec(*plan, (complexFormat*)field, (complexFormat*)field, HIPFFT_FORWARD);
  cudaF(applyNorm,(complexFormat*)field, 1./sqrt(cudaVarLocal->rows*cudaVarLocal->cols));
  cudaF(cudaConvertFO,(complexFormat*)field);
  cudaF(multiplyPatternPhase_Device,(complexFormat*)field, M_PI*lambda*d/(imagesize*imagesize), 2*d*M_PI/lambda - M_PI/2);
}

cuFunc(multiplyPropagatePhase,(cudaVars* vars, complexFormat* amp, Real a, Real b),(vars,amp,a,b),{
  cudaIdx();
  complexFormat phasefactor;
  Real phase = a*sqrt(1-(pow(x-(cuda_row>>1),2)+pow(y-(cuda_column>>1),2))*b);
  phasefactor.x = cos(phase);
  phasefactor.y = sin(phase);
  amp[index] = hipCmulf(amp[index],phasefactor);
})
void angularSpectrumPropagate(void* field, Real imagesize_over_lambda, Real z_over_lambda){
  myCufftExec(*plan, (complexFormat*)field, (complexFormat*)field, HIPFFT_FORWARD);
  cudaF(applyNorm,(complexFormat*)field, 1./(cudaVarLocal->rows*cudaVarLocal->cols));
  cudaF(cudaConvertFO,(complexFormat*)field);
  cudaF(multiplyPropagatePhase,(complexFormat*)field, 2*M_PI*z_over_lambda, 1./(imagesize_over_lambda*imagesize_over_lambda));
  cudaF(cudaConvertFO,(complexFormat*)field);
  myCufftExec(*plan, (complexFormat*)field, (complexFormat*)field, HIPFFT_BACKWARD);
}


void experimentConfig::createBeamStop(){
  C_circle cir;
  cir.x0=row/2;
  cir.y0=column/2;
  cir.r=beamStopSize;
  decltype(cir) *cuda_spt;
  cuda_spt = (decltype(cir)*)memMngr.borrowCache(sizeof(cir));
  hipMemcpy(cuda_spt, &cir, sizeof(cir), hipMemcpyHostToDevice);
  beamstop = (Real*)memMngr.borrowCache(row*column*sizeof(Real));
  cudaF(createMask,beamstop, cuda_spt,1);
  memMngr.returnCache(cuda_spt);
}
void experimentConfig::propagate(void* datain, void* dataout, bool isforward){
  myCufftExec( *plan, (complexFormat*)datain, (complexFormat*)dataout, isforward? HIPFFT_FORWARD: HIPFFT_BACKWARD);
  cudaF(applyNorm,(complexFormat*)dataout, isforward? forwardFactor: inverseFactor);
}
void experimentConfig::multiplyPatternPhase(void* amp, Real distance){
  if(costheta == 1){
    cudaF(multiplyPatternPhase_Device,(complexFormat*)amp,
         pixelsize*pixelsize*M_PI/(distance*lambda),  2*distance*M_PI/lambda-M_PI/2);
  }else{
    cudaF(multiplyPatternPhaseOblique_Device,(complexFormat*)amp,
         pixelsize*pixelsize*M_PI/(distance*lambda),  2*distance*M_PI/lambda-M_PI/2, costheta);
  }
}
void experimentConfig::multiplyPatternPhase_reverse(void* amp, Real distance){
  if(costheta == 1){
    cudaF(multiplyPatternPhase_Device,(complexFormat*)amp,
        -pixelsize*pixelsize*M_PI/(distance*lambda), -2*distance*M_PI/lambda+M_PI/2);
  }else{
    cudaF(multiplyPatternPhaseOblique_Device,(complexFormat*)amp,
        -pixelsize*pixelsize*M_PI/(distance*lambda), -2*distance*M_PI/lambda+M_PI/2, costheta);
  }
}
void experimentConfig::multiplyPatternPhase_factor(void* amp, Real factor1, Real factor2){
  if(costheta == 1){
    cudaF(multiplyPatternPhase_Device,(complexFormat*)amp, factor1, factor2-M_PI/2);
  }else{
    cudaF(multiplyPatternPhaseOblique_Device,(complexFormat*)amp, factor1, factor2-M_PI/2, costheta);
  }
}
void experimentConfig::multiplyFresnelPhase(void* amp, Real distance){
  Real fresfactor = M_PI*lambda*distance/(pow(pixelsize*row,2));
  if(costheta == 1){
    cudaF(multiplyFresnelPhase_Device,(complexFormat*)amp, fresfactor);
  }else{
    cudaF(multiplyFresnelPhaseOblique_Device,(complexFormat*)amp, fresfactor, 1./costheta);
  }
}
void experimentConfig::multiplyFresnelPhase_factor(void* amp, Real factor){
  if(costheta == 1){
    cudaF(multiplyFresnelPhase_Device,(complexFormat*)amp, factor);
  }else{
    cudaF(multiplyFresnelPhaseOblique_Device,(complexFormat*)amp, factor, 1./costheta);
  }
}
void experimentConfig::calculateParameters(){
  enhancement = pow(pixelsize,2)*sqrt(row*column)/(lambda*d); // this guarentee energy conservation
  fresnelFactor = lambda*d/pow(pixelsize,2)/row/column;
  forwardFactor = fresnelFactor*enhancement;
  inverseFactor = 1./row/column/forwardFactor;
}
