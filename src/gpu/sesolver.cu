#include "hip/hip_runtime.h"
#include "cudaDefs_h.cu"
cuFunc(initV,(Real* V, Real val), (V, val), {
  cuda3Idx()
  int &nx = cuda_row;
  int &ny = cuda_column;
  int &nz = cuda_height;
  //Real xmid = nx/2-0.5-2;
  //Real ymid = ny/2-0.5-1;
  //Real zmid = nz/2-0.5;
  //Real r2 = sq(x-xmid) + sq(y-ymid) + sq(z-zmid);
  //V[index] = -val/sqrt(r2);
  Real xmid1 = nx/3-0.5;
  Real ymid1 = ny/3-0.5;
  Real zmid1 = nz/2-0.5;
  Real xmid2 = nx*2/3-0.5;
  Real ymid2 = ny*2/3-0.5;
  Real zmid2 = nz/2-0.5;
  Real r21 = sq(x-xmid1) + sq(y-ymid1) + sq(z-zmid1);
  Real r22 = sq(x-xmid2) + sq(y-ymid2) + sq(z-zmid2);
  V[index] += val/sqrt(r21) + val/sqrt(r22);
  if(r21 < 20 || r22 < 20) printf("V=%f\n", V[index]);
})

cuFunc(Hpsifunc, (Real * psi, Real *V, Real *Hpsi, Real Eshift),
       (psi, V, Hpsi, Eshift), {
         cuda3Idx();
         Hpsi[index] = (Eshift + 6 + V[index]) * psi[index];
         if (z < cuda_height - 1) {
           Hpsi[index] -= psi[index + cuda_row * cuda_column];
         }
         if (z > 0) {
           Hpsi[index] -= psi[index - cuda_row * cuda_column];
         }
         if (x < cuda_row - 1) {
           Hpsi[index] -= psi[index + 1];
         }
         if (x > 0) {
           Hpsi[index] -= psi[index - 1];
         }
         if (y < cuda_column - 1) {
           Hpsi[index] -= psi[index + cuda_row];
         }
         if (y > 0) {
           Hpsi[index] -= psi[index - cuda_row];
         }
       });
cuFunc(getXZSlice, (Real * slice, Real *data, int nx, int ny, int nz, int iy),
       (slice, data, nx, ny, nz, iy), {
         int index = blockIdx.x * blockDim.x + threadIdx.x;
         if (index >= nx * nz)
           return;
         int x = index % nx;
         int z = index / nx;
         slice[index] = data[x + nx * iy + nx * ny * z];
       });
cuFunc(getYZSlice, (Real * slice, Real *data, int nx, int ny, int nz, int ix),
       (slice, data, nx, ny, nz, ix), {
         int index = blockIdx.x * blockDim.x + threadIdx.x;
         if (index >= ny * nz)
           return;
         int y = index % ny;
         int z = index / ny;
         int idx = ix + nx * y + nx * ny * z;
         slice[index] = data[idx];
       })
