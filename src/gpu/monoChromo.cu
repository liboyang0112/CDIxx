#include "hip/hip_runtime.h"
#include "monoChromo.h"
#include "cudaConfig.h"
#include "common.h"
#include "cub_wrap.h"
#include "cuPlotter.h"
#include <fstream>
#include <iostream>
int nearestEven(Real x){
  return round(x/2)*2;
}

__global__ void multiplyReal(complexFormat* a, complexFormat* b, Real* c){
  cudaIdx();
  c[index] = a[index].x*b[index].x;
}

void monoChromo::init(int nrow, int ncol, int nlambda_, Real* lambdas_, Real* spectra_){
  nlambda = nlambda_;
  spectra = spectra_;
  row = nrow;
  column = ncol;
  rows = (int*)ccmemMngr.borrowCache(sizeof(int)*nlambda);
  cols = (int*)ccmemMngr.borrowCache(sizeof(int)*nlambda);
  plt.init(row,column);
  locplan = ccmemMngr.borrowCache(sizeof(hipfftHandle)*nlambda);
  for(int i = 0; i < nlambda; i++){
    rows[i] = nearestEven(row*lambdas_[i]);
    cols[i] = nearestEven(column*lambdas_[i]);
    new((hipfftHandle*)locplan+i)hipfftHandle();
    hipfftPlan2d ( (hipfftHandle*)locplan+i, rows[i], cols[i], FFTformat);
  }
}
void monoChromo::init(int nrow, int ncol, Real* lambdasi, Real* spectrumi, Real endlambda){
  row = nrow;
  column = ncol;
  Real currentLambda = 1;
  int currentPoint = 0;
  int jump = 1;
  Real stepsize = 2./row*jump;
  nlambda = (endlambda-1)/stepsize+1;
  spectra = (Real*) ccmemMngr.borrowCache(nlambda*sizeof(Real));
  int i = 0;
  Real tot = 0;
  while(currentLambda < endlambda){
    int count = 0;
    Real intensity = 0;
    while(lambdasi[currentPoint] < currentLambda+stepsize/2){
      count++;
      intensity += spectrumi[currentPoint];
      currentPoint++;
    }
    if(count >=2 ){ //use average
      spectra[i] = intensity/count;
    }else{ //use interpolation
      if(currentLambda == lambdasi[currentPoint-1]){
        spectra[i] = spectrumi[currentPoint-1];
      }
      else if(currentLambda > lambdasi[currentPoint-1]){
        Real dlambda = lambdasi[currentPoint]-lambdasi[currentPoint-1];
        Real dx = (currentLambda - lambdasi[currentPoint-1])/dlambda;
        spectra[i] = spectrumi[currentPoint-1]*(1-dx) + spectrumi[currentPoint]*(dx);
      }else{
        Real dlambda = lambdasi[currentPoint-1]-lambdasi[currentPoint-2];
        Real dx = (currentLambda - lambdasi[currentPoint-2])/dlambda;
        spectra[i] = spectrumi[currentPoint-2]*(1-dx) + spectrumi[currentPoint-1]*(dx);
      }
    }
    tot+=spectra[i];
    i++;
    currentLambda+=stepsize;
  }
  nlambda = i;
  rows = (int*)ccmemMngr.borrowCache(sizeof(int)*nlambda);
  cols = (int*)ccmemMngr.borrowCache(sizeof(int)*nlambda);
  plt.init(row,column);
  locplan = ccmemMngr.borrowCache(sizeof(hipfftHandle)*nlambda);
  for(int i = 0; i < nlambda; i++){
    rows[i] = row+i*2*jump;
    cols[i] = column+i*2*jump;
    printf("%d: (%d,%d)=%f\n",i, rows[i],cols[i],spectra[i]/=tot);
    new(&(((hipfftHandle*)locplan)[i]))hipfftHandle();
    hipfftPlan2d ( &(((hipfftHandle*)locplan)[i]), rows[i], cols[i], FFTformat);
  }
}
void monoChromo::resetSpectra(){
  for(int i = 0; i < nlambda; i++){
    spectra[i] = 1./nlambda;
  }
}
void monoChromo::writeSpectra(const char* filename){
  std::ofstream spectrafile;
  spectrafile.open(filename,ios::out);
  for(int i = 0; i < nlambda; i++){
    spectrafile<<Real(rows[i])/row<<" "<<spectra[i]<<std::endl;
  }
  spectrafile.close();
}
void monoChromo::generateMWL(void* d_input, void* d_patternSum, void* single, Real oversampling){
  Real *d_pattern = (Real*) memMngr.borrowCache(row*column*sizeof(Real));
  complexFormat *d_intensity = (complexFormat*)memMngr.borrowCache(rows[nlambda-1]*cols[nlambda-1]*sizeof(complexFormat));
  complexFormat* d_patternAmp = (complexFormat*)memMngr.borrowCache(row*column*sizeof(Real)*2);
  for(int i = 0; i < nlambda; i++){
    int thisrow = rows[i];
    int thiscol = cols[i];
    init_cuda_image(thisrow, thiscol, 65536, 1);
    cudaF(createWaveFront)((Real*)d_input, 0, (complexFormat*)d_intensity, row/oversampling, column/oversampling);
    myCufftExec( ((hipfftHandle*)locplan)[i], d_intensity,d_intensity, HIPFFT_FORWARD);
    cudaF(cudaConvertFO)(d_intensity);
    init_cuda_image(row, column, 65536, 1);
    cudaF(crop)(d_intensity,d_patternAmp,thisrow,thiscol);
    cudaF(applyNorm)(d_patternAmp, sqrt(spectra[i])/sqrt(thiscol*thisrow));
    if(i==0) {
      cudaF(getMod2)((Real*)d_patternSum, d_patternAmp);
      if(single!=0) {
        cudaF(extendToComplex)((Real*)d_patternSum, (complexFormat*)single);
        cudaF(applyNorm)((complexFormat*)single, 1./spectra[i]);
      }
    }else{
      cudaF(getMod2)(d_pattern, d_patternAmp);
      cudaF(add)((Real*)d_patternSum, (Real*)d_pattern, 1);
    }
  }
  memMngr.returnCache(d_pattern);
  memMngr.returnCache(d_intensity);
  memMngr.returnCache(d_patternAmp);
}
__global__ void printpix(Real* input, int x, int y){
  printf("%f", input[x*cuda_column+y]);
}
__global__ void printpixreal(complexFormat* input, int x, int y){
  printf("%f,", input[x*cuda_column+y].x);
}
void monoChromo::solveMWL(void* d_input, void* d_output, bool restart, int nIter, bool updateX, bool updateA)
{
  if(nlambda<0) printf("nlambda not initialized: %d\n",nlambda);
  size_t sz = row*column*sizeof(complexFormat);
  complexFormat *fftb = (complexFormat*)memMngr.borrowCache(sz);
  init_fft(row,column);
  init_cuda_image(row, column, 65536, 1);
  Real dt = 2;
  Real friction = 0.2;
  if(restart) hipMemcpy(d_output, d_input, sz, hipMemcpyDeviceToDevice);
  complexFormat *deltab = (complexFormat*)memMngr.borrowCache(sz);
  complexFormat *fbi = (complexFormat*)memMngr.borrowCache(sz);
  complexFormat *momentum = (complexFormat*)memMngr.borrowCache(sz);
  hipMemset(momentum, 0, sz);
  complexFormat *padded = (complexFormat*) memMngr.borrowCache(sizeof(complexFormat)*rows[nlambda-1]*cols[nlambda-1]);
  complexFormat *deltabprev = 0;
  Real *multiplied = 0;
  Real *momentum_a = 0;
  float step_a = 0;
  if(updateA){
    deltabprev = (complexFormat*)memMngr.borrowCache(sz);
    multiplied = (Real*)memMngr.borrowCache(sz/2);
    cudaF(getMod2)(multiplied, (complexFormat*)d_input);
    Real mod2ref = findSum(multiplied, row*column);
    printf("normalization: %f\n",mod2ref);
    step_a = 1./(mod2ref*nlambda);
    if(step_a<=0 || step_a!=step_a) abort();
    momentum_a = (Real*) ccmemMngr.borrowCache(nlambda*sizeof(Real));
    memset(momentum_a,0,nlambda*sizeof(Real));
  }
  Real stepsize = 0.5;
  if(!updateX) {
    myCufftExec( *plan, (complexFormat*)d_output, fftb, HIPFFT_BACKWARD);
    cudaF(cudaConvertFO)(fftb);
  }
  for(int i = 0; i < nIter; i++){
    bool updateAIter = (updateA&&(i%5==0) || updateX==0) && (i > 0);
    if(updateAIter){
      auto tmp_swap = deltabprev;
      deltabprev = deltab;
      deltab = tmp_swap;
    }
    Real sumspect = 0;
    if(updateX) {
      myCufftExec( *plan, (complexFormat*)d_output, fftb, HIPFFT_BACKWARD);
      cudaF(cudaConvertFO)(fftb);
    }
    hipMemcpy(deltab, d_input, sz, hipMemcpyDeviceToDevice);
    cudaF(add)(deltab, (complexFormat*)d_output, -spectra[0]);
    if(updateAIter) {
      cudaF(multiplyReal)(deltabprev, (complexFormat*)d_output, multiplied);
      Real sum =findSum(multiplied, row*column, false);
      if(fabs(sum) > 1e3) {
        sum =findSum(multiplied, row*column, false);
        printf("WARING recalculated sum %f\n", sum);
        exit(0);
      }
      Real step = step_a*sum;
      momentum_a[0] += step*dt;
      momentum_a[0]*=(1-friction*dt);
      //spectra[0]+=step;
      spectra[0]+=momentum_a[0]*dt;
      if(spectra[0]<=0) spectra[0] = 1e-6;
      if(spectra[0]>0.03) spectra[0] = 0.03;
      sumspect+=spectra[0];
    }
    for(int j = 1; j < nlambda; j++){
      if(spectra[j]<=0) continue;
      size_t N = rows[j]*cols[j];
      init_cuda_image(rows[j], cols[j], 65536, 1);
      cudaF(pad)(fftb, padded, row, column);
      cudaF(cudaConvertFO)(padded);
      cudaF(applyNorm)(padded, 1./N);
      myCufftExec( ((hipfftHandle*)locplan)[j], padded, padded, HIPFFT_FORWARD);
      init_cuda_image(row, column, 65536, 1);
      cudaF(crop)(padded, fbi, rows[j], cols[j]);
      cudaF(add)(deltab, fbi, -spectra[j]);
      if(updateAIter) {
        cudaF(multiplyReal)(deltabprev, fbi, multiplied);
        Real sum = findSum(multiplied,row*column);
        if(fabs(sum) > 1e3) {
          sum = findSum(multiplied,row*column);
          printf("WARING recalculated sum %f\n", sum);
          exit(0);
        }
        Real step = step_a*Real(rows[j])/row*sum;
        //if(i > 100 && j == 86){
        //plt.plotComplex(deltabprev, REAL, 0, 1, "deltabprev", 1);
        //plt.plotComplex(fbi, REAL, 0, 1, "fbi", 1);
        //plt.plotFloat(multiplied, MOD, 0, 1, "multiplied", 1);
        //exit(0);
        //}
        //printf("momentum[%d]=%f+%f=%f\n",j, momentum_a[j], step*dt, momentum_a[j]+step*dt);
        momentum_a[j] += step*dt;
        momentum_a[j]*=(1-friction*dt);
        //printf("spectra[%d]=%f+%f=%f\n",j, spectra[j], momentum_a[j]*dt, spectra[j]+momentum_a[j]*dt);
        //printf("spectra[%d]=%f+%f=%f\n",j, spectra[j], step, spectra[j]+step);
        spectra[j]+=momentum_a[j]*dt;
        //spectra[j]+=step;
        if(spectra[j]<=0) spectra[j] = 1e-6;
        if(spectra[j]>0.3) spectra[j] = 0.3;
        sumspect+=spectra[j];
      }
    }
    if(updateAIter){
      //if(i > 10) exit(0);
      for(int j = 0; j < nlambda; j++){
        spectra[j]-=(sumspect-1)/nlambda;
        if(spectra[j]<=0) spectra[j] = 1e-6;
        if(spectra[j]>0.3) spectra[j] = 0.3;
      }
    }
    if(updateX){
      cudaF(add)((complexFormat*)momentum, deltab, stepsize*spectra[0]);
      if(i==nIter-1) {
        plt.plotComplex(deltab, MOD, 0, 1, "residual", 1);
        //cudaF(add)((complexFormat*)d_input, deltab, -1);
        //break;
      }
      for(int j = 1; j < nlambda; j++){
        if(spectra[j]<=0) continue;
        init_cuda_image(rows[j], cols[j], 65536, 1);
        cudaF(pad)((complexFormat*)deltab, padded, row, column);
        myCufftExec( ((hipfftHandle*)locplan)[j], padded, padded, HIPFFT_BACKWARD);
        cudaF(cudaConvertFO)(padded);
        init_cuda_image(row, column, 65536, 1);
        cudaF(crop)(padded, fbi, rows[j], cols[j]);
        cudaF(cudaConvertFO)(fbi);
        cudaF(applyNorm)(fbi, 1./(row*column));
        myCufftExec( *plan, fbi, fbi, HIPFFT_FORWARD);
        cudaF(add)((complexFormat*)momentum, fbi, stepsize*spectra[j]);
      }
      cudaF(applyNorm)((complexFormat*)momentum, 1-friction*dt);
      cudaF(add)((complexFormat*)d_output, momentum, dt);
      cudaF(forcePositive)((complexFormat*)d_output);

    }
  }
  if(updateA){
    ccmemMngr.returnCache(momentum_a);
    memMngr.returnCache(deltabprev);
  }
  if(multiplied) memMngr.returnCache(multiplied);
  memMngr.returnCache(momentum);
  memMngr.returnCache(padded);
  memMngr.returnCache(fbi);
  memMngr.returnCache(fftb);
  memMngr.returnCache(deltab);

}
