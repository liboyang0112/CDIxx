#include "hip/hip_runtime.h"
#include <complex>
#include <cassert>
#include <stdio.h>
#include <time.h>
#include <random>
#include <chrono>

#include <stdio.h>
#include "common.h"
#include <ctime>
#include "cudaConfig.h"
#include "experimentConfig.h"
#include "mnistData.h"
#include "tvFilter.h"
#include "cuPlotter.h"
#include "cub_wrap.h"
#include "cdi.h"


//#define Bits 16

__device__ __host__ Real gaussian(Real x, Real y, Real sigma){
  Real r2 = sq(x) + sq(y);
  return exp(-r2/2/sq(sigma));
}

Real gaussian_norm(Real x, Real y, Real sigma){
  return 1./(2*M_PI*sigma*sigma)*gaussian(x,y,sigma);
}

cuFunc(takeMod2Diff,(complexFormat* a, Real* b, Real *output, Real *bs),(a,b,output,bs),{
  cudaIdx()
    Real mod2 = sq(a[index].x)+sq(a[index].y);
  Real tmp = b[index]-mod2;
  if(bs&&bs[index]>0.5) tmp=0;
  else if(b[index]>vars->scale) tmp = vars->scale-mod2;
  output[index] = tmp;
})

cuFunc(takeMod2Sum,(complexFormat* a, Real* b),(a,b),{
  cudaIdx()
    Real tmp = b[index]+sq(a[index].x)+sq(a[index].y);
  if(tmp<0) tmp=0;
  b[index] = tmp;
})


__device__ void ApplyHIOSupport(bool insideS, complexFormat &rhonp1, complexFormat &rhoprime, Real beta){
  if(insideS){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x -= beta*rhoprime.x;
    rhonp1.y -= beta*rhoprime.y;
  }
}

__device__ void ApplyRAARSupport(bool insideS, complexFormat &rhonp1, complexFormat &rhoprime, Real beta){
  if(insideS){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x = beta*rhonp1.x+(1-2*beta)*rhoprime.x;
    rhonp1.y = beta*rhonp1.y+(1-2*beta)*rhoprime.y;
  }
}

__device__ void ApplyPOSERSupport(bool insideS, complexFormat &rhonp1, complexFormat &rhoprime){
  if(insideS && rhoprime.x > 0){
    rhonp1.x = rhoprime.x;
  }else{
    rhonp1.x = 0;
  }
  rhonp1.y = 0;
}
__device__ void ApplyERSupport(bool insideS, complexFormat &rhonp1, complexFormat &rhoprime){
  if(insideS){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x = 0;
    rhonp1.y = 0;
  }
}


__device__ void ApplyPOSHIOSupport(bool insideS, complexFormat &rhonp1, complexFormat &rhoprime, Real beta){
  if(rhoprime.x > 0 && (insideS/* || rhoprime[0]<30./rcolor*/)){
    rhonp1.x = rhoprime.x;
  }else{
    rhonp1.x -= beta*rhoprime.x;
  }
  rhonp1.y -= beta*rhoprime.y;
}
CDI::CDI(const char* configfile):experimentConfig(configfile){
  verbose(4, print())
    if(runSim) d = oversampling_spt*pixelsize*beamspotsize/lambda; //distance to guarentee setups.oversampling
}
void CDI::propagatepupil(complexFormat* datain, complexFormat* dataout, bool isforward){
  myCufftExec( *plan, datain, dataout, isforward? HIPFFT_FORWARD: HIPFFT_BACKWARD);
  applyNorm(dataout, isforward? forwardFactorpupil: inverseFactorpupil);
}
void CDI::propagateMid(complexFormat* datain, complexFormat* dataout, bool isforward){
  myCufftExec( *plan, datain, dataout, isforward? HIPFFT_FORWARD: HIPFFT_BACKWARD);
  applyNorm(dataout, isforward? forwardFactorMid: inverseFactorMid);
}
void CDI::multiplyPatternPhaseMid(complexFormat* amp, Real distance){
  multiplyPatternPhase_factor(amp, resolution*resolution*M_PI/(distance*lambda), 2*distance*M_PI/lambda);
}
void CDI::multiplyFresnelPhaseMid(complexFormat* amp, Real distance){
  Real fresfactor = M_PI*lambda*distance/(sq(resolution*row));
  multiplyFresnelPhase_factor(amp, fresfactor);
}
void CDI::allocateMem(){
  if(objectWave) return;
  printf("allocating memory\n");
  int sz = row*column*sizeof(Real);
  objectWave = (complexFormat*)memMngr.borrowCache(sz*2);
  patternWave = (complexFormat*)memMngr.borrowCache(sz*2);
  autoCorrelation = (complexFormat*)memMngr.borrowCache(sz*2);
  patternData = (Real*)memMngr.borrowCache(sz);
  printf("initializing cuda image\n");
  resize_cuda_image(row,column);
  init_cuda_image(rcolor,1./exposure);
  init_fft(row,column);
  printf("initializing cuda plotter\n");
  plt.init(row,column);
}
void CDI::readObjectWave(){
  if(domnist){
    row = column = 256;
    mnist_dat = new cuMnist(mnistData,1, 3, row, column);
    allocateMem();
    return;
  }
  int objrow,objcol;
  Real* d_intensity = 0;
  Real* d_phase = 0;
  readComplexWaveFront(intensityModulation?common.Intensity:0, phaseModulation?common.Phase:0, d_intensity, d_phase, objrow,objcol);
  row = objrow*oversampling;
  column = objcol*oversampling;
  allocateMem();
  createWaveFront( d_intensity, d_phase, (complexFormat*)objectWave, objrow, objcol);
  if(d_phase) memMngr.returnCache(d_phase);
  if(d_intensity) memMngr.returnCache(d_intensity);
}
void CDI::readPattern(){
  Real* pattern = readImage(common.Pattern, row, column);
  if(cropPattern) {
    Real* tmp = (Real*)memMngr.borrowCache(row*column*sizeof(Real));
    hipMemcpy(tmp, pattern, row*column*sizeof(Real), hipMemcpyHostToDevice);
    int rowtmp = row;
    int coltmp = column;
    row = column = cropPattern;
    allocateMem();
    crop( tmp, patternData, rowtmp, coltmp);
    memMngr.returnCache(tmp);
  }else{
    allocateMem();
    hipMemcpy(patternData, pattern, row*column*sizeof(Real), hipMemcpyHostToDevice);
  }
  ccmemMngr.returnCache(pattern);
  cudaConvertFO(patternData);
  applyNorm(patternData, 1./exposure);
  printf("Created pattern data\n");
}
void CDI::calculateParameters(){
  experimentConfig::calculateParameters();
  if(dopupil) {
    Real k = row*sq(pixelsize)/(lambda*d);
    dpupil = d*k/(k+1);
    resolution = lambda*dpupil/(row*pixelsize);
    printf("Resolution=%4.2fum\n", resolution);
    enhancementpupil = sq(pixelsize)*sqrt(row*column)/(lambda*dpupil); // this guarentee energy conservation
    fresnelFactorpupil = lambda*dpupil/sq(pixelsize)/row/column;
    forwardFactorpupil = fresnelFactorpupil*enhancementpupil;
    inverseFactorpupil = 1./row/column/forwardFactorpupil;
    enhancementMid = sq(resolution)*sqrt(row*column)/(lambda*(d-dpupil)); // this guarentee energy conservation
    fresnelFactorMid = lambda*(d-dpupil)/sq(resolution)/row/column;
    forwardFactorMid = fresnelFactorMid*enhancementMid;
    inverseFactorMid = 1./row/column/forwardFactorMid;
  }
}
void CDI::readFiles(){
  if(runSim) {
    printf("running simulation, reading input images\n");
    readObjectWave();
  }else{
    printf("running reconstruction, reading input pattern\n");
    readPattern();
  }
}
void CDI::setPattern_c(void* pattern){
  cudaConvertFO((complexFormat*)pattern,patternWave);
  getMod2(patternData, patternWave);
  applyRandomPhase(patternWave, useBS?beamstop:0, devstates);
}

void CDI::setPattern(void* pattern){
  cudaConvertFO((Real*)pattern,patternData);
  createWaveFront(patternData, 0, patternWave, 1);
  applyRandomPhase(patternWave, useBS?beamstop:0, devstates);
}
void CDI::init(){
  allocateMem();
  if(useBS) createBeamStop();
  calculateParameters();
  //inittvFilter(row,column);
  createSupport();
  devstates = (hiprandStateMRG32k3a *)memMngr.borrowCache(column * row * sizeof(hiprandStateMRG32k3a));
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  initRand(devstates, seed);
}
void CDI::prepareIter(){
  if(runSim) {
    if(domnist){
      void* intensity = memMngr.borrowCache(row*column*sizeof(Real));
      void* phase = 0;
      ((cuMnist*)mnist_dat)->cuRead(intensity);
      if(phaseModulation) {
        phase = memMngr.borrowCache(row*column*sizeof(Real));
        ((cuMnist*)mnist_dat)->cuRead(phase);
      }
      createWaveFront((Real*)intensity, (Real*)phase, (complexFormat*)objectWave, 1);
      memMngr.returnCache(intensity);
      if(phaseModulation) memMngr.returnCache(phase);
      initSupport();
    }
    if(isFresnel) multiplyFresnelPhase(objectWave, d);
    verbose(2,plt.plotComplex(objectWave, MOD2, 0, 1, "inputIntensity", 0));
    verbose(2,plt.plotComplex(objectWave, PHASE, 0, 1, "inputPhase", 0));
    verbose(4,printf("Generating diffraction pattern\n"));
    propagate(objectWave,patternWave, 1);
    convertFOPhase( patternWave);
    plt.plotComplex(patternWave, PHASE, 1, 1, "init_pattern_phase", 0);
    getMod2(patternData, patternWave);
    if(simCCDbit){
      verbose(4,printf("Applying Poisson noise\n"));
      plt.plotFloat(patternData, MOD, 1, exposure, "theory_pattern", 0);
      plt.plotFloat(patternData, MOD, 1, exposure, "theory_pattern_log", 1);
      auto img = readImage("theory_pattern.png", row, column);
      hipMemcpy(patternData, img, row*column*sizeof(Real),hipMemcpyHostToDevice);
      ccmemMngr.returnCache(img);
      applyPoissonNoise_WO(patternData, noiseLevel, devstates,1);
      applyNorm(patternData, 1./exposure);
      cudaConvertFO(patternData);
    }
    cudaConvertFO(patternData);
    applyNorm(patternData, exposure);
    plt.saveFloat(patternData, "sim_pattern");
    applyNorm(patternData, 1./exposure);
    cudaConvertFO(patternData);
  }
  if(restart){
    complexFormat *wf = (complexFormat*) readComplexImage(common.restart);
    hipMemcpy(patternWave, wf, row*column*sizeof(complexFormat), hipMemcpyHostToDevice);
    verbose(2,plt.plotComplex(patternWave, MOD2, 1, exposure, "restart_pattern", 1));
    ccmemMngr.returnCache(wf);
  }else {
    createWaveFront(patternData, 0, patternWave, 1);
    applyRandomPhase(patternWave, useBS?beamstop:0, devstates);
  }
  verbose(1,plt.plotFloat(patternData, MOD, 1, exposure, "init_logpattern", 1, 0, 1));
  plt.plotFloat(patternData, MOD, 1, exposure, ("init_pattern"+save_suffix).c_str(), 0);
  cudaConvertFO( patternData);
  applyNorm( patternData, exposure);
  cudaConvertFO( patternData);
  applyNorm( patternData, 1./exposure);
}
void CDI::checkAutoCorrelation(){
  size_t sz = row*column*sizeof(Real);
  auto tmp = (complexFormat*)memMngr.useOnsite(sz);
  myCufftExecR2C( *planR2C, patternData, tmp);
  fillRedundantR2C(tmp, autoCorrelation, 1./sqrt(row*column));
  plt.plotComplex(autoCorrelation, IMAG, 1, 1, "autocorrelation_imag", 1);
  plt.plotComplex(autoCorrelation, REAL, 1, exposure, "autocorrelation_real", 1);
  plt.plotComplex(autoCorrelation, MOD, 1, exposure, "autocorrelation", 1);
}
void CDI::createSupport(){
  rect re;
  re.startx = (oversampling_spt-1)/2*row/oversampling_spt-1;
  re.starty = (oversampling_spt-1)/2*column/oversampling_spt-1;
  re.endx = row-re.startx;
  re.endy = column-re.starty;
  cuda_spt = (rect*)memMngr.borrowCache(sizeof(rect));
  hipMemcpy(cuda_spt, &re, sizeof(rect), hipMemcpyHostToDevice);
  support = (Real*)memMngr.borrowCache(row*column*sizeof(Real));
  createMask(support, cuda_spt,0);
  memMngr.returnCache(cuda_spt);
}
void CDI::initSupport(){
  createMask(support, cuda_spt,0);
}
void CDI::saveState(){
  size_t sz = row*column*sizeof(complexFormat);
  void* outputData = ccmemMngr.borrowCache(sz);
  hipMemcpy(outputData, patternWave, sz, hipMemcpyDeviceToHost);
  writeComplexImage(common.restart, outputData, row, column);//save the step
  ccmemMngr.returnCache(outputData);
}

cuFunc(applySupportOblique,(complexFormat *gkp1, complexFormat *gkprime, Algorithm algo, Real *spt, int iter = 0, Real fresnelFactor = 0, Real costheta_r = 1),(gkp1,gkprime,algo,spt,iter,fresnelFactor,costheta_r),{
  cudaIdx()
    bool inside = spt[index] > vars->threshold;
  complexFormat &gkp1data = gkp1[index];
  complexFormat &gkprimedata = gkprime[index];
  if(algo==RAAR) ApplyRAARSupport(inside,gkp1data,gkprimedata,vars->beta_HIO);
  else if(algo==ER) ApplyERSupport(inside,gkp1data,gkprimedata);
  else if(algo==HIO) ApplyHIOSupport(inside,gkp1data,gkprimedata,vars->beta_HIO);
  if(fresnelFactor>1e-4 && iter < 400) {
    if(inside){
      Real phase = M_PI*fresnelFactor*(sq((x-(cuda_row>>1))*costheta_r)+sq(y-(cuda_column>>1)));
      //Real mod = hipCabs(gkp1data);
      Real mod = fabs(gkp1data.x*cos(phase)+gkp1data.y*sin(phase)); //use projection (Error reduction)
      gkp1data.x=mod*cos(phase);
      gkp1data.y=mod*sin(phase);
    }
  }
})


cuFunc(applySupport,(complexFormat *gkp1, complexFormat *gkprime, Algorithm algo, Real *spt, int iter, Real fresnelFactor),(gkp1,gkprime,algo,spt,iter,fresnelFactor),{

  cudaIdx()
    bool inside = spt[index] > vars->threshold;
  complexFormat &gkp1data = gkp1[index];
  complexFormat &gkprimedata = gkprime[index];
  if(algo==RAAR) ApplyRAARSupport(inside,gkp1data,gkprimedata,vars->beta_HIO);
  else if(algo==ER) ApplyERSupport(inside,gkp1data,gkprimedata);
  else if(algo==POSER) ApplyPOSERSupport(inside,gkp1data,gkprimedata);
  else if(algo==POSHIO) ApplyPOSHIOSupport(inside,gkp1data,gkprimedata,vars->beta_HIO);
  else if(algo==HIO) ApplyHIOSupport(inside,gkp1data,gkprimedata,vars->beta_HIO);
  if(fresnelFactor>1e-4 && iter < 400) {
    if(inside){
      Real phase = M_PI*fresnelFactor*(sq(x-(cuda_row>>1))+sq(y-(cuda_column>>1)));
      //Real mod = hipCabs(gkp1data);
      Real mod = fabs(gkp1data.x*cos(phase)+gkp1data.y*sin(phase)); //use projection (Error reduction)
      gkp1data.x=mod*cos(phase);
      gkp1data.y=mod*sin(phase);
    }
  }
})
complexFormat* CDI::phaseRetrieve(){
  int vidhandle = 0;
  if(saveVideoEveryIter){
    vidhandle = plt.initVideo("recon_intensity.mp4","avc1", 24);
    plt.showVid = -1;
  }
  Real beta = -1;
  Real gammas = -1./beta;
  Real gammam = 1./beta;
  Real gaussianSigma = 2.5;

  size_t sz = row*column*sizeof(complexFormat);
  complexFormat *cuda_gkp1 = (complexFormat*)objectWave;

  complexFormat *cuda_gkprime;
  Real *cuda_diff;
  Real *cuda_objMod;
  cuda_diff = (Real*) memMngr.borrowCache(sz/2);
  cuda_gkprime = (complexFormat*)memMngr.borrowCache(sz);
  cuda_objMod = (Real*)memMngr.borrowCache(sz/2);
  hipMemcpy(cuda_diff, patternData, sz/2, hipMemcpyDeviceToDevice);

  //cudaConvertFO( cuda_diff);
  //FISTA(cuda_diff, cuda_diff, 0.01, 80, 0);
  //cudaConvertFO( cuda_diff);
  //plt.plotFloat(cuda_diff, MOD, 1, exposure, "smoothed_pattern",1);
  AlgoParser algo(algorithm);
  int size = floor(gaussianSigma*6);
  size = ((size>>1)<<1)+1;
  Real*  d_gaussianKernel = (Real*) memMngr.borrowCache(size*size*sizeof(Real));
  for(int iter = 0; ; iter++){
    int ialgo = algo.next();
    if(ialgo<0) break;
    //start iteration
    if(ialgo == shrinkWrap){
      getMod2(cuda_objMod,cuda_gkp1);
      applyGaussConv(cuda_objMod, support, d_gaussianKernel, gaussianSigma);
      cudaVarLocal->threshold = findMax(support,row*column)*shrinkThreshold;
      hipMemcpy(cudaVar, cudaVarLocal, sizeof(cudaVars),hipMemcpyHostToDevice);
      if(gaussianSigma>1) {
        gaussianSigma*=0.99;
      }
      continue;
    }
    if(simCCDbit) applyMod(patternWave,cuda_diff, useBS? beamstop:0, !reconAC || iter > 1000,iter, noiseLevel);
    else applyModAbs(patternWave,cuda_diff);
    propagate(patternWave, cuda_gkprime, 0);
    if(costheta == 1) applySupport(cuda_gkp1, cuda_gkprime, (Algorithm)ialgo, support, iter, isFresnel? fresnelFactor:0);
    else applySupportOblique(cuda_gkp1, cuda_gkprime, (Algorithm)ialgo, support, iter, isFresnel? fresnelFactor:0, 1./costheta);
    //update mask
    /* //TV regularization of object field, not quite effective
    getReal(cuda_objMod,cuda_gkp1);
    FISTA(cuda_objMod, cuda_objMod, 2e-2, 1, 0);
    assignReal( cuda_objMod,cuda_gkp1);
    getImag(cuda_objMod,cuda_gkp1);
    FISTA(cuda_objMod, cuda_objMod, 2e-2, 1, 0);
    assignImag( cuda_objMod,cuda_gkp1);
    */
    propagate( cuda_gkp1, patternWave, 1);
    if(saveVideoEveryIter && iter%saveVideoEveryIter == 0){
      plt.toVideo = vidhandle;
      plt.plotComplex(cuda_gkp1, MOD2, 0, 1, ("recon_intensity"+to_string(iter)).c_str(), 0, isFlip, 1);
      plt.toVideo = -1;
    }
    if(iter%100==0) {
      std::string iterstr = to_string(iter);
      if(saveIter){
        plt.plotComplex(cuda_gkp1, MOD2, 0, 1, ("recon_intensity"+iterstr).c_str(), 0, isFlip);
        plt.plotComplex(cuda_gkp1, PHASE, 0, 1, ("recon_phase"+iterstr).c_str(), 0, isFlip);
        plt.plotComplex(patternWave, MOD2, 1, exposure, ("recon_pattern"+iterstr).c_str(), 0);
      }
      if(0&&iter > 1){  //Do Total variation denoising during the reconstruction, disabled because not quite effective.
        takeMod2Diff(patternWave,patternData, cuda_diff, useBS? beamstop:0);
        cudaConvertFO(cuda_diff);
        FISTA(cuda_diff, cuda_diff, 0.01, 80, 0);
        //plt.plotFloat(cuda_diff, MOD, 0, exposure, ("smootheddiff"+iterstr).c_str(), 1);
        cudaConvertFO(cuda_diff);
        plt.plotFloat(cuda_diff, MOD, 1, exposure, ("diff"+iterstr).c_str(), 1);
        takeMod2Sum(patternWave, cuda_diff);
        //plt.plotFloat(cuda_diff, MOD, 1, exposure, ("smoothed"+iterstr).c_str(), 1);
      }
    }
  }
  if(saveVideoEveryIter) plt.saveVideo(vidhandle);
  if(d_gaussianKernel) memMngr.returnCache(d_gaussianKernel);
  verbose(2,plt.plotComplex(patternWave, MOD2, 1, exposure, "recon_pattern", 1, 0))
  if(verbose >= 4){
    cudaConvertFO((complexFormat*)cuda_gkp1, cuda_gkprime);
    propagate(cuda_gkprime, cuda_gkprime, 1);
    plt.plotComplex(cuda_gkprime, PHASE, 1, 1, "recon_pattern_phase", 0, 0);
  }
  plt.plotComplex(cuda_gkp1, MOD2, 0, 1, ("recon_intensity"+save_suffix).c_str(), 0, isFlip);
  plt.plotComplex(cuda_gkp1, PHASE, 0, 1, ("recon_phase"+save_suffix).c_str(), 0, isFlip);
  bitMap( support, support, cudaVarLocal->threshold);
  plt.plotFloat(support, MOD, 0, 1, "support", 0);
  auto mid = findMiddle(support, row*column);
  resize_cuda_image(row/oversampling_spt,column/oversampling_spt);
  complexFormat* tmp = (complexFormat*)memMngr.borrowCache(sizeof(complexFormat*)*(row/oversampling_spt)*(column/oversampling_spt));
  printf("mid= %f,%f\n",mid.x,mid.y);
  crop(cuda_gkp1, tmp, row, column,mid.x,mid.y);
  plt.init(row/oversampling_spt,column/oversampling_spt);
  plt.plotComplex(tmp, MOD2, 0, 1, ("recon_intensity_cropped"+save_suffix).c_str(), 0, isFlip);
  plt.plotComplex(tmp, PHASE, 0, 1, ("recon_phase_cropped"+save_suffix).c_str(), 0, isFlip);
  resize_cuda_image(row,column);
  plt.init(row,column);
  memMngr.returnCache(tmp);

  if(isFresnel) multiplyFresnelPhase(cuda_gkp1, -d);
  plt.plotComplex(cuda_gkp1, PHASE, 0, 1, ("recon_phase_fresnelRemoved"+save_suffix).c_str(), 0, isFlip);
  getMod2( cuda_objMod, patternWave);
  myCufftExecR2C( *planR2C, cuda_objMod, (complexFormat*)cuda_diff);
  fillRedundantR2C((complexFormat*)cuda_diff, cuda_gkprime, 1./sqrt(row*column));
  plt.plotComplex(cuda_gkprime, MOD, 1, exposure, "autocorrelation_recon", 1);
  add( cuda_objMod, patternData, -1);
  plt.plotFloat(cuda_objMod, MOD, 1, exposure, "residual",1);
  applyMod(patternWave,patternData,useBS?beamstop:0,1,nIter, noiseLevel);
  memMngr.returnCache(cuda_gkprime);
  memMngr.returnCache(cuda_objMod);
  memMngr.returnCache(cuda_diff);

  return patternWave;
}
