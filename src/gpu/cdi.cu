#include "hip/hip_runtime.h"
#include <complex>
#include <cassert>
#include <stdio.h>
#include <time.h>
#include <random>

#include <stdio.h>
#include <libconfig.h++>
#include "hipfft/hipfft.h"
#include "common.h"
#include <ctime>
#include "cudaConfig.h"
#include "experimentConfig.h"
#include "tvFilter.h"
#include "cuPlotter.h"
#include "mnistData.h"
#include "cdi.h"

#include <cub/device/device_reduce.cuh>

struct CustomMax
{
  __device__ __forceinline__
    Real operator()(const Real &a, const Real &b) const {
      return (b > a) ? b : a;
    }
};

Real findMax(Real* d_in, int num_items)
{
  Real *d_out = (Real*)memMngr.borrowCache(sizeof(Real));

  void            *d_temp_storage = NULL;
  size_t          temp_storage_bytes = 0;
  CustomMax max_op;
  gpuErrchk(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, max_op, 0));
  d_temp_storage = (Real*)memMngr.borrowCache(temp_storage_bytes);

  // Run
  gpuErrchk(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, max_op, 0));
  Real output;
  hipMemcpy(&output, d_out, sizeof(Real), hipMemcpyDeviceToHost);

  memMngr.returnCache(d_out);
  memMngr.returnCache(d_temp_storage);
  return output;
}

//#define Bits 16

Real gaussian(Real x, Real y, Real sigma){
  Real r2 = pow(x,2) + pow(y,2);
  return exp(-r2/2/pow(sigma,2));
}

Real gaussian_norm(Real x, Real y, Real sigma){
  return 1./(2*M_PI*sigma*sigma)*gaussian(x,y,sigma);
}





__global__ void takeMod2Diff(complexFormat* a, Real* b, Real *output, Real *bs){
  cudaIdx()
    Real mod2 = pow(a[index].x,2)+pow(a[index].y,2);
  Real tmp = b[index]-mod2;
  if(bs&&bs[index]>0.5) tmp=0;
  else if(b[index]>0.99) tmp = 0.99-mod2;
  output[index] = tmp;
}

__global__ void takeMod2Sum(complexFormat* a, Real* b){
  cudaIdx()
    Real tmp = b[index]+pow(a[index].x,2)+pow(a[index].y,2);
  if(tmp<0) tmp=0;
  b[index] = tmp;
}


__device__ void ApplyHIOSupport(bool insideS, complexFormat &rhonp1, complexFormat &rhoprime, Real beta){
  if(insideS){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x -= beta*rhoprime.x;
    rhonp1.y -= beta*rhoprime.y;
  }
}

__device__ void ApplyRAARSupport(bool insideS, complexFormat &rhonp1, complexFormat &rhoprime, Real beta){
  if(insideS){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x = beta*rhonp1.x+(1-2*beta)*rhoprime.x;
    rhonp1.y = beta*rhonp1.y+(1-2*beta)*rhoprime.y;
  }
}

__device__ void ApplyERSupport(bool insideS, complexFormat &rhonp1, complexFormat &rhoprime){
  if(insideS){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x = 0;
    rhonp1.y = 0;
  }
}


__device__ void ApplyPOSHIOSupport(bool insideS, complexFormat &rhonp1, complexFormat &rhoprime, Real beta){
  if(rhoprime.x > 0 && (insideS/* || rhoprime[0]<30./rcolor*/)){
    rhonp1.x = rhoprime.x;
  }else{
    rhonp1.x -= beta*rhoprime.x;
  }
  rhonp1.y -= beta*rhoprime.y;
}
CDI::CDI(const char* configfile):experimentConfig(configfile){
  verbose(2, print())
    if(runSim) d = oversampling_spt*pixelsize*beamspotsize/lambda; //distance to guarentee setups.oversampling
}
void CDI::propagatepupil(complexFormat* datain, complexFormat* dataout, bool isforward){
  myCufftExec( *plan, datain, dataout, isforward? HIPFFT_FORWARD: HIPFFT_BACKWARD);
  applyNorm<<<numBlocks,threadsPerBlock>>>(dataout, isforward? forwardFactorpupil: inverseFactorpupil);
}
void CDI::propagateMid(complexFormat* datain, complexFormat* dataout, bool isforward){
  myCufftExec( *plan, datain, dataout, isforward? HIPFFT_FORWARD: HIPFFT_BACKWARD);
  applyNorm<<<numBlocks,threadsPerBlock>>>(dataout, isforward? forwardFactorMid: inverseFactorMid);
}
void CDI::multiplyPatternPhaseMid(complexFormat* amp, Real distance){
  multiplyPatternPhase_factor(amp, resolution*resolution*M_PI/(distance*lambda), 2*distance*M_PI/lambda);
}
void CDI::multiplyFresnelPhaseMid(complexFormat* amp, Real distance){
  Real fresfactor = M_PI*lambda*distance/(pow(resolution*row,2));
  multiplyFresnelPhase_factor(amp, fresfactor);
}
void CDI::allocateMem(){
  if(objectWave) return;
  printf("allocating memory\n");
  int sz = row*column*sizeof(Real);
  objectWave = (complexFormat*)memMngr.borrowCache(sz*2);
  patternWave = (complexFormat*)memMngr.borrowCache(sz*2);
  autoCorrelation = (complexFormat*)memMngr.borrowCache(sz*2);
  patternData = (Real*)memMngr.borrowCache(sz);
  printf("initializing cuda image\n");
  init_cuda_image(row,column,rcolor,1./exposure);
  init_fft(row,column);
  printf("initializing cuda plotter\n");
  plt.init(row,column);
}
void CDI::readObjectWave(){
  if(domnist){
    row = column = 256;
    mnist_dat = new cuMnist(mnistData.c_str(), 3, row, column);
    allocateMem();
    return;
  }
  Real* intensity = readImage(common.Intensity.c_str(), row, column);
  size_t sz = row*column*sizeof(Real);
  row*=oversampling;
  column*=oversampling;
  allocateMem();
  Real* d_intensity = (Real*)memMngr.borrowCache(sz); //use the memory allocated;
  hipMemcpy(d_intensity, intensity, sz, hipMemcpyHostToDevice);
  ccmemMngr.returnCache(intensity);
  Real* d_phase = 0;
  if(phaseModulation) {
    int tmp;
    Real* phase = readImage(common.Phase.c_str(), tmp,tmp);
    d_phase = support;
    gpuErrchk(hipMemcpy(d_phase, phase, sz, hipMemcpyHostToDevice));
    ccmemMngr.returnCache(phase);
  }
  createWaveFront<<<numBlocks,threadsPerBlock>>>(d_intensity, d_phase, (complexFormat*)objectWave, oversampling);
  memMngr.returnCache(d_intensity);
  // if(setups.useRectHERALDO){
  //   pixeltype *rowp;
  //   for(int i = 0; i < row ; i++){
  //     rowp = intensity.ptr<pixeltype>(i);
  //     for(int j = 0; j < column ; j++){
  //       if(i > row/2 || j > column/2) rowp[j] = rcolor-1;
  //     }
  //   }
  // }
  //if(setups.useGaussionLumination){
  //  //setups.spt = &re;
  //  //if(!setups.useShrinkMap) setups.spt = &cir3;
  //  //diffraction image, either from simulation or from experiments.
  //  auto f = [&](int x, int y, fftw_format &data){
  //    auto tmp = (complex<Real>*)&data;
  //    bool inside = cir3.isInside(x,y);
  //    if(!inside) *tmp = 0.;
  //    *tmp *= gaussian(x-cir2.x0,y-cir2.y0,cir3.r);
  //    //if(cir2.isInside(x,y))printf("%f, ",gaussian(x-cir2.x0,y-cir2.y0,cir2.r/2));
  //  };
  //  imageLoop<decltype(f)>(gkp1,&f,0);
  //}
  //if(setups.useGaussionHERALDO){
  //  auto f = [&](int x, int y, fftw_format &data){
  //    auto tmp = (complex<Real>*)&data;
  //    if(cir2.isInside(x,y)) 
  //      *tmp *= gaussian(x-cir2.x0,y-cir2.y0,cir2.r*4);
  //    else *tmp = gaussian(x-cir2.x0,y-cir2.y0,cir2.r*4);
  //    if(x < row*1/3 && y < row*1/3) *tmp = 0;
  //    //if(cir2.isInside(x,y))printf("%f, ",gaussian(x-cir2.x0,y-cir2.y0,cir2.r/2));
  //  };
  //  imageLoop<decltype(f)>(gkp1,&f,0);
  //}
  //if(useRectHERALDO)
  //  setRectHERALDO<<<numBlocks,threadsPerBlock>>>(objectWave, oversampling);
}
void CDI::readPattern(){
  Real* pattern = readImage(common.Pattern.c_str(), row, column);
  allocateMem();
  hipMemcpy(patternData, pattern, row*column*sizeof(Real), hipMemcpyHostToDevice);
  ccmemMngr.returnCache(pattern);
  cudaConvertFO<<<numBlocks,threadsPerBlock>>>(patternData);
  applyNorm<<<numBlocks,threadsPerBlock>>>(patternData, 1./exposure);
  printf("Created pattern data\n");
}
void CDI::calculateParameters(){
  experimentConfig::calculateParameters();
  Real beta_HIO = 0.9;
  hipMemcpyToSymbol(HIP_SYMBOL(cuda_beta_HIO),&beta_HIO,sizeof(beta_HIO));
  if(dopupil) {
    Real k = row*pow(pixelsize,2)/(lambda*d);
    dpupil = d*k/(k+1);
    resolution = lambda*dpupil/(row*pixelsize);
    printf("Resolution=%4.2fum\n", resolution);
    enhancementpupil = pow(pixelsize,2)*sqrt(row*column)/(lambda*dpupil); // this guarentee energy conservation
    fresnelFactorpupil = lambda*dpupil/pow(pixelsize,2)/row/column;
    forwardFactorpupil = fresnelFactorpupil*enhancementpupil;
    inverseFactorpupil = 1./row/column/forwardFactorpupil;
    enhancementMid = pow(resolution,2)*sqrt(row*column)/(lambda*(d-dpupil)); // this guarentee energy conservation
    fresnelFactorMid = lambda*(d-dpupil)/pow(resolution,2)/row/column;
    forwardFactorMid = fresnelFactorMid*enhancementMid;
    inverseFactorMid = 1./row/column/forwardFactorMid;
  }
}
void CDI::readFiles(){
  if(runSim) {
    printf("running simulation, reading input images\n");
    readObjectWave();
  }else{
    printf("running reconstruction, reading input pattern\n");
    readPattern();
  }
}
void CDI::setPattern_c(void* pattern){
  cudaF(cudaConvertFO)((complexFormat*)pattern,patternWave);
  cudaF(getMod2)(patternData, patternWave);
  applyRandomPhase<<<numBlocks,threadsPerBlock>>>(patternWave, useBS?beamstop:0, devstates);
}

void CDI::setPattern(void* pattern){
  cudaF(cudaConvertFO)((Real*)pattern,patternData);
  cudaF(createWaveFront)(patternData, 0, patternWave, 1);
  applyRandomPhase<<<numBlocks,threadsPerBlock>>>(patternWave, useBS?beamstop:0, devstates);
}
void CDI::init(){
  allocateMem();
  if(useBS) createBeamStop();
  calculateParameters();
  inittvFilter(row,column);
  createSupport();
  devstates = (hiprandStateMRG32k3a *)memMngr.borrowCache(column * row * sizeof(hiprandStateMRG32k3a));
  cudaF(initRand)(devstates);
}
void CDI::prepareIter(){
  if(runSim) {
    if(domnist){
      void* intensity = memMngr.borrowCache(row*column*sizeof(Real));
      void* phase = 0;
      mnist_dat->cuRead(intensity);
      if(phaseModulation) {
        phase = memMngr.borrowCache(row*column*sizeof(Real));
        mnist_dat->cuRead(phase);
      }
      cudaF(createWaveFront)((Real*)intensity, (Real*)phase, (complexFormat*)objectWave, 1);
      memMngr.returnCache(intensity);
      if(phaseModulation) memMngr.returnCache(phase);
      initSupport();
    }
    if(isFresnel) multiplyFresnelPhase(objectWave, d);
    verbose(2,plt.plotComplex(objectWave, MOD2, 0, 1, "inputIntensity", 0));
    verbose(2,plt.plotComplex(objectWave, PHASE, 0, 1, "inputPhase", 0));
    verbose(4,printf("Generating diffraction pattern\n"));
    propagate(objectWave, patternWave, 1);
    getMod2<<<numBlocks,threadsPerBlock>>>(patternData, patternWave);
    if(simCCDbit){
      verbose(4,printf("Applying Poisson noise\n"))
        verbose(1,plt.plotFloat(patternData, MOD, 1, exposure, "theory_pattern", 1));
      cudaF(applyPoissonNoise_WO)(patternData, noiseLevel, devstates);
    }
  }
  if(restart){
    complexFormat *wf = (complexFormat*) readComplexImage(common.restart.c_str());
    hipMemcpy(patternWave, wf, row*column*sizeof(complexFormat), hipMemcpyHostToDevice);
    verbose(2,plt.plotComplex(patternWave, MOD2, 1, exposure, "restart_pattern", 1))
      ccmemMngr.returnCache(wf);
  }else {
    createWaveFront<<<numBlocks,threadsPerBlock>>>(patternData, 0, patternWave, 1);
    verbose(1,plt.plotFloat(patternData, MOD, 1, exposure, "init_logpattern", 1))
      plt.plotFloat(patternData, MOD, 1, exposure, ("init_pattern"+save_suffix).c_str(), 0);
    applyRandomPhase<<<numBlocks,threadsPerBlock>>>(patternWave, useBS?beamstop:0, devstates);
  }
}
void CDI::checkAutoCorrelation(){
  size_t sz = row*column*sizeof(Real);
  auto tmp = (complexFormat*)memMngr.useOnsite(sz*2);
  myCufftExecR2C( *planR2C, patternData, (complexFormat*)tmp);// re-use the memory allocated for pupil
  cudaF(fillRedundantR2C)((complexFormat*)tmp, autoCorrelation, 1./sqrt(row*column));
  plt.plotComplex(autoCorrelation, IMAG, 1, 1, "autocorrelation_imag", 1); // only positive values are shown
  plt.plotComplex(autoCorrelation, REAL, 1, 1, "autocorrelation_real", 1); // only positive values are shown
  plt.plotComplex(autoCorrelation, MOD, 1, 1, "autocorrelation", 1);
}
void CDI::createSupport(){
  rect re;
  re.startx = (oversampling_spt-1)/2*row/oversampling_spt-1;
  re.starty = (oversampling_spt-1)/2*column/oversampling_spt-1;
  re.endx = row-re.startx-2;
  re.endy = column-re.starty-2;
  cuda_spt = (rect*)memMngr.borrowCache(sizeof(rect));
  hipMemcpy(cuda_spt, &re, sizeof(rect), hipMemcpyHostToDevice);
  support = (Real*)memMngr.borrowCache(row*column*sizeof(Real));
  createMask<<<numBlocks,threadsPerBlock>>>(support, cuda_spt,0);
}
void CDI::initSupport(){
  createMask<<<numBlocks,threadsPerBlock>>>(support, cuda_spt,0);
}
void CDI::saveState(){
  size_t sz = row*column*sizeof(complexFormat);
  void* outputData = ccmemMngr.borrowCache(sz);
  hipMemcpy(outputData, patternWave, sz, hipMemcpyDeviceToHost);
  writeComplexImage(common.restart.c_str(), outputData, row, column);//save the step
  ccmemMngr.returnCache(outputData);
}

__global__ void applySupportOblique(complexFormat *gkp1, complexFormat *gkprime, Algorithm algo, Real *spt, int iter = 0, Real fresnelFactor = 0, Real costheta_r = 1){
  cudaIdx()
    bool inside = spt[index] > cuda_threshold;
  complexFormat &gkp1data = gkp1[index];
  complexFormat &gkprimedata = gkprime[index];
  if(algo==RAAR) ApplyRAARSupport(inside,gkp1data,gkprimedata,cuda_beta_HIO);
  else if(algo==ER) ApplyERSupport(inside,gkp1data,gkprimedata);
  else if(algo==HIO) ApplyHIOSupport(inside,gkp1data,gkprimedata,cuda_beta_HIO);
  if(fresnelFactor>1e-4 && iter < 400) {
    if(inside){
      Real phase = M_PI*fresnelFactor*(pow((x-(cuda_row>>1))*costheta_r,2)+pow(y-(cuda_column>>1),2));
      //Real mod = hipCabs(gkp1data);
      Real mod = fabs(gkp1data.x*cos(phase)+gkp1data.y*sin(phase)); //use projection (Error reduction)
      gkp1data.x=mod*cos(phase);
      gkp1data.y=mod*sin(phase);
    }
  }
}


__global__ void applySupport(complexFormat *gkp1, complexFormat *gkprime, Algorithm algo, Real *spt, int iter = 0, Real fresnelFactor = 0){

  cudaIdx()
    bool inside = spt[index] > cuda_threshold;
  complexFormat &gkp1data = gkp1[index];
  complexFormat &gkprimedata = gkprime[index];
  if(algo==RAAR) ApplyRAARSupport(inside,gkp1data,gkprimedata,cuda_beta_HIO);
  else if(algo==ER) ApplyERSupport(inside,gkp1data,gkprimedata);
  //else if(algo==HIO) ApplyPOSHIOSupport(inside,gkp1data,gkprimedata,cuda_beta_HIO);
  else if(algo==HIO) ApplyHIOSupport(inside,gkp1data,gkprimedata,cuda_beta_HIO);
  if(fresnelFactor>1e-4 && iter < 400) {
    if(inside){
      Real phase = M_PI*fresnelFactor*(pow(x-(cuda_row>>1),2)+pow(y-(cuda_column>>1),2));
      //Real mod = hipCabs(gkp1data);
      Real mod = fabs(gkp1data.x*cos(phase)+gkp1data.y*sin(phase)); //use projection (Error reduction)
      gkp1data.x=mod*cos(phase);
      gkp1data.y=mod*sin(phase);
    }
  }
}

complexFormat* CDI::phaseRetrieve(){
  Real beta = -1;
  Real gammas = -1./beta;
  Real gammam = 1./beta;
  Real gaussianSigma = 2.5;

  size_t sz = row*column*sizeof(complexFormat);
  complexFormat *cuda_gkp1 = (complexFormat*)objectWave;

  complexFormat *cuda_gkprime;
  Real *cuda_diff;
  Real *cuda_objMod;
  cuda_diff = (Real*) memMngr.borrowCache(sz/2);
  cuda_gkprime = (complexFormat*)memMngr.borrowCache(sz);
  cuda_objMod = (Real*)memMngr.borrowCache(sz/2);
  hipMemcpy(cuda_diff, patternData, sz/2, hipMemcpyDeviceToDevice);

  AlgoParser algo(algorithm);
  Real* d_gaussianKernel = 0;
  Real* gaussianKernel = 0;
  for(int iter = 0; ; iter++){
    int ialgo = algo.next();
    if(ialgo<0) break;
    //start iteration
    cudaF(applyMod)(patternWave,cuda_diff, useBS? beamstop:0, !reconAC || iter > 1000,iter, noiseLevel);
    propagate(patternWave, cuda_gkprime, 0);
    if(costheta == 1) cudaF(applySupport)(cuda_gkp1, cuda_gkprime, (Algorithm)ialgo, support, iter, isFresnel? fresnelFactor:0);
    else cudaF(applySupportOblique)(cuda_gkp1, cuda_gkprime, (Algorithm)ialgo, support, iter, isFresnel? fresnelFactor:0, 1./costheta);
    //update mask
    if(iter%20==0){
      cudaF(getMod)(cuda_objMod,cuda_gkp1);
      if(iter > 0 && useShrinkMap){
        int size = floor(gaussianSigma*6); // r=3 sigma to ensure the contribution outside kernel is negligible (0.01 of the maximum)
        size = size/2;
        int width = size*2+1;
        int kernelsz = width*width*sizeof(Real);
        if(!d_gaussianKernel){
          d_gaussianKernel = (Real*) memMngr.borrowCache(kernelsz);
          gaussianKernel =  (Real*) ccmemMngr.borrowCache(kernelsz);
        }
        Real total = 0;
        Real weight;
        for(int i = 0; i < width*width; i++) {
          weight = gaussian((i/width-size),i%width-size, gaussianSigma);
          total+= weight;
          gaussianKernel[i] = weight;
        }
        for(int i = 0; i < width*width; i++)
          gaussianKernel[i] /= total;
        hipMemcpy(d_gaussianKernel, gaussianKernel, kernelsz, hipMemcpyHostToDevice);
        applyConvolution<<<numBlocks,threadsPerBlock, pow(size*2+threadsPerBlock.x,2)*sizeof(Real)>>>(cuda_objMod, support, d_gaussianKernel, size, size);

        Real threshold = findMax(support, row*column);
        threshold*=shrinkThreshold;
        hipMemcpyToSymbol(HIP_SYMBOL(cuda_threshold), &threshold, sizeof(threshold));

        if(gaussianSigma>1.5) {
          gaussianSigma*=0.99;
        }
      }
      //if(iter%100==0){
      //tvFilterWrap(cuda_objMod, 2e8, 20);
      //applyMod<<<numBlocks,threadsPerBlock>>>(cuda_gkp1, cuda_objMod);
      //}
    }
    propagate( cuda_gkp1, patternWave, 1);
    if(iter%100==0) {
      std::string iterstr = to_string(iter);
      if(saveIter){
        plt.plotComplex(cuda_gkp1, MOD2, 0, 1, ("recon_intensity"+iterstr).c_str(), 0);
        plt.plotComplex(cuda_gkp1, PHASE, 0, 1, ("recon_phase"+iterstr).c_str(), 0);
        plt.plotComplex(patternWave, MOD2, 1, exposure, ("recon_pattern"+iterstr).c_str(), 1);
      }
      if(0){  //Do Total variation denoising during the reconstruction, disabled because not quite effective.
        takeMod2Diff<<<numBlocks,threadsPerBlock>>>(patternWave,patternData, cuda_diff, useBS? beamstop:0);
        cudaConvertFO<<<numBlocks,threadsPerBlock>>>(cuda_diff);
        tvFilterWrap(cuda_diff, noiseLevel, 200);
        cudaConvertFO<<<numBlocks,threadsPerBlock>>>(cuda_diff);
        plt.plotFloat(cuda_diff, MOD, 1, 1, ("smootheddiff"+iterstr).c_str(), 1);
        takeMod2Sum<<<numBlocks,threadsPerBlock>>>(patternWave, cuda_diff);
        plt.plotFloat(cuda_diff, MOD, 1, 1, ("smoothed"+iterstr).c_str(), 1);
      }
    }
  }
  if(gaussianKernel) ccmemMngr.returnCache(gaussianKernel);
  if(d_gaussianKernel) memMngr.returnCache(d_gaussianKernel);
  verbose(2,plt.plotComplex(patternWave, MOD2, 1, exposure, "recon_pattern", 1))
    if(verbose >= 4){
      cudaF(cudaConvertFO)((complexFormat*)cuda_gkp1, cuda_gkprime);
      propagate(cuda_gkprime, cuda_gkprime, 1);
      plt.plotComplex(cuda_gkprime, PHASE, 1, 1, "recon_pattern_phase", 0);
    }
  applyMod<<<numBlocks,threadsPerBlock>>>(patternWave,patternData,useBS?beamstop:0,1,nIter, noiseLevel);
  plt.plotComplex(cuda_gkp1, MOD2, 0, 1, ("recon_intensity"+save_suffix).c_str(), 0);
  plt.plotComplex(cuda_gkp1, PHASE, 0, 1, ("recon_phase"+save_suffix).c_str(), 0);
  if(isFresnel) multiplyFresnelPhase(cuda_gkp1, -d);
  plt.plotComplex(cuda_gkp1, PHASE, 0, 1, ("recon_phase_fresnelRemoved"+save_suffix).c_str(), 0);
  memMngr.returnCache(cuda_gkprime);
  memMngr.returnCache(cuda_objMod);
  memMngr.returnCache(cuda_diff);

  return patternWave;
}
