#include "hip/hip_runtime.h"
#include "cuPlotter.hpp"
#include "cudaDefs_h.cu"
#include "cudaConfig.hpp"
void cuPlotter::freeCuda(){
  if(cuCache_data) { memMngr.returnCache(cuCache_data); cuCache_data = 0;}
  if(cuCache_float_data) { memMngr.returnCache(cuCache_float_data); cuCache_float_data = 0;}
}

__device__ Real cugetVal(cudaVars*vars, mode m, hipComplex &data, Real decay, bool islog){
  Real target = 0;
  switch(m){
    case IMAG: target = data.y*decay; break;
    case MOD: target = hipCabsf(data)*decay; break;
    case MOD2: target = (data.x*data.x+data.y*data.y)*decay; break;
    case PHASE: target = atan2(data.y,data.x)/2/M_PI+0.5; break;
    case PHASERAD: target = atan2(data.y,data.x); break;
    case REAL:{
      target = data.x*decay;
      if(islog){
        if(target!=0){
          Real ab = fabs(target);
          Real logv = log2f(ab)/log2f(vars->rcolor)+1;
          if(logv < 0) target = 0;
          else target = target*logv/(2*ab);
        }
      }
      return (target+0.5)*vars->rcolor;
    }
    default: ;
  }
  if(target!=0){
    if(islog) target = log2f(target)/log2f(vars->rcolor)+1;
    target*=vars->rcolor;
  }
  return target;
}
__device__ Real cugetVal(cudaVars* vars, mode m, Real &data, Real decay, bool islog){
  Real ret = 0;
  if(m==REAL) {
    ret = data*decay; //-1~1
    if(islog){
      if(ret!=0){
        Real ab = fabs(ret);
        Real logv = log2f(ab)/log2f(vars->rcolor)+1;
        if(logv < 0) ret = 0;
        else ret = ret*logv/(2*ab);
      }
    }
    return (ret+0.5)*vars->rcolor;
  }
  if(m==MOD2) ret = data*data*decay;
  else if(m==MOD) ret = fabs(data)*decay;
  if(ret!=0){
    if(islog) ret = log2f(ret)/log2f(vars->rcolor)+1;
    ret*=vars->rcolor;
  }
  return ret;
}

template <typename T>
__global__ void process(cudaVars* vars, int cuda_row, int cuda_column, void* cudaData, pixeltype* cache, mode m, bool isFrequency=0, Real decay = 1, bool islog = 0, bool isFlip = 0){
  cudaIdx()
  int halfrow = cuda_row>>1;
  int halfcol = cuda_column>>1;
  int targetx = x;
  int targety = y;
  if(isFrequency) {
    targetx = x<halfrow?x+halfrow:(x-halfrow);
    targety = y<halfcol?y+halfcol:(y-halfcol);
  }
  if(isFlip){
    targetx = cuda_row-x-1;
  }
  Real target = cugetVal(vars,m, ((T*)cudaData)[index],decay,islog);
  if(target < 0) target = 0;
  else if(target>=vars->rcolor) {
    target=vars->rcolor-1;
  }
  cache[targetx*cuda_column+targety] = floor(target);
}
void cuPlotter::saveFloatData(void* cudaData){
  hipMemcpy(cv_float_data, cudaData, rows*cols*sizeof(Real), hipMemcpyDeviceToHost);
};
void cuPlotter::saveComplexData(void* cudaData){
  hipMemcpy(cv_complex_data, cudaData, rows*cols*sizeof(complexFormat), hipMemcpyDeviceToHost);
};
void* cuPlotter::processFloatData(void* cudaData, const mode m, bool isFrequency, Real decay, bool islog, bool isFlip){
  if(!cuCache_data) cuCache_data = (pixeltype*) memMngr.borrowCache(rows*cols*sizeof(pixeltype));
  process<Real><<<numBlocks,threadsPerBlock>>>(cudaVar, rows, cols, cudaData, cuCache_data, m, isFrequency, decay, islog, isFlip);
  hipMemcpy(cv_data, cuCache_data,rows*cols*sizeof(pixeltype), hipMemcpyDeviceToHost);
  return cv_data;
};
void* cuPlotter::processComplexData(void* cudaData, const mode m, bool isFrequency, Real decay, bool islog, bool isFlip){
  if(!cuCache_data) cuCache_data = (pixeltype*) memMngr.borrowCache(rows*cols*sizeof(pixeltype));
  process<hipComplex><<<numBlocks,threadsPerBlock>>>(cudaVar, rows, cols, cudaData, cuCache_data, m,isFrequency, decay, islog, isFlip);
  hipMemcpy(cv_data, cuCache_data,rows*cols*sizeof(pixeltype), hipMemcpyDeviceToHost);
  return cv_data;
};
