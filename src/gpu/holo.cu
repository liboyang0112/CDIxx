#include "hip/hip_runtime.h"
#include "holo.h"
#include "common.h"
#include "cuPlotter.h"

cuFunc(applySupportBar,(cudaVars* vars, complexFormat* img, Real* spt),(vars,img,spt),{
  cudaIdx();
  if(spt[index] > vars->threshold)
    img[index].x = img[index].y = 0;
})

cuFunc(applySupport,(cudaVars* vars, complexFormat* img, Real* spt),(vars,img,spt),{
  cudaIdx();
  if(spt[index] < vars->threshold)
    img[index].x = img[index].y = 0;
})

cuFunc(dillate, (cudaVars* vars, complexFormat* data, Real* support, int wid, int hit), (vars,data,support,wid,hit),{
  cudaIdx();
  if(abs(data[index].y) < 0.5 && abs(data[index].y) < 0.5) return;
  int idxp = 0;
  for(int xp = 0; xp < cuda_row; xp++)
  for(int yp = 0; yp < cuda_column; yp++)
  {
    if(abs(xp - x) < wid && abs(yp-y) < hit) support[idxp] = 1;
    if(abs(xp - x) > cuda_row/2 || abs(yp-y)>cuda_column/2) support[idxp] = 1;
    idxp++;
  }
})
cuFunc(applyModCorr, (cudaVars* vars, complexFormat* obj, complexFormat* refer, Real* xcorrelation),(vars,obj,refer,xcorrelation),{
  cudaIdx();
  complexFormat objtmp = obj[index];
  complexFormat reftmp = refer[index];
  Real maximum = vars->scale*0.99;
  Real fact = xcorrelation[index]/2 - reftmp.x*objtmp.x - reftmp.y*objtmp.y;
  fact /= reftmp.x*reftmp.x + reftmp.y*reftmp.y;
  obj[index].x = objtmp.x + fact*reftmp.x;
  obj[index].y = objtmp.y + fact*reftmp.y;
})

holo::holo(const char* configfile) : CDI(configfile){}
void holo::allocateMem_holo(){
  size_t sz = row*column*sizeof(Real);
  patternData_holo = (Real*)memMngr.borrowCache(sz);
  patternData_obj = (Real*)memMngr.borrowCache(sz);
  xcorrelation = (Real*)memMngr.borrowCache(sz);
  support_holo = (Real*)memMngr.borrowCache(sz);
  xcorrelation_support = (Real*)memMngr.borrowCache(sz);
  patternWave_holo = (complexFormat*)memMngr.borrowCache(sz*2);
  patternWave_obj = (complexFormat*)memMngr.borrowCache(sz*2);
  objectWave_holo = (complexFormat*)memMngr.borrowCache(sz*2);
}
void holo::calcXCorrelation(){
  cudaF(add, patternData_holo, patternData, -1);
  cudaF(extendToComplex, patternData_holo, patternWave_holo);
  cudaF(add, patternData_holo, patternData, 1);
  myCufftExec(*plan, patternWave_holo,patternWave_holo,HIPFFT_FORWARD);
  cudaF(applyNorm, patternWave_holo, 1./(row*column));

  rect cir;
  cir.startx=row/2-objrow;
  cir.starty=column/2-objcol;
  cir.endx=row/2+objrow;
  cir.endy=column/2+objcol;
  decltype(cir) *cuda_spt;
  cuda_spt = (decltype(cir)*)memMngr.borrowCache(sizeof(cir));
  hipMemcpy(cuda_spt, &cir, sizeof(cir), hipMemcpyHostToDevice);
  cudaF(createMask, xcorrelation_support, cuda_spt, 1);
  
  cudaF(applySupportBar, patternWave_holo, xcorrelation_support);
  myCufftExec(*plan, patternWave_holo,patternWave_holo,HIPFFT_BACKWARD);
  plt.plotComplex(patternWave_holo, REAL, 1, exposurepupil, "xcorrspt", 1);
  cudaF(getReal, xcorrelation, patternWave_holo);

  hipMemset(support_holo, 0, memMngr.getSize(support_holo));
  cudaF(dillate, (complexFormat*)objectWave, support_holo, row/oversampling-objrow, column/oversampling-objcol);
  plt.plotFloat(support_holo, MOD, 0, 1, "holospt");
}
void holo::simulate(){
  readObjectWave();
  init();
  prepareIter();
  allocateMem_holo();
  Real* intensity = readImage(pupil.Intensity.c_str(), objrow, objcol);
  size_t sz = objrow*objcol*sizeof(Real);
  Real* d_intensity = (Real*)memMngr.borrowCache(sz); //use the memory allocated;
  hipMemcpy(d_intensity, intensity, sz, hipMemcpyHostToDevice);
  ccmemMngr.returnCache(intensity);
  Real* d_phase = 0;
  if(phaseModulation_pupil) {
    int tmprow,tmpcol;
    Real* phase = readImage(pupil.Phase.c_str(), tmprow,tmpcol);
    d_phase = (Real*)memMngr.borrowCache(sz);
    size_t tmpsz = tmprow*tmpcol*sizeof(Real);

    if(tmpsz!=sz){
      Real* d_phasetmp = (Real*)memMngr.borrowCache(tmpsz);
      gpuErrchk(hipMemcpy(d_phasetmp,phase,tmpsz,hipMemcpyHostToDevice));
      init_cuda_image(objrow, objcol);
      if(tmpsz > sz){
        cudaF(crop,d_phasetmp, d_phase, tmprow, tmpcol);
      }else{
        cudaF(pad,d_phasetmp, d_phase, tmprow, tmpcol);
      }
      memMngr.returnCache(d_phasetmp);
    }
    else {
      gpuErrchk(hipMemcpy(d_phase,phase,sz,hipMemcpyHostToDevice));
    }
    ccmemMngr.returnCache(phase);
  }
  init_cuda_image(row, column);
  cudaF(createWaveFront, d_intensity, d_phase, objectWave_holo, objrow, objcol, (row/oversampling-objrow)/2, (column/oversampling-objcol)/2);
  cudaF(add, objectWave_holo, (complexFormat*)objectWave, 1);
  plt.plotComplex(objectWave_holo, MOD2, 0, 1, "input");
  plt.plotComplex(objectWave_holo, PHASE, 0, 1, "input_phase");
  if(doIteration) phaseRetrieve();
  else propagate(patternWave, objectWave, 0);
  propagate(objectWave_holo, patternWave_holo, 1);
  cudaF(getMod2, patternData_holo, patternWave_holo);
  plt.plotFloat(patternData_holo, MOD, 1, exposurepupil, "holoPattern");
  if(simCCDbit) {
    Real* intensity = readImage("holoPattern.png", row, column);
    hipMemcpy(patternData_holo, intensity, memMngr.getSize(patternData_holo), hipMemcpyHostToDevice);
    ccmemMngr.returnCache(intensity);
    cudaF(applyPoissonNoise_WO, patternData_holo, noiseLevel, devstates, 1);
    cudaF(applyNorm, patternData_holo, 1./exposurepupil);
    cudaF(cudaConvertFO, patternData_holo);
  }
  calcXCorrelation();
  iterate();
};
void holo::iterate(){
  hipMemset(patternWave_obj, 0, memMngr.getSize(patternWave_obj));
  for(int iter = 0; iter < nIter; iter++){
    cudaF(applyModCorr, patternWave_obj, patternWave, xcorrelation);
    cudaF(add, patternWave_obj, patternWave, 1);
    cudaF(applyMod, patternWave_obj, patternData_holo, useBS? beamstop:0, 1, iter, noiseLevel);
    cudaF(add, patternWave_obj, patternWave, -1);
    myCufftExec(*plan, patternWave_obj, patternWave_obj, HIPFFT_BACKWARD);
    cudaF(applyNorm, patternWave_obj, 1./(row*column));
    cudaF(applySupportBar, patternWave_obj, support_holo);
    myCufftExec(*plan, patternWave_obj, patternWave_obj, HIPFFT_FORWARD);
  }
  propagate(patternWave_obj,objectWave_holo,0);
  plt.plotComplex(objectWave_holo, MOD2, 0, 1, "object");
  plt.plotComplex(objectWave_holo, PHASE, 0, 1, "object_phase");
}
