#include "hip/hip_runtime.h"
#include "cudaConfig.hpp"
#include "cudaDefs_h.cu"
#include "imgio.hpp"
#include <hiprand/hiprand_kernel.h>
#include <cub_wrap.hpp>
#include <hipfft/hipfft.h>
cudaVars* cudaVar = 0;
cudaVars* cudaVarLocal = 0;
dim3 numBlocks;
const dim3 threadsPerBlock = 256;
complexFormat *cudaData = 0;
static hipfftHandle *plan = 0, *planR2C = 0;
int3 cuda_imgsz = {0,0,1};
void cuMemManager::c_malloc(void*& ptr, size_t sz) { gpuErrchk(hipMalloc((void**)&ptr, sz)); }
void cuMemManager::c_memset(void*& ptr, size_t sz) { gpuErrchk(hipMemset(ptr, 0, sz)); }
cuMemManager memMngr;
int getCudaRows(){
  return cuda_imgsz.x;
}
int getCudaCols(){
  return cuda_imgsz.y;
}
void myMemcpyH2D(void* d, void* s, size_t sz){
  hipMemcpy(d, s, sz, hipMemcpyHostToDevice);
}
void myMemcpyD2D(void* d, void* s, size_t sz){
  hipMemcpy(d, s, sz, hipMemcpyDeviceToDevice);
}
void myMemcpyD2H(void* d, void* s, size_t sz){
  hipMemcpy(d, s, sz, hipMemcpyDeviceToHost);
}
void clearCuMem(void* ptr, size_t sz){
  hipMemset(ptr, 0, sz);
}
void resize_cuda_image(int rows, int cols){
  cuda_imgsz.x = rows;
  cuda_imgsz.y = cols;
  numBlocks.x=(rows*cols-1)/threadsPerBlock.x+1;
}
void resize_cuda_volumn(int rows, int cols, int layers){
  cuda_imgsz.x = rows;
  cuda_imgsz.y = cols;
  cuda_imgsz.z = layers;
  numBlocks.x=(rows*cols*layers-1)/threadsPerBlock.x+1;
}
void init_cuda_image(int rcolor, Real scale){
  const int sz = sizeof(cudaVars);
  if(!cudaVar){
    cudaVar = (cudaVars*) memMngr.borrowCache(sz);
    myMalloc(cudaVars, cudaVarLocal, 1);
    cudaVarLocal->threshold = 0.5;
    cudaVarLocal->beta_HIO = 1;
    if(scale==scale) cudaVarLocal->scale = scale;
    if(rcolor!=0) cudaVarLocal->rcolor = rcolor;
    else cudaVarLocal->rcolor=65535;
    hipMemcpy(cudaVar, cudaVarLocal, sz, hipMemcpyHostToDevice);
    return;
  }
  if(rcolor!=0) cudaVarLocal->rcolor = rcolor;
  if(scale==scale) cudaVarLocal->scale = scale;
  hipMemcpy(cudaVar, cudaVarLocal, sz, hipMemcpyHostToDevice);
};
size_t getGPUFreeMem(){
    size_t freeBytes, totalBytes;
    hipMemGetInfo(&freeBytes, &totalBytes);
    return freeBytes >> 20;
}
void setThreshold(Real val){
  hipMemcpy(&cudaVar->threshold, &val, sizeof(cudaVarLocal->threshold),hipMemcpyHostToDevice);
}
void* newRand(size_t sz){
  return memMngr.borrowCache(sz * sizeof(hiprandStateMRG32k3a));
}

void gpuerr(){
  gpuErrchk(hipGetLastError());
}

using namespace std;
static int rows_fft, cols_fft, batch_fft;
__device__ __host__ bool rect::isInside(int x, int y){
  if(x > startx && x <= endx && y > starty && y <= endy) return true;
  return false;
}
__device__ __host__ bool C_circle::isInside(int x, int y){
  Real dr = hypot(Real(x-x0),Real(y-y0));
  if(dr < r) return true;
  return false;
}
void init_fft(int rows, int cols, int batch){
  printf("init fft: %d %d, old dim=%d, %d\n", rows, cols, rows_fft, cols_fft);
  if(rows!=rows_fft||cols!=cols_fft||batch_fft!=batch){
    if(!plan){
      plan = new hipfftHandle();
      planR2C = new hipfftHandle();
    }else{
      hipfftDestroy(*plan);
      hipfftDestroy(*planR2C);
    }
    if(cols == 1){
      int dim[2] = {rows, batch};
      hipfftPlanMany ( plan, 1, &rows, dim, 1, rows, dim, 1, rows, FFTformat, batch);
      //hipfftPlan1d ( planR2C, rows, FFTformatR2C, batch);
    }else{
      hipfftPlan2d ( plan, rows, cols, FFTformat);
      hipfftPlan2d ( planR2C, rows, cols, FFTformatR2C);
    }
    cols_fft = cols;
    rows_fft = rows;
    batch_fft = batch;
  }
}
void createPlan(int* handle, int row, int col){
  hipfftPlan2d (handle, row, col, FFTformat);
}
void createPlan1d(int* handle, int n){
  hipfftPlan1d(handle, n, FFTformat, 1);
}
void destroyPlan(int handle){
  hipfftDestroy(handle);
}
void myFFTM(int handle, void* in, void* out){
  myCufftExec(handle, (hipComplex*)in, (hipComplex*)out, HIPFFT_FORWARD);
}
void myIFFTM(int handle, void* in, void* out){
  myCufftExec(handle, (hipComplex*)in, (hipComplex*)out, HIPFFT_BACKWARD);
}
void myFFT(void* in, void* out){
  myCufftExec(*plan, (hipComplex*)in, (hipComplex*)out, HIPFFT_FORWARD);
}
void myIFFT(void* in, void* out){
  myCufftExec(*plan, (hipComplex*)in, (hipComplex*)out, HIPFFT_BACKWARD);
}
void myFFTR2C(void* in, void* out){
  myCufftExecR2C(*planR2C, (Real*)in, (hipComplex*)out);
}
cuFuncTemplate(cudaConvertFO, (T* data, T* out),(data,out==0?data:out),{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= (cuda_row*cuda_column)/2) return;
    int x = index%cuda_row;
    int y = index/cuda_row;
    index = x*cuda_column+y;
    int indexp = (x >= (cuda_row>>1)? x - (cuda_row>>1) : (x + (cuda_row>>1)))*cuda_column + y + (cuda_column>>1);
    T tmp = data[index];
    out[index]=data[indexp];
    out[indexp]=tmp;
    })
template void cudaConvertFO<Real>(Real*, Real*);
template<> void cudaConvertFO<complexFormat>(complexFormat* data, complexFormat* out){
  cudaConvertFOWrap<<<numBlocks, threadsPerBlock>>>(addVar((hipComplex*)data, (hipComplex*)(out==0?data:out)));
}

cuFuncTemplate(transpose, (T* data, T* out),(data,out==0?data:out),{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= (cuda_row*cuda_column)/2) return;
    int indexp = cuda_row*cuda_column - index;
    T tmp = data[index];
    out[index]=data[indexp];
    out[indexp]=tmp;
    })
template void transpose<Real>(Real*, Real*);
template<> void transpose<complexFormat>(complexFormat* data, complexFormat* out){
  transposeWrap<<<numBlocks, threadsPerBlock>>>(addVar((hipComplex*)data, (hipComplex*)(out==0?data:out)));
}

template <typename T1, typename T2>
__global__ void assignValWrap(int cuda_row, int cuda_column, int cuda_height, T1* out, T2* input){
  cuda1Idx()
    out[index] = input[index];
}
template <typename T1, typename T2>
void assignVal(T1* out, T2* input){
  assignValWrap<<<numBlocks,threadsPerBlock>>>(cuda_imgsz.x, cuda_imgsz.y, cuda_imgsz.z, out, input);
}
template void assignVal<Real,Real>(Real*, Real*);
template void assignVal<Real,double>(Real*, double*);
template<> void assignVal<complexFormat, complexFormat>(complexFormat* out, complexFormat* input){
  assignValWrap<<<numBlocks,threadsPerBlock>>>(cuda_imgsz.x, cuda_imgsz.y, cuda_imgsz.z, (hipComplex*)out,(hipComplex*)input);
}

cuFuncTemplate(crop,(T* src, T* dest, int row, int col, Real midx, Real midy),(src,dest,row,col,midx,midy),{
    cudaIdx()
    int shiftx = int(row*midx);
    if(shiftx + cuda_row/2 > row/2) shiftx = (row-cuda_row)/2;
    else if(shiftx - cuda_row/2 < - row/2) shiftx = (cuda_row-row)/2;
    int shifty = int(col*midy);
    if(shifty + cuda_column/2 > col/2) shifty = (row-cuda_column)/2;
    else if(shifty - cuda_column/2 < - col/2) shifty = (cuda_column-col)/2;
    int targetindex = (x+(row-cuda_row)/2+shiftx)*col + y+(col-cuda_column)/2+shifty;
    dest[index] = src[targetindex];
    })
template void crop<Real>(Real*, Real*, int, int, Real, Real);
template<> void crop<complexFormat>(complexFormat* src, complexFormat* dest, int row, int col, Real midx, Real midy){
  cropWrap<<<numBlocks, threadsPerBlock>>>(addVar((hipComplex*)src, (hipComplex*)dest, row, col, midx, midy));
}


cuFuncc(multiplyShift,(complexFormat* object, Real shiftx, Real shifty),(hipComplex* object, Real shiftx, Real shifty),((hipComplex*)object,shiftx,shifty),{
    cudaIdx();
    Real phi = -2*M_PI*(shiftx*(x-cuda_row/2)/cuda_row+shifty*(y-cuda_column/2)/cuda_column);
    hipComplex tmp = {cos(phi),sin(phi)};
    object[index] = hipCmulf(object[index],tmp);
    })

cuFuncc(applyNorm,(complexFormat* data, Real factor),(hipComplex* data, Real factor),((hipComplex*)data,factor),{
    cuda1Idx()
    data[index].x*=factor;
    data[index].y*=factor;
    })
cuFunc(applyNorm,(Real* data, Real factor),(data,factor),{
    cuda1Idx()
    data[index]*=factor;
    })

void shiftWave(complexFormat* wave, Real shiftx, Real shifty){
  myFFT(wave, wave);
  cudaConvertFO(wave);
  multiplyShift(wave, shiftx, shifty);
  cudaConvertFO(wave);
  applyNorm(wave, 1./(cuda_imgsz.x*cuda_imgsz.y));
  myIFFT(wave, wave);
}

cuFuncc(rotateToReal,(complexFormat* data),(hipComplex* data),((hipComplex*)data),{
    cuda1Idx();
    data[index].x = hipCabsf(data[index]);
    data[index].y = 0;
    })

cuFuncc(removeImag,(complexFormat* data),(hipComplex* data),((hipComplex*)data),{
    cuda1Idx();
    data[index].y = 0;
    })

void shiftMiddle(complexFormat* wave){
  cudaConvertFO(wave);
  myFFT(wave, wave);
  rotateToReal(wave);
  applyNorm(wave, 1./(cuda_imgsz.x*cuda_imgsz.y));
  myIFFT(wave, wave);
  cudaConvertFO(wave);
}

__global__ void createGaussKernel(Real* data, int sz, Real sigma){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= sz*sz) return;
  int dx = idx/sz-(sz>>1);
  int dy = (idx%sz)-(sz>>1);
  data[idx] = exp(-Real(dx*dx+dy*dy)/(sigma*sigma));
}

void createGauss(Real* data, int sz, Real sigma){
  createGaussKernel<<<(sz*sz-1)/threadsPerBlock.x+1,threadsPerBlock>>>(data, sz, sigma);
  applyNormWrap<<<(sz*sz-1)/threadsPerBlock.x+1,threadsPerBlock>>> (addVar(data, Real(1./findSum(data))));
}
void applyGaussConv(Real* input, Real* output, Real* gaussMem, Real sigma, int size){
  if(size == 0) size = int(floor(sigma*6))>>1; // r=3 sigma to ensure the contribution outside kernel is negligible (0.01 of the maximum)
  int width = (size<<1)+1;
  createGauss(gaussMem, width, sigma);
  applyConvolution((sq(width-1+16)+(width*width))*sizeof(Real), input, output, gaussMem, size, size);
}

cuFuncc(fillRedundantR2C,(complexFormat* data, complexFormat* dataout, Real factor),(hipComplex* data, hipComplex* dataout, Real factor),((hipComplex*)data,(hipComplex*)dataout,factor),{
    cudaIdx()
    int targetIndex = x*(cuda_column/2+1)+y;
    if(y <= cuda_column/2) {
    dataout[index].x = data[targetIndex].x*factor;
    dataout[index].y = data[targetIndex].y*factor;
    return;
    }
    if(x==0) {
    targetIndex = cuda_column-y;
    }else{
    targetIndex = (cuda_row-x)*(cuda_column/2+1)+cuda_column-y;
    }
    dataout[index].x = data[targetIndex].x*factor;
    dataout[index].y = -data[targetIndex].y*factor;
    })

cuFuncShared(applyConvolution,(Real *input, Real *output, Real* kernel, int kernelwidth, int kernelheight),
    (input,output,kernel,kernelwidth,kernelheight),
    {
    extern __shared__ float tile[];
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int blockindex = threadIdx.x*blockDim.y+threadIdx.y;
    int tilewidth = kernelwidth*2+blockDim.x;
    int tilesize = tilewidth*tilewidth;
    int blocksize = blockDim.x*blockDim.y;
    for(int filltarget = blockindex; filltarget < tilesize; filltarget+=blocksize){
    int fillx = x - threadIdx.x - kernelwidth + filltarget/tilewidth;
    int filly = y - threadIdx.y - kernelwidth + filltarget%tilewidth;
    if(fillx < 0 || filly < 0 || filly >= cuda_column || fillx >= cuda_row) tile[filltarget] = 0;
    else tile[filltarget] = input[fillx*cuda_column+filly];
    }
    if(blockindex < (2*kernelwidth+1)*(2*kernelheight+1)) tile[tilesize+blockindex] = kernel[blockindex];
    if(x >= cuda_row || y >= cuda_column) return;
    int index = x*cuda_column+y;
    __syncthreads();
    int Idx = (threadIdx.x)*(tilewidth) + threadIdx.y;
    int IdxK = 0;
    Real n_output = 0;
    for(int x1 = -kernelwidth; x1 <= kernelwidth; x1++){
      for(int y1 = -kernelheight; y1 <= kernelheight; y1++){
        n_output+=tile[Idx++]*tile[tilesize+IdxK++];
      }
      Idx+=tilewidth-2*kernelheight-1;
    }
    output[index] = n_output;
    })

cuFunc(interpolate,(Real* out, Real* data0, Real* data1, Real dx),(out, data0,data1,dx),{
    cuda1Idx()
    out[index] = data0[index]*(1-dx) + data1[index]*dx;
    })
cuFuncc(interpolate,(complexFormat* out, complexFormat* data0, complexFormat* data1, Real dx),(hipComplex* out, hipComplex* data0, hipComplex* data1, Real dx),((hipComplex*)out, (hipComplex*)data0,(hipComplex*)data1,dx),{
    cuda1Idx()
    out[index].x = data0[index].x*(1-dx) + data1[index].x*dx;
    out[index].y = data0[index].y*(1-dx) + data1[index].y*dx;
    })
cuFunc(adamUpdateV,(Real* v, Real* grad, Real beta2),(v,grad,beta2),{
    cuda1Idx()
    Real tmp = grad[index];
    v[index] = tmp*tmp*(1-beta2) + beta2*v[index];
    })
cuFuncc(adamUpdateV,(Real* v, complexFormat* grad, Real beta2),(Real* v, hipComplex* grad, Real beta2),(v,(hipComplex*)grad,beta2),{
    cuda1Idx()
    Real tmp = grad[index].x;
    v[index] = tmp*tmp*(1-beta2) + beta2*v[index];
    })
cuFuncc(adamUpdate,(complexFormat* xn, complexFormat* m, Real* v, Real lr, Real eps),(hipComplex* xn, hipComplex* m, Real* v, Real lr, Real eps),((hipComplex*)xn,(hipComplex*)m,v,lr,eps),{
    cuda1Idx()
    xn[index].x += lr*m[index].x/(sqrt(v[index])+eps);
    })
cuFuncc(ceiling,(complexFormat* data, Real ceilval),(hipComplex* data, Real ceilval),((hipComplex*)data,ceilval),{
    cuda1Idx()
    Real factor = ceilval/hypot(data[index].x, data[index].y);
    if(factor>1) return;
    data[index].x*=factor;
    data[index].y*=factor;
    })
cuFuncc(multiplyReal,(Real* store, complexFormat* src, complexFormat* target),(Real* store, hipComplex* src, hipComplex* target),(store,(hipComplex*)src,(hipComplex*)target),{
    cuda1Idx();
    store[index] = src[index].x*target[index].x;
    })

cuFuncc(multiply,(complexFormat* src, complexFormat* target),(hipComplex* src, hipComplex* target),((hipComplex*)src,(hipComplex*)target),{
    cuda1Idx()
    src[index] = hipCmulf(src[index], target[index]);
    })

cuFuncc(multiplyConj,(complexFormat* src, complexFormat* target),(hipComplex* src, hipComplex* target),((hipComplex*)src,(hipComplex*)target),{
    cuda1Idx()
    src[index] = hipCmulf(src[index], hipConjf(target[index]));
    })

cuFuncc(multiply,(complexFormat* src, Real* target),(hipComplex* src, Real* target),((hipComplex*)src,target),{
    cuda1Idx()
    src[index].x = src[index].x*target[index];
    src[index].y = src[index].y*target[index];
    })

cuFuncc(forcePositive,(complexFormat* a),(hipComplex* a),((hipComplex*)a),{
    cuda1Idx()
    if(a[index].x<0) a[index].x=0;
    a[index].y = 0;
    })

cuFunc(forcePositive,(Real* a),(a),{
    cuda1Idx()
    if(a[index]<0) a[index]=0;
    })

cuFuncc(multiply,(complexFormat* store, complexFormat* src, complexFormat* target),(hipComplex* store, hipComplex* src, hipComplex* target),((hipComplex*)store,(hipComplex*)src,(hipComplex*)target),{
    cuda1Idx()
    store[index] = hipCmulf(src[index], target[index]);
    })

cuFunc(multiply,(Real* store, Real* src, Real* target),(store,src,target),{
    cuda1Idx()
    store[index] = src[index]* target[index];
    })

cuFuncc(multiplyPropagatePhase,(complexFormat* amp, Real a, Real b),(hipComplex* amp, Real a, Real b),((hipComplex*)amp,a,b),{ // a=z/lambda, b = (lambda/s)^2, s is the image size
    cudaIdx();
    hipComplex phasefactor;
    Real phase = a*sqrt(1-(sq(x-(cuda_row>>1))+sq(y-(cuda_column>>1)))*b);
    phasefactor.x = cos(phase);
    phasefactor.y = sin(phase);
    amp[index] = hipCmulf(amp[index],phasefactor);
    })

cuFuncc(extendToComplex,(Real* a, complexFormat* b),(Real* a, hipComplex* b),(a,(hipComplex*)b),{
    cuda1Idx()
    b[index].x = a[index];
    b[index].y = 0;
    })

cuFunc(add,(Real* a, Real* b, Real c),(a,b,c),{
    cuda1Idx()
    a[index]+=b[index]*c;
    })

cuFunc(add,(Real* store, Real* a, Real* b, Real c),(store, a,b,c),{
    cuda1Idx()
    store[index] = a[index]+b[index]*c;
    })

cuFuncc(createWaveFront,(Real* d_intensity, Real* d_phase, complexFormat* objectWave, int row, int col, int shiftx, int shifty, Real phaseFactor),(Real* d_intensity, Real* d_phase, hipComplex* objectWave, int row, int col, int shiftx, int shifty, Real phaseFactor),(d_intensity,d_phase,(hipComplex*)objectWave,row,col,shiftx,shifty,phaseFactor),{
    cudaIdx()
    int marginx = (cuda_row-row)/2+shiftx;
    int marginy = (cuda_column-col)/2+shifty;
    if(x<marginx || x >= marginx+row || y < marginy || y >= marginy+col){
    objectWave[index].x = objectWave[index].y = 0;
    return;
    }
    int targetindex = (x-marginx)*col + y-marginy;
    Real phase = phaseFactor*sqSum(x-marginx-(row>>1),y-marginy-(col>>1));
    Real mod = d_intensity?sqrtf(max(0.,d_intensity[targetindex])):1;
    if(d_phase){
    phase += (d_phase[targetindex]-0.5)*2*M_PI;
    }
    if(phase){
    objectWave[index].x = mod*cos(phase);
    objectWave[index].y = mod*sin(phase);
    }else{
    objectWave[index].x = mod;
    objectWave[index].y = 0;
    }
})

cuFuncc(createWaveFront,(Real* d_intensity, Real* d_phase, complexFormat* objectWave, Real oversampling, Real shiftx, Real shifty, Real phaseFactor),(Real* d_intensity, Real* d_phase, hipComplex* objectWave, Real oversampling, Real shiftx, Real shifty, Real phaseFactor),(d_intensity,d_phase,(hipComplex*)objectWave,oversampling,shiftx,shifty,phaseFactor),{
    cudaIdx()
    Real marginratio = (1-1./oversampling)/2;
    int marginx = (marginratio+shiftx)*cuda_row;
    int marginy = (marginratio+shifty)*cuda_column;
    if(x<marginx || x >= cuda_row-marginx || y < marginy || y >= cuda_column-marginy){
    objectWave[index].x = objectWave[index].y = 0;
    return;
    }
    int row = ceil(cuda_row/oversampling);
    int col = ceil(cuda_column/oversampling);
    int targetindex = (x-marginx)*col + y-marginy;
    Real phase = phaseFactor*sqSum(x-marginx-(row>>1),y-marginy-(col>>1));
    Real mod = d_intensity?sqrtf(max(0.,d_intensity[targetindex])):1;
    if(d_phase) phase += (d_phase[targetindex]-0.5)*2*M_PI;
    //Real phase = d_phase? (d_phase[targetindex]-0.5) : 0;
    if(phase){
    objectWave[index].x = mod*cos(phase);
    objectWave[index].y = mod*sin(phase);
    }else{
    objectWave[index].x = mod;
    objectWave[index].y = 0;
    }
})

void readComplexWaveFront(const char* intensityFile, const char* phaseFile, Real* &d_intensity, Real* &d_phase, int &objrow, int &objcol){
  size_t sz = 0;
  if(intensityFile) {
    Real* intensity = readImage(intensityFile, objrow, objcol);
    sz = objrow*objcol*sizeof(Real);
    if(!d_intensity) d_intensity = (Real*)memMngr.borrowCache(sz); //use the memory allocated;
    hipMemcpy(d_intensity, intensity, sz, hipMemcpyHostToDevice);
    ccmemMngr.returnCache(intensity);
  }
  if(phaseFile) {
    int tmprow,tmpcol;
    Real* phase = readImage(phaseFile, tmprow,tmpcol);
    if(!intensityFile) {
      sz = tmprow*tmpcol*sizeof(Real);
      objrow = tmprow;
      objcol = tmpcol;
    }
    if(!d_phase) d_phase = (Real*)memMngr.borrowCache(sz);
    size_t tmpsz = tmprow*tmpcol*sizeof(Real);
    if(tmpsz!=sz){
      Real* d_phasetmp = (Real*)memMngr.borrowCache(tmpsz);
      gpuErrchk(hipMemcpy(d_phasetmp,phase,tmpsz,hipMemcpyHostToDevice));
      resize_cuda_image(objrow, objcol);
      if(tmpsz > sz){
        crop(d_phasetmp, d_phase, tmprow, tmpcol);
      }else{
        pad(d_phasetmp, d_phase, tmprow, tmpcol);
      }
      memMngr.returnCache(d_phasetmp);
    }
    else {
      gpuErrchk(hipMemcpy(d_phase,phase,sz,hipMemcpyHostToDevice));
    }

    ccmemMngr.returnCache(phase);
  }
  gpuErrchk(hipGetLastError());
}
cuFunc(initRand,(void* state, unsigned long long seed),(state,seed),{
    cuda1Idx()
    hiprand_init(seed,index,0,(hiprandStateMRG32k3a*)state+index);
    })

cuFunc(applyPoissonNoise_WO,(Real* wave, Real noiseLevel, void* state, Real scale),
    (wave,noiseLevel,state,scale),{
    cuda1Idx()
    if(scale==0) scale = vars->scale;
    wave[index]=scale*(int(wave[index]*vars->rcolor/scale) + hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel)-noiseLevel)/vars->rcolor;
    })
cuFunc(ccdRecord, (uint16_t* data, Real* wave, int noiseLevel, void* state, Real exposure),
    (data,wave,noiseLevel,state,exposure),{
    cuda1Idx()
    int dataint = hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel + vars->rcolor*wave[index]*exposure);
    if(dataint >= vars->rcolor) dataint = vars->rcolor-1;
    data[index] = dataint-noiseLevel;
    });
cuFuncc(ccdRecord, (uint16_t* data, complexFormat* wave, int noiseLevel, void* state, Real exposure),(uint16_t* data, hipComplex* wave, int noiseLevel, void* state, Real exposure),
    (data,(hipComplex*)wave,noiseLevel,state,exposure),{
    cuda1Idx()
    hipComplex tmp = wave[index];
    int dataint = hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel + vars->rcolor*(tmp.x*tmp.x+tmp.y*tmp.y)*exposure);
    if(dataint >= vars->rcolor) dataint = vars->rcolor-1;
    data[index] = dataint-noiseLevel;
    });
cuFunc(ccdRecord, (Real* data, Real* wave, int noiseLevel, void* state, Real exposure, int rcolor),
    (data,wave,noiseLevel,state,exposure, rcolor),{
    cuda1Idx()
    if(rcolor == 0) rcolor = vars->rcolor;
    int dataint = hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel + rcolor*wave[index]*exposure);
    if(dataint >= rcolor) dataint = rcolor-1;
    data[index] = Real(dataint-noiseLevel)/rcolor;
    });
cuFuncc(ccdRecord, (Real* data, complexFormat* wave, int noiseLevel, void* state, Real exposure),(Real* data, hipComplex* wave, int noiseLevel, void* state, Real exposure),
    (data,(hipComplex*)wave,noiseLevel,state,exposure),{
    cuda1Idx()
    hipComplex tmp = wave[index];
    int dataint = hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel + vars->rcolor*(tmp.x*tmp.x+tmp.y*tmp.y)*exposure);
    if(dataint >= vars->rcolor) dataint = vars->rcolor-1;
    data[index] = Real(dataint-noiseLevel)/vars->rcolor;
    });
cuFuncc(ccdRecord, (complexFormat* data, complexFormat* wave, int noiseLevel, void* state, Real exposure),(hipComplex* data, hipComplex* wave, int noiseLevel, void* state, Real exposure),
    ((hipComplex*)data,(hipComplex*)wave,noiseLevel,state,exposure),{
    cuda1Idx()
    hipComplex tmp = wave[index];
    int dataint = hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel + vars->rcolor*(tmp.x*tmp.x+tmp.y*tmp.y)*exposure);
    if(dataint >= vars->rcolor) dataint = vars->rcolor-1;
    data[index].x = Real(dataint-noiseLevel)/vars->rcolor;
    data[index].y = 0;
    });
cuFunc(applyPoissonNoise,(Real* wave, Real noiseLevel, void* state, Real scale),
    (wave,noiseLevel,state,scale),{
    cuda1Idx()
    hiprand_init(1,index,0,&((hiprandStateMRG32k3a*)state)[index]);
    if(scale==0) scale = vars->scale;
    wave[index]+=scale*(hiprand_poisson(&((hiprandStateMRG32k3a*)state)[index], noiseLevel)-noiseLevel)/vars->rcolor;
    })

cuFuncc(getMod,(Real* mod, complexFormat* amp),(Real* mod, hipComplex* amp),(mod,(hipComplex*)amp),{
    cuda1Idx()
    mod[index] = hipCabsf(amp[index]);
    })
cuFuncc(getReal,(Real* mod, complexFormat* amp, Real norm),(Real* mod, hipComplex* amp, Real norm),(mod,(hipComplex*)amp, norm),{
    cuda1Idx()
    mod[index] = amp[index].x*norm;
    })
cuFuncc(addReal,(Real* mod, complexFormat* amp, Real norm),(Real* mod, hipComplex* amp, Real norm),(mod,(hipComplex*)amp, norm),{
    cuda1Idx()
    mod[index] += amp[index].x*norm;
    })
cuFuncc(getImag,(Real* mod, complexFormat* amp),(Real* mod, hipComplex* amp),(mod,(hipComplex*)amp),{
    cuda1Idx()
    mod[index] = amp[index].y;
    })
cuFuncc(assignReal,(Real* mod, complexFormat* amp),(Real* mod, hipComplex* amp),(mod,(hipComplex*)amp),{
    cuda1Idx()
    amp[index].x = mod[index];
    })
cuFuncc(assignImag,(Real* mod, complexFormat* amp),(Real* mod, hipComplex* amp),(mod,(hipComplex*)amp),{
    cuda1Idx()
    amp[index].y = mod[index];
    })
cuFuncc(getMod2,(Real* mod2, complexFormat* amp),(Real* mod2, hipComplex* amp),(mod2,(hipComplex*)amp),{
    cuda1Idx()
    hipComplex tmp = amp[index];
    mod2[index] = tmp.x*tmp.x + tmp.y*tmp.y;
    })
cuFuncc(getMod2,(complexFormat* mod2, complexFormat* amp),(hipComplex* mod2, hipComplex* amp),((hipComplex*)mod2,(hipComplex*)amp),{
    cuda1Idx()
    hipComplex tmp = amp[index];
    mod2[index].x = tmp.x*tmp.x + tmp.y*tmp.y;
    mod2[index].y = 0;
    })
cuFuncc(addMod2,(Real* mod2, complexFormat* amp, Real norm),(Real* mod2, hipComplex* amp, Real norm),(mod2,(hipComplex*)amp,norm),{
    cuda1Idx()
    hipComplex tmp = amp[index];
    mod2[index] += tmp.x*tmp.x*norm + tmp.y*tmp.y*norm;
    })
cuFunc(getMod2,(Real* mod2, Real* mod),(mod2,mod),{
    cuda1Idx()
    mod2[index] = sq(mod[index]);
    })

cuFuncc(bitMap,(Real* store, complexFormat* amp, Real threshold),(Real* store, hipComplex* amp, Real threshold),(store,(hipComplex*)amp, threshold),{
    cuda1Idx()
    if(threshold == 0) threshold = vars->threshold;
    hipComplex tmp = amp[index];
    store[index] = tmp.x*tmp.x+tmp.y*tmp.y > threshold*threshold;
    })

cuFunc(bitMap,(Real* store, Real* amp, Real threshold),(store,amp, threshold),{
    cuda1Idx()
    if(threshold == 0) threshold = vars->threshold;
    store[index] = amp[index] > threshold;
    })

cuFunc(applyThreshold,(Real* store, Real* input, Real threshold),(store,input,threshold),{
    cuda1Idx()
    store[index] = input[index] > threshold? input[index] : 0;
    })

cuFunc(linearConst,(Real* store, Real* data, Real fact, Real shift),(store, data, fact, shift),{
    cuda1Idx();
    store[index] = fact*data[index]+shift;
    })

cuFuncc(applyModAbs,(complexFormat* source, Real* target, void* state),(hipComplex* source, Real* target, void* state),((hipComplex*)source, target, state),{
    cuda1Idx();
    Real mod = hypot(source[index].x, source[index].y);
    Real rat = target[index];
    if(rat > 0) rat = sqrt(rat);
    else rat = 0;
    if(mod==0) {
    Real randphase = state?hiprand_uniform((hiprandStateMRG32k3a*)state + index)*2*M_PI:0;
    source[index].x = rat*cos(randphase);
    source[index].y = rat*sin(randphase);
    return;
    }
    rat /= mod;
    source[index].x *= rat;
    source[index].y *= rat;
    })
cuFuncc(applyModAbsinner,(complexFormat* source, Real* target,  int row, int col, Real norm, void* state),(hipComplex* source, Real* target,  int row, int col, Real norm, void* state),((hipComplex*)source,target,row,col,norm, state),{
    cudaIdx()
    int targetx = x >= cuda_row/2 ? x - (cuda_row - row) : x;
    int targety = y >= cuda_column/2 ? y - (cuda_column - col) : y;
    Real rat = target[index]*norm;
    index = targetx*col+targety;
    Real mod = hypot(source[index].x, source[index].y);
    if(rat > 0) rat = sqrt(rat);
    else rat = 0;
    if(mod==0) {
    Real randphase = state?hiprand_uniform((hiprandStateMRG32k3a*)state+index)*2*M_PI:0;
    source[index].x = rat*cos(randphase);
    source[index].x = rat*sin(randphase);
    return;
    }
    rat /= mod;
    source[index].x *= rat;
    source[index].y *= rat;
    })

cuFuncc(applyMod,(complexFormat* source, Real* target, Real *bs, bool loose, int iter, int noiseLevel),(hipComplex* source, Real* target, Real *bs, bool loose, int iter, int noiseLevel), ((hipComplex*)source, target, bs, loose, iter, noiseLevel),{
    cuda1Idx()
    Real maximum = vars->scale*0.95;
    Real mod2 = target[index];
    if(mod2<0) mod2=0;
    if(loose && bs && bs[index]>0.5) {
    //if(iter > 500) return;
    //else mod2 = maximum+1;
    return;
    }
    Real tolerance = (1.+sqrtf(noiseLevel))*vars->scale/vars->rcolor; // fluctuation caused by bit depth and noise
    hipComplex sourcedata = source[index];
    Real srcmod2 = sourcedata.x*sourcedata.x + sourcedata.y*sourcedata.y;
    if(mod2>=maximum) {
    if(loose) mod2 = max(maximum,srcmod2);
    else tolerance*=1000;
    }
    if(srcmod2 == 0){
    source[index].x = sqrt(mod2);
    source[index].y = 0;
    return;
    }
    Real diff = mod2-srcmod2;
    Real val = mod2;
    if(diff>tolerance){
      val -= tolerance;
    }else if(diff < -tolerance ){
      val += tolerance;
    }
    val = sqrt(val/srcmod2);
    source[index].x = (0.+val)/1.*sourcedata.x;
    source[index].y = (0.+val)/1.*sourcedata.y;
    })
cuFuncc(add,(complexFormat* a, complexFormat* b, Real c ),(hipComplex* a, hipComplex* b, Real c ),((hipComplex*)a,(hipComplex*)b,c),{
    cuda1Idx()
    a[index].x+=b[index].x*c;
    a[index].y+=b[index].y*c;
    })
cuFuncc(convertFOPhase, (complexFormat* data),(hipComplex* data),((hipComplex*)data),{
    cudaIdx()
    if((x+y)%2==1) {
    data[index].x = -data[index].x;
    data[index].y = -data[index].y;
    }
    })
cuFuncc(add,(complexFormat* store, complexFormat* a, complexFormat* b, Real c ),(hipComplex* store, hipComplex* a, hipComplex* b, Real c ),((hipComplex*)store,(hipComplex*)a,(hipComplex*)b,c),{
    cuda1Idx()
    store[index].x=a[index].x + b[index].x*c;
    store[index].y=a[index].y + b[index].y*c;
    })
cuFunc(addRemoveOE, (Real* src, Real* sub, Real mult), (src, sub,mult), {
    cuda1Idx();
    if(sub[index] < 0.99){
    src[index]+=sub[index]*mult;
    }else{
    src[index] = 0;
    }
    });
cuFuncc(applyRandomPhase,(complexFormat* wave, Real* beamstop, void* state),(hipComplex* wave, Real* beamstop, void* state),
    ((hipComplex*)wave, beamstop, state),{
    cuda1Idx()
    hipComplex tmp = wave[index];
    if(beamstop && beamstop[index]>vars->threshold) {
    tmp.x = tmp.y = 0;
    }
    else{
    Real mod = hipCabsf(wave[index]);
    Real randphase = hiprand_uniform((hiprandStateMRG32k3a*)state+index)*2*M_PI;
    tmp.x = mod*cos(randphase);
    tmp.y = mod*sin(randphase);
    }
    wave[index] = tmp;
    })

__device__ hipComplex getFact(Real phase, int l){
  hipComplex nom;
  if(phase != 0){
    nom.x = cos(phase)-1;
    nom.y = sin(phase);
    Real mod2 = sqSum(nom.x, nom.y);
    nom.x = nom.x/mod2;
    nom.y = nom.y/mod2;  //omitted a - sign
    Real nomy = 2*sin(phase*l/2);
    nom.x = nom.y*nomy;
    nom.y = nom.x*nomy;
  }else{
    nom.x = l;
    nom.y = 0;
  }
  return nom;
}
cuFunc(stretch,(Real* src, Real* dest, Real rat, int prec),(src,dest,rat,prec),{
    cudaIdx()
    int targetx = Real(x-cuda_row/2)/rat+cuda_row/2;
    int targety = Real(y-cuda_column/2)/rat+cuda_column/2;
    Real f = cuda_row*cuda_column*rat*rat;
    dest[index] = 0;
    Real sum = 0;
    Real sum1 = 0;
    for(int tx = targetx - prec; tx < targetx+prec; tx++){
    Real phase = 2*M_PI*(Real(x-cuda_row/2)/rat-tx+cuda_row/2)/cuda_row;
    hipComplex factor1 = getFact(phase, cuda_row);
    for(int ty = targety - prec; ty < targety+prec; ty++){
    phase = 2*M_PI*(Real(y-cuda_column/2)/rat-ty+cuda_column/2)/cuda_column;
    hipComplex factor2 = getFact(phase, cuda_row);
    factor2 = hipCmulf(factor1,factor2);
    if(x == 1 && y == 1) {
    sum += factor2.x/f;
    sum1 += factor2.y/f;
    printf("%d, %d, %f, %f\n", tx, ty, factor2.x/f, factor2.y/f);
    }
    dest[index] += src[tx*cuda_row+ty]*factor2.x /f;
    }
    }
    if(x == 1 && y == 1) printf("sum: %f, %f, %f\n", sum, sum1, sqSum(sum, sum1));
})
cuFunc(cropinner,(Real* src, Real* dest, int row, int col, Real norm),(src,dest,row,col,norm),{
    cudaIdx()
    int targetx = x >= cuda_row/2 ? x + row - cuda_row : x;
    int targety = y >= cuda_column/2 ? y + col - cuda_column : y;
    int targetidx = targetx * col + targety;
    dest[index] = src[targetidx]*norm;
    })
cuFunc(mergePixel, (Real* output, Real* input, int row, int col, int nmerge),(output, input, row, col, nmerge),{
    cudaIdx()
    int idx0 = x*nmerge*col+y*nmerge;
    Real out = 0;
    for(int dx = 0; dx < nmerge; dx ++){
    for(int dy = 0; dy < nmerge; dy ++){
    out += input[idx0];
    idx0++;
    }
    idx0+=col-nmerge;
    }
    output[index] = out/(nmerge*nmerge);
    })

cuFuncc(cropinner,(complexFormat* src, complexFormat* dest, int row, int col, Real norm),(hipComplex* src, hipComplex* dest, int row, int col, Real norm),((hipComplex*)src,(hipComplex*)dest,row,col,norm),{
    cudaIdx()
    int targetx = x >= cuda_row/2 ? x + row - cuda_row : x;
    int targety = y >= cuda_column/2 ? y + col - cuda_column : y;
    int targetidx = targetx * col + targety;
    dest[index].x = src[targetidx].x*norm;
    dest[index].y = src[targetidx].y*norm;
    })
cuFunc(padinner,(Real* src, Real* dest, int row, int col, Real norm),(src,dest,row,col,norm),{
    cudaIdx()
    if((x >= row/2 && x < (cuda_row - row/2)) || (y >= col/2 && y < (cuda_column - col/2))){
    dest[index] = 0;
    return;
    }
    int targetx = x >= cuda_row/2 ? x - (cuda_row - row) : x;
    int targety = y >= cuda_column/2 ? y - (cuda_column - col) : y;
    dest[index] = src[targetx*col+targety]*norm;
    })

cuFuncc(padinner, (complexFormat* src, complexFormat* dest, int row, int col, Real norm),(hipComplex* src, hipComplex* dest, int row, int col, Real norm), ((hipComplex*)src, (hipComplex*)dest, row, col, norm),{
    cudaIdx()
    if((x >= row/2 && x < (cuda_row - row/2)) || (y >= col/2 && y < (cuda_column - col/2))){
    dest[index].x = dest[index].y = 0;
    return;
    }
    int targetx = x >= cuda_row/2 ? x - (cuda_row - row) : x;
    int targety = y >= cuda_column/2 ? y - (cuda_column - col) : y;
    int targetidx = targetx*col+targety;
    dest[index].x = src[targetidx].x*norm;
    dest[index].y = src[targetidx].y*norm;
    })

cuFunc(paste, (Real* out, Real* in, int colout, int posx, int posy, bool replace),(out, in, colout, posx, posy, replace),{
    cudaIdx();
    int tidx = (x+posx)*colout + y + posy;
    if(tidx < 0) return;
    Real data = in[index];
    if(!replace) data += out[tidx];
    out[tidx] = data>1?1:data;
    })
cuFuncc(paste, (complexFormat* out, complexFormat* in, int colout, int posx, int posy, bool replace),(hipComplex* out, hipComplex* in, int colout, int posx, int posy, bool replace),((hipComplex*)out, (hipComplex*)in, colout, posx, posy, replace),{
    cudaIdx();
    int tidx = (x+posx)*colout + y + posy;
    hipComplex data = in[index];
    if(!replace) {
    data.x += out[tidx].x;
    data.y += out[tidx].y;
    }
    out[tidx] = data;
    })
//-------experimentConfig.cc-begin
// pixsize*pixsize*M_PI/(d*lambda) and 2*d*M_PI/lambda
cuFuncc(multiplyPatternPhase_Device,(complexFormat* amp, Real r_d_lambda, Real d_r_lambda),(hipComplex* amp, Real r_d_lambda, Real d_r_lambda),((hipComplex*)amp,r_d_lambda,d_r_lambda),{
    cudaIdx()
    Real phase = (sq(x-(cuda_row>>1))+sq(y-(cuda_column>>1)))*r_d_lambda+d_r_lambda;
    hipComplex p = {cos(phase),sin(phase)};
    amp[index] = hipCmulf(amp[index], p);
    })

cuFuncc(multiplyPatternPhaseOblique_Device,(complexFormat* amp, Real r_d_lambda, Real d_r_lambda, Real costheta),(hipComplex* amp, Real r_d_lambda, Real d_r_lambda, Real costheta),((hipComplex*)amp,r_d_lambda,d_r_lambda,costheta),{ // pixsize*pixsize*M_PI/(d*lambda) and 2*d*M_PI/lambda and costheta = z/r
    cudaIdx()
    Real phase = (sq((x-(cuda_row>>1)*costheta))+sq(y-(cuda_column>>1)))*r_d_lambda+d_r_lambda;
    hipComplex p = {cos(phase),sin(phase)};
    amp[index] = hipCmulf(amp[index], p);
    })

cuFuncc(multiplyFresnelPhase_Device,(complexFormat* amp, Real phaseFactor),(hipComplex* amp, Real phaseFactor),((hipComplex*)amp,phaseFactor),{ // pixsize*pixsize*M_PI/(d*lambda) and 2*d*M_PI/lambda
    cudaIdx()
    Real phase = phaseFactor*(sq(x-(cuda_row>>1))+sq(y-(cuda_column>>1)));
    hipComplex p = {cos(phase),sin(phase)};
    if(hipCabsf(amp[index])!=0) amp[index] = hipCmulf(amp[index], p);
    })

cuFuncc(multiplyFresnelPhaseOblique_Device,(complexFormat* amp, Real phaseFactor, Real costheta_r),(hipComplex* amp, Real phaseFactor, Real costheta_r),((hipComplex*)amp,phaseFactor,costheta_r),{ // costheta_r = 1./costheta = r/z
    cudaIdx()
    Real phase = phaseFactor*(sq((x-(cuda_row>>1))*costheta_r)+sq(y-(cuda_column>>1)));
    hipComplex p = {cos(phase),sin(phase)};
    if(hipCabsf(amp[index])!=0) amp[index] = hipCmulf(amp[index], p);
    })

//-------experimentConfig.cc-end

//-------cdi.cc-begin

cuFuncc(takeMod2Diff,(complexFormat* a, Real* b, Real *output, Real *bs),(hipComplex* a, Real* b, Real *output, Real *bs),((hipComplex*)a,b,output,bs),{
    cuda1Idx()
    Real mod2 = sq(a[index].x)+sq(a[index].y);
    Real tmp = b[index]-mod2;
    if(bs&&bs[index]>0.5) tmp=0;
    else if(b[index]>vars->scale) tmp = vars->scale-mod2;
    output[index] = tmp;
    })

cuFuncc(takeMod2Sum,(complexFormat* a, Real* b),(hipComplex* a, Real* b),((hipComplex*)a,b),{
    cuda1Idx()
    Real tmp = b[index]+sq(a[index].x)+sq(a[index].y);
    if(tmp<0) tmp=0;
    b[index] = tmp;
    })
__device__ void ApplyHIOSupport(bool insideS, hipComplex &rhonp1, hipComplex &rhoprime, Real beta){
  if(insideS){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x -= beta*rhoprime.x;
    rhonp1.y -= beta*rhoprime.y;
  }
}
__device__ void ApplyFHIOSupport(bool insideS, hipComplex &rhonp1, hipComplex &rhoprime){
  if(insideS){
    rhonp1.x += 1.9*(rhoprime.x-rhonp1.x);
    rhonp1.y += 1.9*(rhoprime.y-rhonp1.y);
  }else{
    rhonp1.x -= 1.2*rhoprime.x;
    rhonp1.y -= 1.2*rhoprime.y;
  }
}
__device__ void ApplyRAARSupport(bool insideS, hipComplex &rhonp1, hipComplex &rhoprime, Real beta){
  if(insideS){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{

    rhonp1.x = beta*rhonp1.x+(1-2*beta)*rhoprime.x;
    rhonp1.y = beta*rhonp1.y+(1-2*beta)*rhoprime.y;
//    rhonp1.x = beta*(rhonp1.x-rhoprime.x);
//    rhonp1.y = beta*(rhonp1.y-rhoprime.y);
  }
}
__device__ void ApplyPOSERSupport(bool insideS, hipComplex &rhonp1, hipComplex &rhoprime){
  if(insideS && rhoprime.x > 0){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x = 0;
    rhonp1.y = 0;
  }
}
__device__ void ApplyERSupport(bool insideS, hipComplex &rhonp1, hipComplex &rhoprime){
  if(insideS){
    rhonp1.x += 1.9*(rhoprime.x-rhonp1.x);
    rhonp1.y += 1.9*(rhoprime.y-rhonp1.y);
  }else{
    rhonp1.x = 0;
    rhonp1.y = 0;
  }
}
__device__ void ApplyPOSHIOSupport(bool insideS, hipComplex &rhonp1, hipComplex &rhoprime, Real beta){
  if(rhoprime.x > 0 && insideS){
    rhonp1.x += 1.9*(rhoprime.x-rhonp1.x);
    //rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x -= beta*rhoprime.x;
  }
  rhonp1.y -= beta*rhoprime.y;
}
__device__ void ApplyPOS0HIOSupport(bool insideS, hipComplex &rhonp1, hipComplex &rhoprime, Real beta){
  if(rhoprime.x > 0 && insideS){
    rhonp1.x = rhoprime.x;
    //rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x -= beta*rhoprime.x;
  }
  rhonp1.y -= beta*rhoprime.y;
}
cuFuncc(applySupportOblique,(complexFormat *gkp1, complexFormat *gkprime, int algo, Real *spt, int iter, Real fresnelFactor, Real costheta_r),(hipComplex* gkp1, hipComplex* gkprime, int algo, Real *spt, int iter, Real fresnelFactor, Real costheta_r),((hipComplex*)gkp1,(hipComplex*)gkprime,algo,spt,iter,fresnelFactor,costheta_r),{
    cudaIdx()
    bool inside = spt[index] > vars->threshold;
    hipComplex &gkp1data = gkp1[index];
    hipComplex &gkprimedata = gkprime[index];
    if(algo==RAAR) ApplyRAARSupport(inside,gkp1data,gkprimedata,vars->beta_HIO);
    else if(algo==ER) ApplyERSupport(inside,gkp1data,gkprimedata);
    else if(algo==HIO) ApplyHIOSupport(inside,gkp1data,gkprimedata,vars->beta_HIO);
    if(fresnelFactor>1e-4 && iter < 400) {
    if(inside){
    Real phase = M_PI*fresnelFactor*(sq((x-(cuda_row>>1))*costheta_r)+sq(y-(cuda_column>>1)));
    //Real mod = hipCabs(gkp1data);
    Real mod = fabs(gkp1data.x*cos(phase)+gkp1data.y*sin(phase)); //use projection (Error reduction)
    gkp1data.x=mod*cos(phase);
    gkp1data.y=mod*sin(phase);
    }
    }
    })
cuFunc(applySupport,(void *gkp1, void *gkprime, int algo, Real *spt, int iter, Real fresnelFactor),(gkp1,gkprime,algo,spt,iter,fresnelFactor),{
    cudaIdx();
    bool inside = spt[index] > vars->threshold;
    hipComplex &gkp1data = ((hipComplex*)gkp1)[index];
    hipComplex &gkprimedata = ((hipComplex*)gkprime)[index];
    if(algo==RAAR) ApplyRAARSupport(inside,gkp1data,gkprimedata,vars->beta_HIO);
    else if(algo==ER) ApplyERSupport(inside,gkp1data,gkprimedata);
    else if(algo==POSER) ApplyPOSERSupport(inside,gkp1data,gkprimedata);
    else if(algo==POSHIO) ApplyPOSHIOSupport(inside,gkp1data,gkprimedata,vars->beta_HIO);
    else if(algo==HIO) ApplyHIOSupport(inside,gkp1data,gkprimedata,vars->beta_HIO);
    else if(algo==FHIO) ApplyFHIOSupport(inside,gkp1data,gkprimedata);
    if(fresnelFactor>1e-4 && iter < 400) {
    if(inside){
    Real phase = M_PI*fresnelFactor*(sq(x-(cuda_row>>1))+sq(y-(cuda_column>>1)));
    //Real mod = hipCabs(gkp1data);
    Real mod = fabs(gkp1data.x*cos(phase)+gkp1data.y*sin(phase)); //use projection (Error reduction)
    gkp1data.x=mod*cos(phase);
    gkp1data.y=mod*sin(phase);
    }
    }
    })
//-------cdi.cc-end

//-------FISTA.cc-------begin
cuFunc(partialx, (Real* b, Real* p), (b,p),{
    cuda1Idx()
    int x = index/cuda_column;
    Real target;
    if(x == cuda_row-1) target = b[index]-b[index%cuda_column];
    else target = b[index]-b[index+cuda_column];
    //if(fabs(target) > 3e-2) target = 0;
    p[index] = target;
    })
cuFunc(partialy, (Real* b, Real* p), (b,p),{
    cuda1Idx()
    int y = index%cuda_column;
    Real target;
    if(y == cuda_column-1) target = b[index]-b[index-cuda_column+1];
    else target = b[index]-b[index+1];
    //if(fabs(target) > 3e-2) target = 0;
    p[index] = target;
    })
cuFunc(diffMax, (Real* p, Real* q), (p,q),{
    cuda1Idx()
    Real mod = hypot(p[index],q[index]);
    if(mod <= 1) return;
    p[index] /= mod;
    q[index] /= mod;
    })
cuFunc(calcLpq, (Real* out, Real* p, Real* q), (out,p,q),{
    cudaIdx()
    Real tmp = p[index]+q[index];
    if(x >= 1) tmp -= p[index-cuda_column];
    else tmp-=p[index+(cuda_row-1)*cuda_column];
    if(y >= 1) tmp -= q[index-1];
    else tmp-=q[index+cuda_column-1];
    out[index] = tmp;
    })
//-------FISTA.cc-------end
//-------monoChromo.cc-------begin
cuFuncc(updateMomentum,(complexFormat* force, complexFormat* mom, Real dx),(hipComplex* force, hipComplex* mom, Real dx),((hipComplex*)force, (hipComplex*)mom , dx),{
    cuda1Idx()
    Real m = mom[index].x;
    Real f = force[index].x;
    // interpolate with walls
    //if(m * f < 0) m = f*(1-dx);
    //else m = m*dx + f*(1-dx);
    //m = m*dx + f*(1-dx);
    if(m * f < 0) m = f*dx;
    else m = m + f*dx;
    mom[index].x = m;
    })

cuFuncc(overExposureZeroGrad, (complexFormat* deltab, complexFormat* b, int noiseLevel),(hipComplex* deltab, hipComplex* b, int noiseLevel),((hipComplex*)deltab, (hipComplex*)b, noiseLevel),{
    cuda1Idx();
    if(b[index].x >= vars->scale*0.99 && deltab[index].x < 0) deltab[index].x = 0;
    //if(fabs(deltab[index].x)*vars->rcolor < sqrtf(noiseLevel)) deltab[index].x = 0;
    deltab[index].y = 0;
    })

cuFuncc(multiplyPixelWeight, (complexFormat* img, Real* weights),(hipComplex* img, Real* weights),((hipComplex*)img, weights),{
    cudaIdx();
    int shift = max(abs(x+0.5-cuda_row/2), abs(y+0.5-cuda_column/2));
    img[index].x *= weights[shift];
    })
cuFuncc(multiplyReal_inner,(complexFormat* a, complexFormat* b, Real* c, int d),(hipComplex* a, hipComplex* b, Real* c, int d),((hipComplex*)a,(hipComplex*)b,c,d),{
    cudaIdx();
    int removeCent = 50;
    if(x < d || x >= cuda_row - d || y < d || y > cuda_column - d
        ||(abs(x-cuda_row/2) < removeCent && abs(y-cuda_column/2) < removeCent)
      )
    c[index] = 0;
    else c[index] = a[index].x*b[index].x;
    })
cuFuncc(assignRef_d, (complexFormat* wavefront, uint32_t* mmap, complexFormat* rf, int n), (hipComplex* wavefront, uint32_t* mmap, hipComplex* rf, int n),((hipComplex*)wavefront, mmap, (hipComplex*)rf, n), {
    cuda1Idx()
    if(index >= n) return;
    rf[index] = wavefront[mmap[index]];
    })
cuFuncc(expandRef, (complexFormat* rf, complexFormat* amp, uint32_t* mmap, int row, int col, int row0, int col0),(hipComplex* rf, hipComplex* amp, uint32_t* mmap, int row, int col, int row0, int col0),((hipComplex*)rf, (hipComplex*)amp, mmap, row, col, row0, col0),{
    cuda1Idx()
    int idx = mmap[index];
    int x = idx/col0 + (row-row0)/2;
    int y = idx%col0 + (col-col0)/2;
    amp[x*col+y] = rf[index];
    })
cuFuncc(expandRef, (complexFormat* rf, complexFormat* amp, uint32_t* mmap, int row, int col, int row0, int col0, complexFormat a),(hipComplex* rf, hipComplex* amp, uint32_t* mmap, int row, int col, int row0, int col0, hipComplex a),((hipComplex*)rf, (hipComplex*)amp, mmap, row, col, row0, col0, *((hipComplex*)&a)),{
    cuda1Idx()
    int idx = mmap[index];
    int x = idx/col0 + (row-row0)/2;
    int y = idx%col0 + (col-col0)/2;
    amp[x*col+y] = hipCmulf(rf[index],a);
    })
cuFuncc(saveRef, (complexFormat* rf, complexFormat* amp, uint32_t* mmap, int row, int col, int row0, int col0, Real norm),(hipComplex* rf, hipComplex* amp, uint32_t* mmap, int row, int col, int row0, int col0, Real norm),((hipComplex*)rf, (hipComplex*)amp, mmap, row, col, row0, col0, norm),{
    cuda1Idx()
    int idx = mmap[index];
    int x = idx/col0 + (row-row0)/2;
    int y = idx%col0 + (col-col0)/2;
    rf[index].x = amp[x*col+y].x*norm;
    rf[index].y = amp[x*col+y].y*norm;
    })
cuFuncc(saveRef_Real, (complexFormat* rf, complexFormat* amp, uint32_t* mmap, int row, int col, int row0, int col0, int n, Real norm),(hipComplex* rf, hipComplex* amp, uint32_t* mmap, int row, int col, int row0, int col0, int n, Real norm),((hipComplex*)rf, (hipComplex*)amp, mmap, row, col, row0, col0, n, norm),{
    cuda1Idx()
    if(index >= n) return;
    int idx = mmap[index];
    int x = idx/col0 + (row-cuda_row)/2;
    int y = idx%col0 + (col-col0)/2;
    rf[index].x = amp[x*col+y].x*norm;
    rf[index].y = 0;
    })
//-------monoChromo.cc-------end
//-------holo.cc-------begin
cuFuncc(applySupportBarHalf,(complexFormat* img, Real* spt),(hipComplex* img, Real* spt),((hipComplex*)img,spt),{
    cudaIdx();
    int hr = cuda_row>>1;
    int hc = cuda_column>>1;
    if(x > hr) x -= hr;
    else x += hr;
    if(y > hc) y -= hc;
    else y += hc;
    if(spt[index] > vars->threshold || x + y > cuda_row)
    img[index].x = img[index].y = 0;
    })


cuFuncc(applySupportBar_Flip,(complexFormat* img, Real* spt),(hipComplex* img, Real* spt),((hipComplex*)img,spt),{
    cuda1Idx();
    if(spt[index] > vars->threshold){
    img[index].x *= -0.3;
    img[index].y *= -0.3;
    }
    })

cuFuncc(applySupport,(complexFormat* img, Real* spt),(hipComplex* img, Real* spt),((hipComplex*)img,spt),{
    cuda1Idx();
    if(spt[index] < vars->threshold)
    img[index].x = img[index].y = 0;
    })

cuFuncc(dillate, (complexFormat* data, Real* support, int wid, int hit), (hipComplex* data, Real* support, int wid, int hit), ((hipComplex*)data,support,wid,hit),{
    cudaIdx();
    if(abs(data[index].x) < 0.5 && abs(data[index].y) < 0.5) return;
    int idxp = 0;
    for(int xp = 0; xp < cuda_row; xp++)
    for(int yp = 0; yp < cuda_column; yp++)
    {
    if(abs(xp - x) <= wid && abs(yp-y) <= hit) support[idxp] = 0;
    if(abs(x - xp) > cuda_row/2 || abs(y-yp)>cuda_column/2) support[idxp] = 0;
    idxp++;
    }
    })

cuFuncc(applyModCorr, (complexFormat* obj, complexFormat* refer, Real* xcorrelation),(hipComplex* obj ,hipComplex* refer, Real* xcorrelation),((hipComplex*)obj,(hipComplex*)refer,xcorrelation),{
    cuda1Idx();
    hipComplex objtmp = obj[index];
    hipComplex reftmp = refer[index];
    if(reftmp.x == 0 && reftmp.y == 0) return;
    Real fact = xcorrelation[index]/2 - reftmp.x*objtmp.x - reftmp.y*objtmp.y;
    fact /= reftmp.x*reftmp.x + reftmp.y*reftmp.y;
    obj[index].x = objtmp.x + fact*reftmp.x;
    obj[index].y = objtmp.y + fact*reftmp.y;
    })

cuFuncc(devideStar, (complexFormat* obj, complexFormat* refer, complexFormat* xcorrelation),(hipComplex* obj ,hipComplex* refer, hipComplex* xcorrelation),((hipComplex*)obj,(hipComplex*)refer,(hipComplex*)xcorrelation),{
    cuda1Idx();
    hipComplex xctmp = xcorrelation[index];
    hipComplex reftmp = refer[index];
    Real fact = max(sqSum(reftmp.x,reftmp.y),1e-4);
    xctmp = hipCmulf(xctmp, reftmp);
    obj[index].x = xctmp.x / fact;
    obj[index].y = xctmp.y / fact;
    })

//-------holo.cc-------end
cuFuncTemplate(createMask, (Real* data, T* spt, bool isFrequency),(data,spt,isFrequency),{
    cudaIdx()
    if(isFrequency){
    if(x>=cuda_row/2) x-=cuda_row/2;
    else x+=cuda_row/2;
    if(y>=cuda_column/2) y-=cuda_column/2;
    else y+=cuda_column/2;
    }
    data[index]=spt->isInside(x,y);
    })
template void createMask<rect>(Real*, rect*, bool isFrequency);
template void createMask<C_circle>(Real*, C_circle*, bool isFrequency);
cuFuncTemplate(createMaskBar, (Real* data, T* spt, bool isFrequency),(data,spt,isFrequency),{
    cudaIdx()
    if(isFrequency){
    if(x>=cuda_row/2) x-=cuda_row/2;
    else x+=cuda_row/2;
    if(y>=cuda_column/2) y-=cuda_column/2;
    else y+=cuda_column/2;
    }
    data[index]=!spt->isInside(x,y);
    })
template void createMaskBar<rect>(Real*, rect*, bool isFrequency);
template void createMaskBar<C_circle>(Real*, C_circle*, bool isFrequency);
cuFunc(applyMask, (Real* data, Real* mask, Real threshold),(data,mask,threshold),{
    cuda1Idx();
    if(mask[index]<=threshold) data[index] = 0;
    })

cuFuncc(applyMask, (complexFormat* data, Real* mask, Real threshold),(hipComplex* data, Real* mask, Real threshold),((hipComplex*)data,mask,threshold),{
    cuda1Idx();
    if(mask[index]<=threshold) data[index].x = data[index].y = 0;
    })
cuFunc(applyMaskBar, (Real* data, Real* mask, Real threshold),(data,mask,threshold),{
    cuda1Idx();
    if(mask[index]>threshold) data[index] = 0;
    })
cuFuncc(applyMaskBar, (Real* data, complexFormat* mask, Real threshold),(Real* data, hipComplex* mask, Real threshold),(data,(hipComplex*)mask,threshold),{
    cuda1Idx();
    if(mask[index].x>threshold) data[index] = 0;
    })
cuFuncc(applyMaskBar, (complexFormat* data, Real* mask, Real threshold),(hipComplex* data, Real* mask, Real threshold),((hipComplex*)data,mask,threshold),{
    cuda1Idx();
    if(mask[index]>threshold) data[index].x = data[index].y = 0;
    })
cuFuncc(zeroEdge,(complexFormat* a, int n),(hipComplex* a, int n),((hipComplex*)a,n),{
    cudaIdx()
    if(x<n || x>=cuda_row-n || y < n || y >= cuda_column-n)
    a[index] = hipComplex();
    })
cuFunc(zeroEdge,(Real* a, int n),(a,n),{
    cudaIdx()
    if(x<n || x>=cuda_row-n || y < n || y >= cuda_column-n)
    a[index] = 0;
    })
cuFuncc(zeroEdgey,(complexFormat* a, int n),(hipComplex* a, int n),((hipComplex*)a,n),{
    cuda1Idx()
    int y = index%cuda_column;
    if(y < n || y >= cuda_column-n)
    a[index] = hipComplex();
    })

cuFunc(ssimMap,(Real* output, Real* mu1sq, Real* mu2sq, Real* mu1mu2, Real* sigma1sq, Real* sigma2sq, Real* sigma12, Real C1, Real C2),(output, mu1sq, mu2sq, mu1mu2, sigma1sq, sigma2sq, sigma12, C1, C2),{
    cuda1Idx()
    output[index] = (2*mu1mu2[index]+C1)*(2*sigma12[index]+C2)/((mu1sq[index]+mu2sq[index]+C1)*(sigma1sq[index]+sigma2sq[index]+C2));
    if(output[index] > 1) printf("output=(2*%f)*(2*%f)/(%f+%f)(%f+%f)=%f\n", mu1mu2[index], sigma12[index], mu1sq[index], mu2sq[index], sigma1sq[index], sigma2sq[index], output[index]);
    })

cuFuncTemplate(pad,(T* src, T* dest, int row, int col, int shiftx, int shifty),(src, dest, row, col, shiftx, shifty),{
    cudaIdx()
    int marginx = (cuda_row-row)/2+shiftx;
    int marginy = (cuda_column-col)/2+shifty;
    if(x < marginx || x >= row+marginx || y < marginy || y >= col+marginy){
    dest[index] = T();
    return;
    }
    int targetindex = (x-marginx)*col + y-marginy;
    dest[index] = src[targetindex];
    })
template void pad<Real>(Real*, Real*, int, int, int, int);
template<> void pad<complexFormat>(complexFormat* src, complexFormat* dest, int row, int col, int shiftx, int shifty){
  padWrap<<<numBlocks, threadsPerBlock>>>(addVar((hipComplex*)src, (hipComplex*)dest, row, col, shiftx, shifty));
};

cuFuncTemplate(refine,(T* src, T* dest, int refinement),(src,dest,refinement),{
    cudaIdx()
    int indexlu = (x/refinement)*(cuda_row/refinement) + y/refinement;
    int indexld = (x/refinement)*(cuda_row/refinement) + y/refinement+1;
    int indexru = (x/refinement+1)*(cuda_row/refinement) + y/refinement;
    int indexrd = (x/refinement+1)*(cuda_row/refinement) + y/refinement+1;
    Real dx = Real(x%refinement)/refinement;
    Real dy = Real(y%refinement)/refinement;
    dest[index] =
    src[indexlu]*(1-dx)*(1-dy)
    +((y<cuda_column-refinement)?src[indexld]*(1-dx)*dy:0)
    +((x<cuda_row-refinement)?src[indexru]*dx*(1-dy):0)
    +((y<cuda_column-refinement&&x<cuda_row-refinement)?src[indexrd]*dx*dy:0);
    })
template void refine<Real>(Real*, Real*, int);
