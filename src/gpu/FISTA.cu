#include "hip/hip_runtime.h"
#include "tvFilter.h"
#include "memManager.h"
#include "cudaConfig.h"
#include "cuPlotter.h"
#define swap(x, y) \
  auto x##swapTMPVariable = x;\
  x = y;\
  y = x##swapTMPVariable

cuFunc(partialx, (Real* b, Real* p), (b,p),{
  cudaIdx()
  if(x == cuda_row-1) p[index] = 0;
  else p[index] = b[index]-b[index+cuda_column];
})
cuFunc(partialy, (Real* b, Real* p), (b,p),{
  cudaIdx()
  if(y == cuda_column-1) p[index] = 0;
  else p[index] = b[index]-b[index+1];
})
cuFunc(diffMax, (Real* p, Real* q), (p,q),{
  cudaIdx()
  Real mod = hypot(p[index],q[index]);
  if(mod <= 1) return;
  p[index] /= mod;
  q[index] /= mod;
})
cuFunc(calcLpq, (Real* out, Real* p, Real* q), (out,p,q),{
  cudaIdx()
  Real tmp = p[index]+q[index];
  if(y >= 1) tmp -= p[index-cuda_column];
  if(x >= 1) tmp -= q[index-1];
  out[index] = tmp;
})
void FISTA(Real* b, Real* output, Real lambda, int niter, void (applyC)(Real*, Real*)){
  size_t sz = memMngr.getSize(b);
  Real tk = 1;
  Real* pij = (Real*)memMngr.borrowCache(sz);
  Real* qij = (Real*)memMngr.borrowCache(sz);
  Real* lpq = (Real*)memMngr.borrowCache(sz);
  Real* pijprev = (Real*)memMngr.borrowCache(sz);
  Real* qijprev = (Real*)memMngr.borrowCache(sz);
  hipMemset(pij, 0, sz);
  hipMemset(qij, 0, sz);
  applyC(b, output);
  for(int iter = 0; iter < niter ; iter++){
    swap(pij, pijprev);
    swap(qij, qijprev);
    cudaF(applyNorm, output, 0.125/lambda);
    cudaF(partialx, output, pij);
    cudaF(partialy, output, qij);
    cudaF(add, pij, pijprev, 1);
    cudaF(add, qij, qijprev, 1);
    cudaF(diffMax, pij, qij);
    Real tkp1 = 0.5+sqrt(0.25+tk*tk);
    Real fact1 = (tk-1)/tkp1;
    cudaF(applyNorm, pij, 1+fact1);
    cudaF(applyNorm, qij, 1+fact1);
    cudaF(add, pij, pijprev, -fact1);
    cudaF(add, qij, qijprev, -fact1);
    cudaF(calcLpq, lpq, pij, qij);
    cudaF(add, output, b, lpq, -lambda);
    //applyC(output, output);
  }
};
